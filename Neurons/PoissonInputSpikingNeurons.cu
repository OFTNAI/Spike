#include "hip/hip_runtime.h"
#include "PoissonInputSpikingNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"
#include <algorithm> // For random shuffle
using namespace std;


// PoissonInputSpikingNeurons Constructor
PoissonInputSpikingNeurons::PoissonInputSpikingNeurons() {

	random_state_manager = NULL;

	rate = 0;
	rates = NULL;
	
	d_rates = NULL;

}


// PoissonInputSpikingNeurons Destructor
PoissonInputSpikingNeurons::~PoissonInputSpikingNeurons() {

	free(random_state_manager);

	free(rates);

	CudaSafeCall(hipFree(d_rates));

}


int PoissonInputSpikingNeurons::AddGroup(neuron_parameters_struct * group_params){

	int new_group_id = InputSpikingNeurons::AddGroup(group_params);

	poisson_input_spiking_neuron_parameters_struct * poisson_input_spiking_group_params = (poisson_input_spiking_neuron_parameters_struct*)group_params;

	rate = poisson_input_spiking_group_params->rate;

	return new_group_id;

}


void PoissonInputSpikingNeurons::set_up_rates() {

	rates = (float*)realloc(rates, sizeof(float)*total_number_of_neurons);
	for (int i = total_number_of_neurons - number_of_neurons_in_new_group; i < total_number_of_neurons; i++) {
		rates[i] = rate;
	}

	total_number_of_transformations_per_object = 1;
	total_number_of_objects = 1;
	total_number_of_input_stimuli = 1;
}


void PoissonInputSpikingNeurons::setup_random_states_on_device() {


	
	random_state_manager = new RandomStateManager();

	random_state_manager->setup_random_states();
}

void PoissonInputSpikingNeurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps, bool high_fidelity_spike_storage) {

	InputSpikingNeurons::allocate_device_pointers(maximum_axonal_delay_in_timesteps, high_fidelity_spike_storage);

	CudaSafeCall(hipMalloc((void **)&d_rates, sizeof(float)*total_number_of_neurons));

}

void PoissonInputSpikingNeurons::copy_constants_to_device() {
	InputSpikingNeurons::copy_constants_to_device();

	if (rates != NULL) {
		CudaSafeCall(hipMemcpy(d_rates, rates, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
	}
}


void PoissonInputSpikingNeurons::reset_neuron_activities() {

	InputSpikingNeurons::reset_neuron_activities();

}


void PoissonInputSpikingNeurons::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	InputSpikingNeurons::set_threads_per_block_and_blocks_per_grid(threads);

}


void PoissonInputSpikingNeurons::update_membrane_potentials(float timestep, float current_time_in_seconds) {

	poisson_update_membrane_potentials_kernel<<<random_state_manager->block_dimensions, random_state_manager->threads_per_block>>>(random_state_manager->d_states,
														d_rates,
														d_membrane_potentials_v,
														timestep,
														d_thresholds_for_action_potential_spikes,
														total_number_of_neurons,
														current_stimulus_index);

	CudaCheckError();
}


__global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states,
							float *d_rates,
							float *d_membrane_potentials_v,
							float timestep,
							float * d_thresholds_for_action_potential_spikes,
							size_t total_number_of_input_neurons,
							int current_stimulus_index) {

	 
	int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idx = t_idx;
	while (idx < total_number_of_input_neurons){

		int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;

		float rate = d_rates[rate_index];

		if (rate > 0.1) {

			// Creates random float between 0 and 1 from uniform distribution
			// d_states effectively provides a different seed for each thread
			// hiprand_uniform produces different float every time you call it
			float random_float = hiprand_uniform(&d_states[t_idx]);
			
			// if the randomnumber is less than the rate
			if (random_float < (rate * timestep)) {

				// Puts membrane potential above default spiking threshold
				d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;

			} 

		}

		idx += blockDim.x * gridDim.x;

	}
	__syncthreads();
}

