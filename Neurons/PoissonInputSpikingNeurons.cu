#include "hip/hip_runtime.h"
#include "PoissonInputSpikingNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"
#include <algorithm> // For random shuffle
using namespace std;


// PoissonInputSpikingNeurons Constructor
PoissonInputSpikingNeurons::PoissonInputSpikingNeurons() {
	rates = NULL;
	d_rates = NULL;

}


// PoissonInputSpikingNeurons Destructor
PoissonInputSpikingNeurons::~PoissonInputSpikingNeurons() {

}


int PoissonInputSpikingNeurons::AddGroup(neuron_parameters_struct * group_params){

	int new_group_id = InputSpikingNeurons::AddGroup(group_params);

	poisson_input_spiking_neuron_parameters_struct * poisson_input_spiking_group_params = (poisson_input_spiking_neuron_parameters_struct*)group_params;

	rates = (float*)realloc(rates, sizeof(float)*total_number_of_neurons);
	for (int i = total_number_of_neurons - number_of_neurons_in_new_group; i < total_number_of_neurons; i++) {
		rates[i] = poisson_input_spiking_group_params->rate;
	}

	return CORRECTED_PRESYNAPTIC_ID(new_group_id, true);

}

void PoissonInputSpikingNeurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps, bool high_fidelity_spike_storage) {

	InputSpikingNeurons::allocate_device_pointers(maximum_axonal_delay_in_timesteps, high_fidelity_spike_storage);

	CudaSafeCall(hipMalloc((void **)&d_rates, sizeof(float)*total_number_of_neurons));

}

void PoissonInputSpikingNeurons::copy_constants_to_device() {
	InputSpikingNeurons::copy_constants_to_device();

	CudaSafeCall(hipMemcpy(d_rates, rates, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
}


void PoissonInputSpikingNeurons::reset_neurons() {

	InputSpikingNeurons::reset_neurons();

	
}


void PoissonInputSpikingNeurons::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	InputSpikingNeurons::set_threads_per_block_and_blocks_per_grid(threads);

}

int* PoissonInputSpikingNeurons::setup_stimuli_presentation_order(Stimuli_Presentation_Struct * stimuli_presentation_params) {
	
	int* stimuli_presentation_order = InputSpikingNeurons::setup_stimuli_presentation_order(stimuli_presentation_params);

	return stimuli_presentation_order;
}


bool PoissonInputSpikingNeurons::stimulus_is_new_object_for_object_by_object_presentation(int stimulus_index) {
	print_message_and_exit("Object by object presentation currently unsupported at PoissonInputSpikingNeurons level. Please use ImagePoissonInputSpikingNeurons.");
	return false;
}


void PoissonInputSpikingNeurons::update_membrane_potentials(float timestep) {

	poisson_update_membrane_potentials_kernel<<<RandomStateManager::instance()->block_dimensions, RandomStateManager::instance()->threads_per_block>>>(RandomStateManager::instance()->d_states,
														d_rates,
														d_membrane_potentials_v,
														timestep,
														d_thresholds_for_action_potential_spikes,
														total_number_of_neurons,
														current_stimulus_index);

	CudaCheckError();
}


__global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states,
							float *d_rates,
							float *d_membrane_potentials_v,
							float timestep,
							float * d_thresholds_for_action_potential_spikes,
							size_t total_number_of_input_neurons,
							int current_stimulus_index) {

	 
	int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idx = t_idx;
	while (idx < total_number_of_input_neurons){

		int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;

		float rate = d_rates[rate_index];

		if (rate > 0.1) {

			// Creates random float between 0 and 1 from uniform distribution
			// d_states effectively provides a different seed for each thread
			// hiprand_uniform produces different float every time you call it
			float random_float = hiprand_uniform(&d_states[t_idx]);
			
			// if the randomnumber is less than the rate
			if (random_float < (rate * timestep)) {

				// Puts membrane potential above default spiking threshold
				d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;

			} 

		}

		idx += blockDim.x * gridDim.x;

	}
	__syncthreads();
}

