#include "hip/hip_runtime.h"
#include "LIFSpikingNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"


// LIFSpikingNeurons Constructor
LIFSpikingNeurons::LIFSpikingNeurons() {
	// param_a = NULL;
	// param_b = NULL;

	// d_param_a = NULL;
	// d_param_b = NULL;
}


// LIFSpikingNeurons Destructor
LIFSpikingNeurons::~LIFSpikingNeurons() {

}


int LIFSpikingNeurons::AddGroup(neuron_parameters_struct * group_params, int group_shape[2]){

	int new_group_id = SpikingNeurons::AddGroup(group_params, group_shape);

	lif_spiking_neuron_parameters_struct * lif_spiking_group_params = (lif_spiking_neuron_parameters_struct*)group_params;

	// param_a = (float*)realloc(param_a, (total_number_of_neurons*sizeof(float)));
	// param_b = (float*)realloc(param_b, (total_number_of_neurons*sizeof(float)));

	for (int i = total_number_of_neurons - number_of_neurons_in_new_group; i < total_number_of_neurons; i++) {
		// param_a[i] = izhikevich_spiking_group_params->parama;
		// param_b[i] = izhikevich_spiking_group_params->paramb;
	}

	return new_group_id;
}


void LIFSpikingNeurons::initialise_device_pointers() {
 	
 	SpikingNeurons::initialise_device_pointers();

 	// CudaSafeCall(hipMalloc((void **)&d_param_a, sizeof(float)*total_number_of_neurons));
 	// CudaSafeCall(hipMalloc((void **)&d_param_b, sizeof(float)*total_number_of_neurons));
 	
}

void LIFSpikingNeurons::reset_neurons() {

	SpikingNeurons::reset_neurons();	

	// CudaSafeCall(hipMemcpy(d_param_a, param_a, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
	// CudaSafeCall(hipMemcpy(d_param_b, param_b, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
}


__global__ void lif_update_membrane_potentials(float *d_membrane_potentials_v,
								float* d_current_injections,
								float timestep,
								size_t total_number_of_neurons);


void LIFSpikingNeurons::update_membrane_potentials(float timestep) {

	lif_update_membrane_potentials<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(d_membrane_potentials_v,
																	d_current_injections,
																	timestep,
																	total_number_of_neurons);

	CudaCheckError();
}


// State Update
__global__ void lif_update_membrane_potentials(float *d_membrane_potentials_v,
								float* d_current_injections,
								float timestep,
								size_t total_number_of_neurons){

	
	// // Get thread IDs
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < total_number_of_neurons) {

		// We require the equation timestep in ms:
		float eqtimestep = timestep*1000.0f;

		float membrane_potential_Vi = d_membrane_potentials_v[idx];
		float current_injection_Ii = d_current_injections[idx];
		float temp_resting_potential_V0 = -74.0; // Same as after_spike_reset_membrane_potential ???
		float temp_membrane_resistance_R = 40000000.0f;
		float new_membrane_potential = eqtimestep * (temp_resting_potential_V0 - temp_membrane_resistance_R * current_injection_Ii) + (1 - eqtimestep) * membrane_potential_Vi;

		// if (idx == 10) {
		// 	printf("%f\n", new_membrane_potential);
		// }

		d_membrane_potentials_v[idx] = new_membrane_potential;

	}
	__syncthreads();
}


