#include "hip/hip_runtime.h"
#include "GeneratorSpikingNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include <algorithm> // For random shuffle
using namespace std;


// GeneratorSpikingNeurons Constructor
GeneratorSpikingNeurons::GeneratorSpikingNeurons() {
	neuron_id_matrix_for_stimuli = NULL;
	spike_times_matrix_for_stimuli = NULL;
	number_of_spikes_in_stimuli = NULL;

	d_neuron_ids_for_stimulus = NULL;
	d_spike_times_for_stimulus = NULL;

	length_of_longest_stimulus = 0;
}


// GeneratorSpikingNeurons Destructor
GeneratorSpikingNeurons::~GeneratorSpikingNeurons() {
	free(neuron_id_matrix_for_stimuli);
	free(spike_times_matrix_for_stimuli);
	free(number_of_spikes_in_stimuli);
	CudaSafeCall(hipFree(d_neuron_ids_for_stimulus));
	CudaSafeCall(hipFree(d_spike_times_for_stimulus));
}

// Add Group of given size as usual - nothing special in constructor
int GeneratorSpikingNeurons::AddGroup(neuron_parameters_struct * group_params){
	
	int new_group_id = InputSpikingNeurons::AddGroup(group_params);
	return CORRECTED_PRESYNAPTIC_ID(new_group_id, true);

}

// Allocate device pointers for the longest stimulus so that they do not need to be replaced
void GeneratorSpikingNeurons::allocate_device_pointers() {

	InputSpikingNeurons::allocate_device_pointers();

	CudaSafeCall(hipMalloc((void **)&d_neuron_ids_for_stimulus, sizeof(int)*length_of_longest_stimulus));
	CudaSafeCall(hipMalloc((void **)&d_spike_times_for_stimulus, sizeof(float)*length_of_longest_stimulus));
}


void GeneratorSpikingNeurons::reset_neurons() {
	CudaSafeCall(hipMemcpy(d_neuron_ids_for_stimulus, neuron_id_matrix_for_stimuli[current_stimulus_index], sizeof(int)*number_of_spikes_in_stimuli[current_stimulus_index], hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_spike_times_for_stimulus, spike_times_matrix_for_stimuli[current_stimulus_index], sizeof(float)*number_of_spikes_in_stimuli[current_stimulus_index], hipMemcpyHostToDevice));
}

void GeneratorSpikingNeurons::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	InputSpikingNeurons::set_threads_per_block_and_blocks_per_grid(threads);

	int genblocknum = (length_of_longest_stimulus + threads) / threads;
	number_of_neuron_blocks_per_grid.x = genblocknum;
}

void GeneratorSpikingNeurons::check_for_neuron_spikes(float current_time_in_seconds, float timestep) {

	check_for_generator_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(
		d_neuron_ids_for_stimulus,
		d_spike_times_for_stimulus,
		d_last_spike_time_of_each_neuron,
		current_time_in_seconds,
		timestep,
		number_of_spikes_in_stimuli[current_stimulus_index]);

	CudaCheckError();
}

void GeneratorSpikingNeurons::update_membrane_potentials(float timestep){
}

void GeneratorSpikingNeurons::AddStimulus(int spikenumber, int* ids, float* spiketimes){

	++total_number_of_input_stimuli;
	// If the number of spikes in this stimulus is larger than any other ...
	if (spikenumber > length_of_longest_stimulus){
		length_of_longest_stimulus = spikenumber;
	}

	number_of_spikes_in_stimuli = (int*)realloc(number_of_spikes_in_stimuli, sizeof(int)*total_number_of_input_stimuli);
	neuron_id_matrix_for_stimuli = (int**)realloc(neuron_id_matrix_for_stimuli, sizeof(int*)*total_number_of_input_stimuli);
	spike_times_matrix_for_stimuli = (float**)realloc(spike_times_matrix_for_stimuli, sizeof(float*)*total_number_of_input_stimuli);
	
	// Initialize matrices
	neuron_id_matrix_for_stimuli[total_number_of_input_stimuli - 1] = NULL;
	spike_times_matrix_for_stimuli[total_number_of_input_stimuli - 1] = NULL;
	number_of_spikes_in_stimuli[total_number_of_input_stimuli - 1] = 0;
	
	neuron_id_matrix_for_stimuli[total_number_of_input_stimuli - 1] = (int*)realloc(
		neuron_id_matrix_for_stimuli[total_number_of_input_stimuli - 1], 
		sizeof(int)*(spikenumber));
	spike_times_matrix_for_stimuli[total_number_of_input_stimuli - 1] = (float*)realloc(
		spike_times_matrix_for_stimuli[total_number_of_input_stimuli - 1], 
		sizeof(float)*(spikenumber));

	
	// Assign the genid values according to how many neurons exist already
	for (int i = 0; i < spikenumber; i++){
		spike_times_matrix_for_stimuli[total_number_of_input_stimuli - 1][i] = ids[i];
		spike_times_matrix_for_stimuli[total_number_of_input_stimuli - 1][i] = spiketimes[i];
	}
	// Increment the number of entries the generator population
	number_of_spikes_in_stimuli[total_number_of_input_stimuli - 1] = spikenumber;
	
}

// Spiking Neurons
__global__ void check_for_generator_spikes_kernel(int *d_neuron_ids_for_stimulus,
								float *d_spike_times_for_stimulus,
								float* d_last_spike_time_of_each_neuron,
								float current_time_in_seconds,
								float timestep,
								size_t number_of_spikes_in_stimulus) {

	// Get thread IDs
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < number_of_spikes_in_stimulus) {
		if (fabs(current_time_in_seconds - d_spike_times_for_stimulus[idx]) < 0.5 * timestep) {
			__syncthreads();
			d_last_spike_time_of_each_neuron[idx] = current_time_in_seconds;
		}

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}
