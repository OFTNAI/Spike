#include "hip/hip_runtime.h"
#include "SpikingNeurons.h"
#include <stdlib.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"


// SpikingNeurons Constructor
SpikingNeurons::SpikingNeurons() {

	after_spike_reset_membrane_potentials_c = NULL;
	thresholds_for_action_potential_spikes = NULL;

	d_last_spike_time_of_each_neuron = NULL;
	d_membrane_potentials_v = NULL;
	d_thresholds_for_action_potential_spikes = NULL;
	d_resting_potentials = NULL;

	recent_postsynaptic_activities_D = NULL;
	d_recent_postsynaptic_activities_D = NULL;

}


// SpikingNeurons Destructor
SpikingNeurons::~SpikingNeurons() {
	free(recent_postsynaptic_activities_D);
	CudaSafeCall(hipFree(d_recent_postsynaptic_activities_D));
}


int SpikingNeurons::AddGroup(neuron_parameters_struct * group_params){
	
	int new_group_id = Neurons::AddGroup(group_params);

	spiking_neuron_parameters_struct * spiking_group_params = (spiking_neuron_parameters_struct*)group_params;

	after_spike_reset_membrane_potentials_c = (float*)realloc(after_spike_reset_membrane_potentials_c, (total_number_of_neurons*sizeof(float)));
	thresholds_for_action_potential_spikes = (float*)realloc(thresholds_for_action_potential_spikes, (total_number_of_neurons*sizeof(float)));
	recent_postsynaptic_activities_D = (float*)realloc(recent_postsynaptic_activities_D, (total_number_of_neurons*sizeof(float)));

	for (int i = total_number_of_neurons - number_of_neurons_in_new_group; i < total_number_of_neurons; i++) {
		after_spike_reset_membrane_potentials_c[i] = spiking_group_params->resting_potential_v0;
		thresholds_for_action_potential_spikes[i] = spiking_group_params->threshold_for_action_potential_spike;

		//LIF extra
		recent_postsynaptic_activities_D[i] = 0.0f;
	}

	return new_group_id;
}


void SpikingNeurons::allocate_device_pointers() {

	Neurons::allocate_device_pointers();

	CudaSafeCall(hipMalloc((void **)&d_last_spike_time_of_each_neuron, sizeof(float)*total_number_of_neurons));

	CudaSafeCall(hipMalloc((void **)&d_membrane_potentials_v, sizeof(float)*total_number_of_neurons));
	CudaSafeCall(hipMalloc((void **)&d_thresholds_for_action_potential_spikes, sizeof(float)*total_number_of_neurons));
	CudaSafeCall(hipMalloc((void **)&d_resting_potentials, sizeof(float)*total_number_of_neurons));

 	//LIF extra
 	CudaSafeCall(hipMalloc((void **)&d_recent_postsynaptic_activities_D, sizeof(float)*total_number_of_neurons));
}

void SpikingNeurons::reset_neurons() {

	Neurons::reset_neurons();

	CudaSafeCall(hipMemset(d_last_spike_time_of_each_neuron, -1000.0f, total_number_of_neurons*sizeof(float)));

	CudaSafeCall(hipMemcpy(d_membrane_potentials_v, after_spike_reset_membrane_potentials_c, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_thresholds_for_action_potential_spikes, thresholds_for_action_potential_spikes, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_resting_potentials, after_spike_reset_membrane_potentials_c, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));

	//LIF extra
	CudaSafeCall(hipMemcpy(d_recent_postsynaptic_activities_D, recent_postsynaptic_activities_D, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
}


void SpikingNeurons::update_membrane_potentials(float timestep) {
	
}

void SpikingNeurons::update_postsynaptic_activities(float timestep, float current_time_in_seconds) {
	
}


void SpikingNeurons::check_for_neuron_spikes(float current_time_in_seconds) {

	check_for_neuron_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(d_membrane_potentials_v,
																	d_thresholds_for_action_potential_spikes,
																	d_resting_potentials,
																	d_last_spike_time_of_each_neuron,
																	current_time_in_seconds,
																	total_number_of_neurons);

	CudaCheckError();
}


// Spiking Neurons
__global__ void check_for_neuron_spikes_kernel(float *d_membrane_potentials_v,
								float *d_thresholds_for_action_potential_spikes,
								float *d_resting_potentials,
								float* d_last_spike_time_of_each_neuron,
								float current_time_in_seconds,
								size_t total_number_of_neurons) {

	// Get thread IDs
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_neurons) {

		if (d_membrane_potentials_v[idx] >= d_thresholds_for_action_potential_spikes[idx]) {

			// Set current time as last spike time of neuron
			d_last_spike_time_of_each_neuron[idx] = current_time_in_seconds;

			// Reset membrane potential
			d_membrane_potentials_v[idx] = d_resting_potentials[idx];

		}

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();

}
