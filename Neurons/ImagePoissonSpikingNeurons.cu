#include "ImagePoissonSpikingNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include <algorithm> // For random shuffle

#include <fstream>
#include <sstream>
#include <string>
#include "FstreamWrapper.h"


// ImagePoissonSpikingNeurons Constructor
ImagePoissonSpikingNeurons::ImagePoissonSpikingNeurons(const char * fileList, const char * filterParameters, const char * inputDirectory) {

	//JI
	total_number_of_input_images = 0;
	total_number_of_transformations_per_object = 0;
	total_number_of_objects = 0;

	total_number_of_phases = 0;
	total_number_of_wavelengths = 0;
	total_number_of_orientations = 0;
	total_number_of_gabor_types = 0;

	image_width = 0;

	total_number_of_rates = 0;
	total_number_of_rates_per_image = 0;

	group_ids = NULL;


	//OLD VARIABLES


	filterPhases = new vector<float>();
	filterWavelengths = new vector<int>();
	filterOrientations = new vector<float>();
	// buffer = new vector<vector<vector<vector<float> > > >();
	

	set_up_rates(fileList, filterParameters, inputDirectory);

	image_poisson_spiking_neuron_parameters_struct * image_poisson_spiking_group_params = new image_poisson_spiking_neuron_parameters_struct();
	image_poisson_spiking_group_params->rate = 30.0f;

	AddGroupForEachInputImage(image_poisson_spiking_group_params);



}


// ImagePoissonSpikingNeurons Destructor
ImagePoissonSpikingNeurons::~ImagePoissonSpikingNeurons() {

}


int ImagePoissonSpikingNeurons::AddGroup(neuron_parameters_struct * group_params, int group_shape[2]){

	int new_group_id = PoissonSpikingNeurons::AddGroup(group_params, group_shape);

	image_poisson_spiking_neuron_parameters_struct * image_poisson_spiking_group_params = (image_poisson_spiking_neuron_parameters_struct*)group_params;

	for (int i = total_number_of_neurons - number_of_neurons_in_new_group; i < total_number_of_neurons; i++) {

	}

	return new_group_id;

}


void ImagePoissonSpikingNeurons::AddGroupForEachInputImage(neuron_parameters_struct * group_params) {

	int group_shape[] = {image_width, image_width};

	group_ids = (int *)malloc(total_number_of_input_images*sizeof(int));

	image_poisson_spiking_neuron_parameters_struct * image_poisson_spiking_group_params = (image_poisson_spiking_neuron_parameters_struct*)group_params;

	for (int input_image_index = 0; input_image_index < total_number_of_input_images; input_image_index++) {
		image_poisson_spiking_group_params->input_image_index = input_image_index;
		int new_group_id = this->AddGroup(image_poisson_spiking_group_params, group_shape);
		group_ids[input_image_index] = new_group_id;
	}

}


void ImagePoissonSpikingNeurons::allocate_device_pointers() {

	PoissonSpikingNeurons::allocate_device_pointers();

}


void ImagePoissonSpikingNeurons::reset_neurons() {

	PoissonSpikingNeurons::reset_neurons();

}

void ImagePoissonSpikingNeurons::set_up_rates(const char * fileList, const char * filterParameters, const char * inputDirectory) {
	printf("Setting up Input Neuron Rates from gbo files...\n");

	load_image_names_from_file_list(fileList, inputDirectory);
	load_gabor_filter_parameters(filterParameters, inputDirectory);
	load_rates_from_files(inputDirectory);
	copy_rates_to_device();

	printf("\n");
}


void ImagePoissonSpikingNeurons::load_image_names_from_file_list(const char * fileList, const char * inputDirectory) {
    
	// Open file list
	stringstream path;
	path << inputDirectory << '/' << fileList;
	string path_string = path.str();
	
	ifstream fileListStream;
	fileListStream.open(path_string);

	if(fileListStream.fail()) {
		stringstream s;
		s << "Unable to open " << path_string << " for input." << endl;
		cerr << s.str();
		exit(EXIT_FAILURE);
	}
	
	string dirNameBase;						// The "shapeS1T2" part of "shapeS1T2.png"
	int filesLoaded = 0;
	int lastNrOfTransformsFound = 0; // For validation of file list
	
	// cout << "Reading file list:" << endl;
	
	while(getline(fileListStream, dirNameBase)) { 	// Read line from file list

		// printf("total_number_of_transformations_per_object: %d\n", total_number_of_transformations_per_object);
		
		if(dirNameBase.compare("") == 0) {
			continue; // Last line may just be empty bcs of matlab script, should be break; really, but what the hell		
		} else if(dirNameBase.compare("*") == 0) {	
			if(lastNrOfTransformsFound != 0 && lastNrOfTransformsFound != total_number_of_transformations_per_object) {
				cerr << "Number of transforms varied in file list" << endl;
				exit(EXIT_FAILURE);
			}
				
			total_number_of_objects++;
			lastNrOfTransformsFound = total_number_of_transformations_per_object;
			total_number_of_transformations_per_object = 0;
			
			continue;
		} else {
			filesLoaded++;
			total_number_of_transformations_per_object++;
		}
		
		// cout << "#" << filesLoaded << " Loading: " << dirNameBase << endl;
		
		inputNames.push_back(dirNameBase);
	}
	
	total_number_of_transformations_per_object = lastNrOfTransformsFound;
	
	cout << "Objects: " << total_number_of_objects << ", Transforms per Object: " << total_number_of_transformations_per_object << "..." << endl << endl;
	
	total_number_of_input_images = total_number_of_objects * total_number_of_transformations_per_object;
}


void ImagePoissonSpikingNeurons::load_gabor_filter_parameters(const char * filterParameters, const char * inputDirectory) {


	// cout << "Reading filter parameters:" << endl;

	// Open filterParameters
	stringstream path;
	path << inputDirectory << '/' << filterParameters;
	string path_string = path.str();
	
	ifstream filterParametersStream;
	filterParametersStream.open(path_string);

	if(filterParametersStream.fail()) {
		stringstream s;
		s << "Unable to open " << path_string << " for input." << endl;
		cerr << s.str();
		exit(EXIT_FAILURE);
	}

	string dirNameBase;

	int line_index = 0;
	while(getline(filterParametersStream, dirNameBase)) {

		cout << dirNameBase << endl;

		stringstream lineStream(dirNameBase);

		int num;
		while (lineStream.str().size() != 0) {

			if ((lineStream.peek() == ',') || (lineStream.peek() == '[') || (lineStream.peek() == ' ')) {
				lineStream.ignore();
			} else if (lineStream.peek() == ']') {
				break;
			} else {

				lineStream >> num;

				switch (line_index) {
					case 0:
						filterPhases->push_back((float)num);
						break;

					case 1:
						filterWavelengths->push_back(num);
						break;

					case 2:
						filterOrientations->push_back((float)num);
						break;	
					case 3:
						image_width = num;
						break; 
				}

			}	
		} 

		line_index++;

	}

	total_number_of_phases = filterPhases->size();
	total_number_of_wavelengths = filterWavelengths->size();
	total_number_of_orientations = filterOrientations->size();
	total_number_of_gabor_types = total_number_of_phases*total_number_of_wavelengths*total_number_of_orientations;

	total_number_of_rates_per_image = total_number_of_gabor_types * image_width * image_width;
	total_number_of_rates = total_number_of_input_images * total_number_of_rates_per_image;

	printf("\ntotal_number_of_rates: %d\n\n", total_number_of_rates);
}


void ImagePoissonSpikingNeurons::load_rates_from_files(const char * inputDirectory) {


	input_rates = (float *)malloc(total_number_of_rates*sizeof(float));

	for(int image_index = 0; image_index < total_number_of_input_images; image_index++) {

		int image_starting_index = image_index * total_number_of_rates_per_image;
		
		cout << "Loading Rates for Image #" << image_index << endl;
		
		for(int orientation_index = 0; orientation_index < total_number_of_orientations; orientation_index++) {

			for(int wavelength_index = 0; wavelength_index < total_number_of_wavelengths; wavelength_index++) {

				for(int phase_index = 0; phase_index < total_number_of_phases; phase_index++) {

					int gabor_index = calculate_gabor_index(orientation_index,wavelength_index,phase_index);
					int start_index_for_current_gabor_image = image_starting_index + gabor_index * image_width * image_width;

					// printf("ORIENTATION: %d\n", orientation_index);
					// printf("WAVELENGTH: %d\n", wavelength_index);
					// printf("PHASE: %d\n\n", phase_index);
					// printf("GABOR_INDEX: %d\n", gabor_index);
					
					// Read input to network
					ostringstream dirStream;

					dirStream << inputDirectory << "Filtered/" << inputNames[image_index] << ".flt" << "/"
					<< inputNames[image_index] << '.' << filterWavelengths->at(wavelength_index) << '.' 
					<< filterOrientations->at(orientation_index) << '.' << filterPhases->at(phase_index) << ".gbo";
					
					string t = dirStream.str();
					
					// Open&Read gabor filter file
					fstreamWrapper gaborStream;
					
					try {
						
						gaborStream.open(t.c_str(), std::ios_base::in | std::ios_base::binary);

						for(int image_x = 0; image_x < image_width; image_x++)
							for(int image_y = 0; image_y < image_width; image_y++) {
								
								float rate;
								gaborStream >> rate;
								if(rate < 0) {
									cerr << "Negative firing loaded from filter!!!" << endl;
									exit(EXIT_FAILURE);
								}

								int element_index = start_index_for_current_gabor_image + image_x + image_y * image_width;
								
								input_rates[element_index] = rate;
							}
						
					} catch (fstream::failure e) {
						stringstream s;
						s << "Unable to open/read from " << t << " for gabor input: " << e.what();
						cerr << s.str();
						exit(EXIT_FAILURE);
					}
				}
			}
		}
	}
}

void ImagePoissonSpikingNeurons::copy_rates_to_device() {
	CudaSafeCall(hipMalloc((void **)&d_input_rates, sizeof(float)*total_number_of_rates));
	CudaSafeCall(hipMemcpy(d_input_rates, input_rates, sizeof(float)*total_number_of_gabor_types*inputNames.size(), hipMemcpyHostToDevice));
}


int ImagePoissonSpikingNeurons::calculate_gabor_index(int orientationIndex, int wavelengthIndex, int phaseIndex) {
	
	return orientationIndex * (total_number_of_wavelengths * total_number_of_phases) + wavelengthIndex * total_number_of_phases + phaseIndex;
}


