#include "hip/hip_runtime.h"
//	RecordingElectrodes Class C++
//	RecordingElectrodes.cu
//
//  Adapted from CUDACode
//	Authors: Nasir Ahmad and James Isbister
//	Date: 9/4/2016

#include "RecordingElectrodes.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"
#include <string>
#include <time.h>
using namespace std;

// RecordingElectrodes Constructor
RecordingElectrodes::RecordingElectrodes(SpikingNeurons * neurons_parameter, string full_directory_name_for_simulation_data_files_param, const char * prefix_string_param, int number_of_timesteps_per_device_spike_copy_check_param, int device_spike_store_size_multiple_of_total_neurons_param, float proportion_of_device_spike_store_full_before_copy_param) {

	neurons = neurons_parameter;
	full_directory_name_for_simulation_data_files = full_directory_name_for_simulation_data_files_param;
	prefix_string = prefix_string_param;

	number_of_timesteps_per_device_spike_copy_check = number_of_timesteps_per_device_spike_copy_check_param;
	device_spike_store_size_multiple_of_total_neurons = device_spike_store_size_multiple_of_total_neurons_param;
	proportion_of_device_spike_store_full_before_copy = proportion_of_device_spike_store_full_before_copy_param;
	size_of_device_spike_store = device_spike_store_size_multiple_of_total_neurons * neurons->total_number_of_neurons;

	d_per_neuron_spike_counts = NULL;
	
	d_neuron_ids_of_stored_spikes_on_device = NULL;
	h_neuron_ids_of_stored_spikes_on_host = NULL;
	
	d_time_in_seconds_of_stored_spikes_on_device= NULL;
	h_time_in_seconds_of_stored_spikes_on_host = NULL;

	d_total_number_of_spikes_stored_on_device = NULL;
	h_total_number_of_spikes_stored_on_host = 0;

	reset_neuron_ids = NULL;
	reset_neuron_times = NULL;

}


// RecordingElectrodes Destructor
RecordingElectrodes::~RecordingElectrodes() {

	CudaSafeCall(hipFree(d_total_number_of_spikes_stored_on_device));
	CudaSafeCall(hipFree(d_neuron_ids_of_stored_spikes_on_device));
	CudaSafeCall(hipFree(d_time_in_seconds_of_stored_spikes_on_device));

	free(h_neuron_ids_of_stored_spikes_on_host);
	free(h_time_in_seconds_of_stored_spikes_on_host);
	free(h_total_number_of_spikes_stored_on_device);

}


void RecordingElectrodes::allocate_pointers_for_spike_store() {

	h_total_number_of_spikes_stored_on_device = (int*)malloc(sizeof(int));
	
	CudaSafeCall(hipMalloc((void **)&d_neuron_ids_of_stored_spikes_on_device, sizeof(int)*size_of_device_spike_store));
	CudaSafeCall(hipMalloc((void **)&d_time_in_seconds_of_stored_spikes_on_device, sizeof(float)*size_of_device_spike_store));
	CudaSafeCall(hipMalloc((void **)&d_total_number_of_spikes_stored_on_device, sizeof(int)));

	reset_neuron_ids = (int *)malloc(sizeof(int)*size_of_device_spike_store);
	reset_neuron_times = (float *)malloc(sizeof(float)*size_of_device_spike_store);
	for (int i=0; i < size_of_device_spike_store; i++){
		reset_neuron_ids[i] = -1;
		reset_neuron_times[i] = -1.0f;
	}
}


void RecordingElectrodes::reset_pointers_for_spike_store() {

	h_total_number_of_spikes_stored_on_device[0] = 0;
	h_total_number_of_spikes_stored_on_host = 0;

	CudaSafeCall(hipMemset(d_neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*size_of_device_spike_store));
	CudaSafeCall(hipMemset(d_time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*size_of_device_spike_store));
	CudaSafeCall(hipMemset(d_total_number_of_spikes_stored_on_device, 0, sizeof(int)));

}



void RecordingElectrodes::allocate_pointers_for_spike_count() {
	//For counting spikes
	CudaSafeCall(hipMalloc((void **)&d_per_neuron_spike_counts, sizeof(int) * neurons->total_number_of_neurons));
	
}

void RecordingElectrodes::reset_pointers_for_spike_count() {

	CudaSafeCall(hipMemset(d_per_neuron_spike_counts, 0, sizeof(int) * neurons->total_number_of_neurons));

}




void RecordingElectrodes::collect_spikes_for_timestep(float current_time_in_seconds) {
	collect_spikes_for_timestep_kernel<<<neurons->number_of_neuron_blocks_per_grid, neurons->threads_per_block>>>(neurons->d_last_spike_time_of_each_neuron,
														d_total_number_of_spikes_stored_on_device,
														d_neuron_ids_of_stored_spikes_on_device,
														d_time_in_seconds_of_stored_spikes_on_device,
														current_time_in_seconds,
														neurons->total_number_of_neurons);

	CudaCheckError();
}

void RecordingElectrodes::copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(float current_time_in_seconds, int timestep_index, int number_of_timesteps_per_epoch) {

	if (((timestep_index % number_of_timesteps_per_device_spike_copy_check) == 0) || (timestep_index == (number_of_timesteps_per_epoch-1))){

		// Finally, we want to get the spikes back. Every few timesteps check the number of spikes:
		CudaSafeCall(hipMemcpy(&(h_total_number_of_spikes_stored_on_device[0]), &(d_total_number_of_spikes_stored_on_device[0]), (sizeof(int)), hipMemcpyDeviceToHost));

		// Ensure that we don't have too many
		if (h_total_number_of_spikes_stored_on_device[0] > size_of_device_spike_store){
			print_message_and_exit("Spike recorder has been overloaded! Reduce threshold.");
		}

		// Deal with them!
		if ((h_total_number_of_spikes_stored_on_device[0] >= (proportion_of_device_spike_store_full_before_copy * size_of_device_spike_store)) ||  (timestep_index == (number_of_timesteps_per_epoch - 1))){

			// Reallocate host spike arrays to accommodate for new device spikes.
			h_neuron_ids_of_stored_spikes_on_host = (int*)realloc(h_neuron_ids_of_stored_spikes_on_host, sizeof(int)*(h_total_number_of_spikes_stored_on_host + h_total_number_of_spikes_stored_on_device[0]));
			h_time_in_seconds_of_stored_spikes_on_host = (float*)realloc(h_time_in_seconds_of_stored_spikes_on_host, sizeof(float)*(h_total_number_of_spikes_stored_on_host + h_total_number_of_spikes_stored_on_device[0]));

			// Copy device spikes into correct host array location
			CudaSafeCall(hipMemcpy((void*)&h_neuron_ids_of_stored_spikes_on_host[h_total_number_of_spikes_stored_on_host], 
									d_neuron_ids_of_stored_spikes_on_device, 
									(sizeof(int)*h_total_number_of_spikes_stored_on_device[0]), 
									hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpy((void*)&h_time_in_seconds_of_stored_spikes_on_host[h_total_number_of_spikes_stored_on_host], 
									d_time_in_seconds_of_stored_spikes_on_device, 
									sizeof(float)*h_total_number_of_spikes_stored_on_device[0], 
									hipMemcpyDeviceToHost));

			h_total_number_of_spikes_stored_on_host += h_total_number_of_spikes_stored_on_device[0];


			// Reset device spikes
			CudaSafeCall(hipMemset(&(d_total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
			CudaSafeCall(hipMemcpy(d_neuron_ids_of_stored_spikes_on_device, reset_neuron_ids, sizeof(int)*size_of_device_spike_store, hipMemcpyHostToDevice));
			CudaSafeCall(hipMemcpy(d_time_in_seconds_of_stored_spikes_on_device, reset_neuron_times, sizeof(float)*size_of_device_spike_store, hipMemcpyHostToDevice));
			h_total_number_of_spikes_stored_on_device[0] = 0;
		}
	}
}


void RecordingElectrodes::add_spikes_to_per_neuron_spike_count(float current_time_in_seconds) {
	add_spikes_to_per_neuron_spike_count_kernel<<<neurons->number_of_neuron_blocks_per_grid, neurons->threads_per_block>>>(neurons->d_last_spike_time_of_each_neuron,
														d_per_neuron_spike_counts,
														current_time_in_seconds,
														neurons->total_number_of_neurons);
	CudaCheckError();
}




void RecordingElectrodes::write_spikes_to_file(int epoch_number, bool human_readable_storage, bool isTrained) {

	clock_t write_spikes_to_file_start = clock();

	// Get the names
	string phase = "";
	if (isTrained)
		phase = "Trained";
	else
		phase = "Untrained";
	
	string file_IDs = full_directory_name_for_simulation_data_files + prefix_string + "_SpikeIDs_" + phase + "_Epoch" + to_string(epoch_number);
	string file_Times = full_directory_name_for_simulation_data_files + prefix_string + "_SpikeTimes_" + phase + "_Epoch" + to_string(epoch_number);

//	// Append the clock to the file if flag
//	if (append_clock_to_filenames){ file = file + "t" + to_string(clock()) + "_"; }

	if (human_readable_storage){
		// Open the files
		ofstream spikeidfile, spiketimesfile;
		spikeidfile.open((file_IDs + ".txt"), ios::out | ios::binary);
		spiketimesfile.open((file_Times + ".txt"), ios::out | ios::binary);
		

		// Send the data
		for (int i = 0; i < h_total_number_of_spikes_stored_on_host; i++) {
			spikeidfile << to_string(h_neuron_ids_of_stored_spikes_on_host[i]) << endl;
			spiketimesfile << to_string(h_time_in_seconds_of_stored_spikes_on_host[i]) << endl;
		}

		// Close the files
		spikeidfile.close();
		spiketimesfile.close();
	} else {
		// Open the files
		ofstream spikeidfile, spiketimesfile;
		spikeidfile.open((file_IDs + ".bin"), ios::out | ios::binary);
		spiketimesfile.open((file_Times + ".bin"), ios::out | ios::binary);
		

		// Send the data
		spikeidfile.write((char *)h_neuron_ids_of_stored_spikes_on_host, h_total_number_of_spikes_stored_on_host*sizeof(int));
		spiketimesfile.write((char *)h_time_in_seconds_of_stored_spikes_on_host, h_total_number_of_spikes_stored_on_host*sizeof(float));

		// Close the files
		spikeidfile.close();
		spiketimesfile.close();
	}

	//delete_and_reset_recorded_spikes();

	clock_t write_spikes_to_file_end = clock();
	float write_spikes_to_file_total_time = float(write_spikes_to_file_end - write_spikes_to_file_start) / CLOCKS_PER_SEC;
	printf("Spikes written to file.\n Time taken: %f\n", write_spikes_to_file_total_time);
}



void RecordingElectrodes::delete_and_reset_recorded_spikes() {

	// Reset the spike store
	// Host values
	h_total_number_of_spikes_stored_on_host = 0;
	h_total_number_of_spikes_stored_on_device[0] = 0;
	// Free/Clear Device stuff
	// Reset the number on the device
	CudaSafeCall(hipMemset(&(d_total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
	CudaSafeCall(hipMemset(d_neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*neurons->total_number_of_neurons));
	CudaSafeCall(hipMemset(d_time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*neurons->total_number_of_neurons));

	// Free malloced host stuff
	free(h_neuron_ids_of_stored_spikes_on_host);
	free(h_time_in_seconds_of_stored_spikes_on_host);
	h_neuron_ids_of_stored_spikes_on_host = NULL;
	h_time_in_seconds_of_stored_spikes_on_host = NULL;
}



__global__ void add_spikes_to_per_neuron_spike_count_kernel(float* d_last_spike_time_of_each_neuron,
								int* d_per_neuron_spike_counts,
								float current_time_in_seconds,
								size_t total_number_of_neurons) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_neurons) {

		if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
			atomicAdd(&d_per_neuron_spike_counts[idx], 1);
		}

		// if (idx == 1000) printf("d_per_neuron_spike_counts[idx]: %d\n", d_per_neuron_spike_counts[idx]);
		idx += blockDim.x * gridDim.x;
	}
}

// Collect Spikes
__global__ void collect_spikes_for_timestep_kernel(float* d_last_spike_time_of_each_neuron,
								int* d_total_number_of_spikes_stored_on_device,
								int* d_neuron_ids_of_stored_spikes_on_device,
								float* d_time_in_seconds_of_stored_spikes_on_device,
								float current_time_in_seconds,
								size_t total_number_of_neurons){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_neurons) {

		// If a neuron has fired
		if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
			// Increase the number of spikes stored
			// NOTE: atomicAdd return value is actually original (atomic) value BEFORE incrementation!
			//		- So first value is actually 0 not 1!!!
			int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);
			__syncthreads();

			// In the location, add the id and the time
			d_neuron_ids_of_stored_spikes_on_device[i] = idx;
			d_time_in_seconds_of_stored_spikes_on_device[i] = current_time_in_seconds;
		}
		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}



void RecordingElectrodes::write_initial_synaptic_weights_to_file(SpikingSynapses *synapses, bool human_readable_storage) {
	ofstream initweightfile;
	if (human_readable_storage){
		initweightfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkWeights_Initial.txt", ios::out | ios::binary);
		for (int i=0; i < synapses->total_number_of_synapses; i++){
			initweightfile << to_string(synapses->synaptic_efficacies_or_weights[i]) << endl;

		}
		initweightfile.close();
	} else {
		initweightfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkWeights_Initial.bin", ios::out | ios::binary);
		initweightfile.write((char *)synapses->synaptic_efficacies_or_weights, synapses->total_number_of_synapses*sizeof(float));
		initweightfile.close();
	}
}


void RecordingElectrodes::write_network_state_to_file(SpikingSynapses *synapses, bool human_readable_storage) {

	clock_t save_network_state_start = clock();

	// Copy back the data that we might want:
	CudaSafeCall(hipMemcpy(synapses->synaptic_efficacies_or_weights, synapses->d_synaptic_efficacies_or_weights, sizeof(float)*synapses->total_number_of_synapses, hipMemcpyDeviceToHost));
	
	if (human_readable_storage){
		// Creating and Opening all the files
		ofstream synapsepre, synapsepost, weightfile, delayfile;
		weightfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkWeights.txt", ios::out | ios::binary);
		delayfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkDelays.txt", ios::out | ios::binary);
		synapsepre.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkPre.txt", ios::out | ios::binary);
		synapsepost.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkPost.txt", ios::out | ios::binary);
		
		// Writing the data
		for (int i=0; i < synapses->total_number_of_synapses; i++){
			weightfile << to_string(synapses->synaptic_efficacies_or_weights[i]) << endl;
			delayfile << to_string(synapses->delays[i]) << endl;
			synapsepre << to_string(synapses->presynaptic_neuron_indices[i]) << endl;
			synapsepost << to_string(synapses->postsynaptic_neuron_indices[i]) << endl;
		}

		// Close files
		weightfile.close();
		delayfile.close();
		synapsepre.close();
		synapsepost.close();
	} else {
		// Creating and Opening all the files
		ofstream synapsepre, synapsepost, weightfile, delayfile;
		weightfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkWeights.bin", ios::out | ios::binary);
		delayfile.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkDelays.bin", ios::out | ios::binary);
		synapsepre.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkPre.bin", ios::out | ios::binary);
		synapsepost.open(full_directory_name_for_simulation_data_files + prefix_string + "_NetworkPost.bin", ios::out | ios::binary);
		
		// Writing the data
		weightfile.write((char *)synapses->synaptic_efficacies_or_weights, synapses->total_number_of_synapses*sizeof(float));
		delayfile.write((char *)synapses->delays, synapses->total_number_of_synapses*sizeof(int));
		synapsepre.write((char *)synapses->presynaptic_neuron_indices, synapses->total_number_of_synapses*sizeof(int));
		synapsepost.write((char *)synapses->postsynaptic_neuron_indices, synapses->total_number_of_synapses*sizeof(int));

		// Close files
		weightfile.close();
		delayfile.close();
		synapsepre.close();
		synapsepost.close();
	}

	#ifndef QUIETSTART
	clock_t save_network_state_end = clock();
	float save_network_state_total_time = float(save_network_state_end - save_network_state_start) / CLOCKS_PER_SEC;
	printf("Network state saved to file.\n Time taken: %f\n", save_network_state_total_time);
	print_line_of_dashes_with_blank_lines_either_side();
	#endif

}


