#include "hip/hip_runtime.h"
#include "CollectNeuronSpikesRecordingElectrodes.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"
#include <string>
#include <time.h>
using namespace std;

// CollectNeuronSpikesRecordingElectrodes Constructor
CollectNeuronSpikesRecordingElectrodes::CollectNeuronSpikesRecordingElectrodes(SpikingNeurons * neurons_parameter, SpikingSynapses * spiking_synapses, string full_directory_name_for_simulation_data_files_param, const char * prefix_string_param) {}

	// Variables
	number_of_timesteps_per_device_spike_copy_check = 0;
	device_spike_store_size_multiple_of_total_neurons = 0;
	size_of_device_spike_store = 0
	h_total_number_of_spikes_stored_on_host = 0;
	proportion_of_device_spike_store_full_before_copy = proportion_of_device_spike_store_full_before_copy_param;

	// Host Pointers
	h_neuron_ids_of_stored_spikes_on_host = NULL;
	h_total_number_of_spikes_stored_on_device = NULL;
	h_time_in_seconds_of_stored_spikes_on_host = NULL;

	// Device Pointers
	d_neuron_ids_of_stored_spikes_on_device = NULL;
	d_total_number_of_spikes_stored_on_device = NULL;
	d_time_in_seconds_of_stored_spikes_on_device= NULL;

	// Private Host Pointeres
	reset_neuron_ids = NULL;
	reset_neuron_times = NULL;

}


// CollectNeuronSpikesRecordingElectrodes Destructor
CollectNeuronSpikesRecordingElectrodes::~CollectNeuronSpikesRecordingElectrodes() {

	free(h_neuron_ids_of_stored_spikes_on_host);
	free(h_time_in_seconds_of_stored_spikes_on_host);
	free(h_total_number_of_spikes_stored_on_device);

	CudaSafeCall(hipFree(d_neuron_ids_of_stored_spikes_on_device));
	CudaSafeCall(hipFree(d_total_number_of_spikes_stored_on_device));
	CudaSafeCall(hipFree(d_time_in_seconds_of_stored_spikes_on_device));

	free(reset_neuron_ids);
	free(reset_neuron_times);

}


void CollectNeuronSpikesRecordingElectrodes::initialise_collect_neuron_spikes_recording_electrodes(int number_of_timesteps_per_device_spike_copy_check_param, int device_spike_store_size_multiple_of_total_neurons_param, float proportion_of_device_spike_store_full_before_copy_param) {

	number_of_timesteps_per_device_spike_copy_check = number_of_timesteps_per_device_spike_copy_check_param;
	device_spike_store_size_multiple_of_total_neurons = device_spike_store_size_multiple_of_total_neurons_param;
	size_of_device_spike_store = device_spike_store_size_multiple_of_total_neurons * neurons->total_number_of_neurons;
	proportion_of_device_spike_store_full_before_copy = proportion_of_device_spike_store_full_before_copy_param;

	allocate_pointers_for_spike_store();
	reset_pointers_for_spike_store();

}



void CollectNeuronSpikesRecordingElectrodes::allocate_pointers_for_spike_store() {

	h_total_number_of_spikes_stored_on_device = (int*)malloc(sizeof(int));
	
	CudaSafeCall(hipMalloc((void **)&d_neuron_ids_of_stored_spikes_on_device, sizeof(int)*size_of_device_spike_store));
	CudaSafeCall(hipMalloc((void **)&d_time_in_seconds_of_stored_spikes_on_device, sizeof(float)*size_of_device_spike_store));
	CudaSafeCall(hipMalloc((void **)&d_total_number_of_spikes_stored_on_device, sizeof(int)));

	reset_neuron_ids = (int *)malloc(sizeof(int)*size_of_device_spike_store);
	reset_neuron_times = (float *)malloc(sizeof(float)*size_of_device_spike_store);
	for (int i=0; i < size_of_device_spike_store; i++){
		reset_neuron_ids[i] = -1;
		reset_neuron_times[i] = -1.0f;
	}
}


void CollectNeuronSpikesRecordingElectrodes::reset_pointers_for_spike_store() {

	h_total_number_of_spikes_stored_on_device[0] = 0;
	h_total_number_of_spikes_stored_on_host = 0;

	CudaSafeCall(hipMemset(d_neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*size_of_device_spike_store));
	CudaSafeCall(hipMemset(d_time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*size_of_device_spike_store));
	CudaSafeCall(hipMemset(d_total_number_of_spikes_stored_on_device, 0, sizeof(int)));

}



void CollectNeuronSpikesRecordingElectrodes::delete_and_reset_recorded_spikes() {

	// Reset the spike store
	// Host values
	h_total_number_of_spikes_stored_on_host = 0;
	h_total_number_of_spikes_stored_on_device[0] = 0;
	// Free/Clear Device stuff
	// Reset the number on the device
	CudaSafeCall(hipMemset(&(d_total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
	CudaSafeCall(hipMemset(d_neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*neurons->total_number_of_neurons));
	CudaSafeCall(hipMemset(d_time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*neurons->total_number_of_neurons));

	// Free malloced host stuff
	free(h_neuron_ids_of_stored_spikes_on_host);
	free(h_time_in_seconds_of_stored_spikes_on_host);
	h_neuron_ids_of_stored_spikes_on_host = NULL;
	h_time_in_seconds_of_stored_spikes_on_host = NULL;
}




void CollectNeuronSpikesRecordingElectrodes::copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(float current_time_in_seconds, int timestep_index, int number_of_timesteps_per_epoch) {

	if (((timestep_index % number_of_timesteps_per_device_spike_copy_check) == 0) || (timestep_index == (number_of_timesteps_per_epoch-1))){

		// Finally, we want to get the spikes back. Every few timesteps check the number of spikes:
		CudaSafeCall(hipMemcpy(&(h_total_number_of_spikes_stored_on_device[0]), &(d_total_number_of_spikes_stored_on_device[0]), (sizeof(int)), hipMemcpyDeviceToHost));

		// Ensure that we don't have too many
		if (h_total_number_of_spikes_stored_on_device[0] > size_of_device_spike_store){
			print_message_and_exit("Spike recorder has been overloaded! Reduce threshold.");
		}

		// Deal with them!
		if ((h_total_number_of_spikes_stored_on_device[0] >= (proportion_of_device_spike_store_full_before_copy * size_of_device_spike_store)) ||  (timestep_index == (number_of_timesteps_per_epoch - 1))){

			// Reallocate host spike arrays to accommodate for new device spikes.
			h_neuron_ids_of_stored_spikes_on_host = (int*)realloc(h_neuron_ids_of_stored_spikes_on_host, sizeof(int)*(h_total_number_of_spikes_stored_on_host + h_total_number_of_spikes_stored_on_device[0]));
			h_time_in_seconds_of_stored_spikes_on_host = (float*)realloc(h_time_in_seconds_of_stored_spikes_on_host, sizeof(float)*(h_total_number_of_spikes_stored_on_host + h_total_number_of_spikes_stored_on_device[0]));

			// Copy device spikes into correct host array location
			CudaSafeCall(hipMemcpy((void*)&h_neuron_ids_of_stored_spikes_on_host[h_total_number_of_spikes_stored_on_host], 
									d_neuron_ids_of_stored_spikes_on_device, 
									(sizeof(int)*h_total_number_of_spikes_stored_on_device[0]), 
									hipMemcpyDeviceToHost));
			CudaSafeCall(hipMemcpy((void*)&h_time_in_seconds_of_stored_spikes_on_host[h_total_number_of_spikes_stored_on_host], 
									d_time_in_seconds_of_stored_spikes_on_device, 
									sizeof(float)*h_total_number_of_spikes_stored_on_device[0], 
									hipMemcpyDeviceToHost));

			h_total_number_of_spikes_stored_on_host += h_total_number_of_spikes_stored_on_device[0];


			// Reset device spikes
			CudaSafeCall(hipMemset(&(d_total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
			CudaSafeCall(hipMemcpy(d_neuron_ids_of_stored_spikes_on_device, reset_neuron_ids, sizeof(int)*size_of_device_spike_store, hipMemcpyHostToDevice));
			CudaSafeCall(hipMemcpy(d_time_in_seconds_of_stored_spikes_on_device, reset_neuron_times, sizeof(float)*size_of_device_spike_store, hipMemcpyHostToDevice));
			h_total_number_of_spikes_stored_on_device[0] = 0;
		}
	}
}



void CollectNeuronSpikesRecordingElectrodes::write_spikes_to_file(int epoch_number, bool human_readable_storage, bool isTrained) {

	clock_t write_spikes_to_file_start = clock();

	// Get the names
	string phase = "";
	if (isTrained)
		phase = "Trained";
	else
		phase = "Untrained";
	
	string file_IDs = full_directory_name_for_simulation_data_files + prefix_string + "_SpikeIDs_" + phase + "_Epoch" + to_string(epoch_number);
	string file_Times = full_directory_name_for_simulation_data_files + prefix_string + "_SpikeTimes_" + phase + "_Epoch" + to_string(epoch_number);

//	// Append the clock to the file if flag
//	if (append_clock_to_filenames){ file = file + "t" + to_string(clock()) + "_"; }

	if (human_readable_storage){
		// Open the files
		ofstream spikeidfile, spiketimesfile;
		spikeidfile.open((file_IDs + ".txt"), ios::out | ios::binary);
		spiketimesfile.open((file_Times + ".txt"), ios::out | ios::binary);
		

		// Send the data
		for (int i = 0; i < h_total_number_of_spikes_stored_on_host; i++) {
			spikeidfile << to_string(h_neuron_ids_of_stored_spikes_on_host[i]) << endl;
			spiketimesfile << to_string(h_time_in_seconds_of_stored_spikes_on_host[i]) << endl;
		}

		// Close the files
		spikeidfile.close();
		spiketimesfile.close();
	} else {
		// Open the files
		ofstream spikeidfile, spiketimesfile;
		spikeidfile.open((file_IDs + ".bin"), ios::out | ios::binary);
		spiketimesfile.open((file_Times + ".bin"), ios::out | ios::binary);
		

		// Send the data
		spikeidfile.write((char *)h_neuron_ids_of_stored_spikes_on_host, h_total_number_of_spikes_stored_on_host*sizeof(int));
		spiketimesfile.write((char *)h_time_in_seconds_of_stored_spikes_on_host, h_total_number_of_spikes_stored_on_host*sizeof(float));

		// Close the files
		spikeidfile.close();
		spiketimesfile.close();
	}

	//delete_and_reset_recorded_spikes();

	clock_t write_spikes_to_file_end = clock();
	float write_spikes_to_file_total_time = float(write_spikes_to_file_end - write_spikes_to_file_start) / CLOCKS_PER_SEC;
	printf("Spikes written to file.\n Time taken: %f\n", write_spikes_to_file_total_time);
}





// Collect Spikes
__global__ void collect_spikes_for_timestep_kernel(float* d_last_spike_time_of_each_neuron,
								int* d_total_number_of_spikes_stored_on_device,
								int* d_neuron_ids_of_stored_spikes_on_device,
								float* d_time_in_seconds_of_stored_spikes_on_device,
								float current_time_in_seconds,
								size_t total_number_of_neurons){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_neurons) {

		// If a neuron has fired
		if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
			// Increase the number of spikes stored
			// NOTE: atomicAdd return value is actually original (atomic) value BEFORE incrementation!
			//		- So first value is actually 0 not 1!!!
			int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);
			__syncthreads();

			// In the location, add the id and the time
			d_neuron_ids_of_stored_spikes_on_device[i] = idx;
			d_time_in_seconds_of_stored_spikes_on_device[i] = current_time_in_seconds;
		}
		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}