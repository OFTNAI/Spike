#include "hip/hip_runtime.h"
#include "FourLayerVisionSpikingModel.h"


// FourLayerVisionSpikingModel Constructor
FourLayerVisionSpikingModel::FourLayerVisionSpikingModel (float timestep) {


	E2E_L_ON = true;
	E2E_FB_ON = false;
	E2E_L_STDP_ON = false;

	// Network Parameters
	// const int number_of_layers = 4;
	max_number_of_connections_per_pair = 5;
	dim_excit_layer = 64;
	dim_inhib_layer = 32;

	fanInCount_G2E_FF = 30;
	fanInCount_E2E_FF = 100;
	fanInCount_E2I_L = 30;
	fanInCount_I2E_L = 30;
	fanInCount_E2E_L = 10;
	fanInCount_E2E_FB = 10;

	gaussian_synapses_standard_deviation_G2E_FF = 1.0;
	gaussian_synapses_standard_deviation_E2E_FF[number_of_layers-1] = {8.0, 12.0, 16.0};
	gaussian_synapses_standard_deviation_E2I_L = 1.0;
	gaussian_synapses_standard_deviation_I2E_L = 8.0;
	gaussian_synapses_standard_deviation_E2E_L = 4.0;
	gaussian_synapses_standard_deviation_E2E_FB = 16.0;

	biological_conductance_scaling_constant_lambda_G2E_FF = 0.00002;
	biological_conductance_scaling_constant_lambda_E2E_FF = 0.0001;
	biological_conductance_scaling_constant_lambda_E2I_L = 0.002;
	biological_conductance_scaling_constant_lambda_I2E_L = 0.004;
	biological_conductance_scaling_constant_lambda_E2E_L = 0.0001;
	biological_conductance_scaling_constant_lambda_E2E_FB = 0.00001;

	decay_term_tau_g_G2E_FF = 0.15;
	decay_term_tau_g_E2E_FF = 0.15;
	decay_term_tau_g_E2I_L = 0.002;
	decay_term_tau_g_I2E_L = 0.025; //In Ben's model, 0.005 v 0.025 and latter produced better result
	decay_term_tau_g_E2E_L = 0.15;
	decay_term_tau_g_E2E_FB = 0.15;

	// Neuronal Parameters
	max_FR_of_input_Gabor = 100.0f;
	absolute_refractory_period = 0.002;

	//Synaptic Parameters
	weight_range_bottom = 0.0;
	weight_range_top = 1.0;
	learning_rate_rho = 0.1/timestep;//100.0;// 0.1;
	decay_term_tau_C = 0.3;//(In Ben's model, tau_C/tau_D = 3/5 v 15/25 v 75/125, and the first one produces the best result)
	decay_term_tau_D = 0.3;

	E2E_FF_minDelay = 5.0*timestep;
	E2E_FF_maxDelay = 0.01;//3.0f*pow(10, -3);
	E2I_L_minDelay = 5.0*timestep;
	E2I_L_maxDelay = 0.01;//3.0f*pow(10, -3);
	I2E_L_minDelay = 5.0*timestep;
	I2E_L_maxDelay = 0.01;//3.0f*pow(10, -3);
	E2E_FB_minDelay = 5.0*timestep;
	E2E_FB_maxDelay = 0.01;
	E2E_L_minDelay = 5.0*timestep;
	E2E_L_maxDelay = 0.01;


}


// FourLayerVisionSpikingModel Destructor
FourLayerVisionSpikingModel::~FourLayerVisionSpikingModel () {


}



FourLayerVisionSpikingModel::step_1() {

	lif_spiking_neurons = new LIFSpikingNeurons();
	input_neurons = new ImagePoissonInputSpikingNeurons();
	conductance_spiking_synapses = new ConductanceSpikingSynapses();
	evans_stdp = new EvansSTDP();

	/////////// STDP SETUP ///////////
	evans_stdp_parameters_struct * STDP_PARAMS = new evans_stdp_parameters_struct();
	STDP_PARAMS->decay_term_tau_C = decay_term_tau_C;
	STDP_PARAMS->decay_term_tau_D = decay_term_tau_D;
	STDP_PARAMS->learning_rate_rho = learning_rate_rho;
	evans_stdp->Set_STDP_Parameters((SpikingSynapses *) conductance_spiking_synapses, (SpikingNeurons *) lif_spiking_neurons, (SpikingNeurons *) input_neurons, (stdp_parameters_struct *) STDP_PARAMS);

}


FourLayerVisionSpikingModel::step_2 () {


	/// PROBABLY NEED TO INITIALISE RANDOM STATES BEFORE THIS

	/////////// ADD INPUT NEURONS ///////////
	TimerWithMessages * adding_input_neurons_timer = new TimerWithMessages("Adding Input Neurons...\n");

	if (is_optimisation)
		input_neurons->set_up_rates("FileList.txt", "FilterParameters.txt", "../../MatlabGaborFilter/Inputs/", 100.0f);
	else
		input_neurons->set_up_rates("FileList.txt", "FilterParameters.txt", "MatlabGaborFilter/Inputs/", 100.0f);

	image_poisson_input_spiking_neuron_parameters_struct * image_poisson_input_spiking_group_params = new image_poisson_input_spiking_neuron_parameters_struct();
	image_poisson_input_spiking_group_params->rate = 30.0f; // ??????
	input_neurons->AddGroupForEachGaborType(image_poisson_input_spiking_group_params);

	adding_input_neurons_timer->stop_timer_and_log_time_and_message("Input Neurons Added.", true);



	/////////// ADD NEURONS ///////////
	TimerWithMessages * adding_neurons_timer = new TimerWithMessages("Adding Neurons...\n");

	lif_spiking_neuron_parameters_struct * EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS = new lif_spiking_neuron_parameters_struct();
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->group_shape[0] = dim_excit_layer;
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->group_shape[1] = dim_excit_layer;
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->resting_potential_v0 = -0.074f;
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->threshold_for_action_potential_spike = -0.053f;
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->somatic_capcitance_Cm = 500.0*pow(10, -12);
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->somatic_leakage_conductance_g0 = 25.0*pow(10, -9);
	EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS->absolute_refractory_period = absolute_refractory_period;


	lif_spiking_neuron_parameters_struct * INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS = new lif_spiking_neuron_parameters_struct();
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->group_shape[0] = dim_inhib_layer;
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->group_shape[1] = dim_inhib_layer;
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->resting_potential_v0 = -0.082f;
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->threshold_for_action_potential_spike = -0.053f;
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->somatic_capcitance_Cm = 214.0*pow(10, -12);
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->somatic_leakage_conductance_g0 = 18.0*pow(10, -9);
	INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS->absolute_refractory_period = absolute_refractory_period;

	
	for (int l=0;l<number_of_layers;l++){
		EXCITATORY_NEURONS.push_back(simulator.AddNeuronGroup(EXCITATORY_LIF_SPIKING_NEURON_GROUP_PARAMS));
		INHIBITORY_NEURONS.push_back(simulator.AddNeuronGroup(INHIBITORY_LIF_SPIKING_NEURON_GROUP_PARAMS));
		cout<<"Neuron Group "<<EXCITATORY_NEURONS[l]<<": Excitatory layer "<<l<<endl;
		cout<<"Neuron Group "<<INHIBITORY_NEURONS[l]<<": Inhibitory layer "<<l<<endl;
	}


	adding_neurons_timer->stop_timer_and_log_time_and_message("Neurons Added.", true);


	/////////// ADD SYNAPSES ///////////
	TimerWithMessages * adding_synapses_timer = new TimerWithMessages("Adding Synapses...\n");


	conductance_spiking_synapse_parameters_struct * G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = timestep;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = timestep;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = max_number_of_connections_per_pair;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_G2E_FF;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_G2E_FF;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = false;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_G2E_FF;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = 0.0;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_G2E_FF;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
	G2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;


	conductance_spiking_synapse_parameters_struct * E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = E2E_FF_minDelay;//5.0*timestep;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = E2E_FF_maxDelay;//3.0f*pow(10, -3);
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = max_number_of_connections_per_pair;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_E2E_FF;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_E2E_FF;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = true;
//	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_E2E_FF[0];
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = 0.0;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_E2E_FF;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
	E2E_FF_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;


	conductance_spiking_synapse_parameters_struct * E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	if(E2E_FB_ON){
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = E2E_FB_minDelay;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = E2E_FB_maxDelay;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = max_number_of_connections_per_pair;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_E2E_FB;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_E2E_FB;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = true;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_E2E_FB;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = 0.0;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_E2E_FB;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
		E2E_FB_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;
	}


	conductance_spiking_synapse_parameters_struct * E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = E2I_L_minDelay; //5.0*timestep;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = E2I_L_maxDelay; //3.0f*pow(10, -3);
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = 1;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_E2I_L;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_E2I_L;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = false;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_E2I_L;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = 0.0;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_E2I_L;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
	E2I_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;

	conductance_spiking_synapse_parameters_struct * I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = I2E_L_minDelay;//5.0*timestep;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = I2E_L_maxDelay;//3.0f*pow(10, -3);
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = 1;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_I2E_L;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_I2E_L;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = false;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_I2E_L;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = -70.0*pow(10, -3);
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_I2E_L;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
	I2E_L_INHIBITORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;

	conductance_spiking_synapse_parameters_struct * E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS = new conductance_spiking_synapse_parameters_struct();
	if(E2E_L_ON){
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[0] = E2E_L_minDelay;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->delay_range[1] = E2E_L_maxDelay;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->max_number_of_connections_per_pair = 1;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_per_postsynaptic_neuron = fanInCount_E2E_L;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->biological_conductance_scaling_constant_lambda = biological_conductance_scaling_constant_lambda_E2E_L;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->connectivity_type = CONNECTIVITY_TYPE_GAUSSIAN_SAMPLE;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->stdp_on = E2E_L_STDP_ON;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->gaussian_synapses_standard_deviation = gaussian_synapses_standard_deviation_E2E_L;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->reversal_potential_Vhat = 0.0;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->decay_term_tau_g = decay_term_tau_g_E2E_L;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_bottom = weight_range_bottom;
		E2E_L_EXCITATORY_CONDUCTANCE_SPIKING_SYNAPSE_PARAMETERS->weight_range_top = weight_range_top;
	}

}