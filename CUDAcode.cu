#include "hip/hip_runtime.h"
//	CUDA code for SPIKE simulator
//
//
//	Author: Nasir Ahmad
//	Date: 9/12/2015

//  Adapted by Nasir Ahmad and James Isbister
//	Date: 23/3/2016

// For files/manipulations
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm> // For random shuffle
using namespace std;


#include "CUDAcode.h"
#include <time.h>
#include "CUDAErrorCheckHelpers.h"
#include "RecordingElectrodes.h"

#include "GeneratorSpikingNeurons.h"

// Silences the printfs
// #define QUIETSTART

__global__ void init(unsigned int seed, hiprandState_t* states, size_t numNeurons);



//			lastactive = vector- indicating last time synapse emitted current
void GPUDeviceComputation (
					Neurons * neurons,
					Connections * connections,
					PoissonSpikingNeurons * input_neurons,

					float total_time_per_epoch,
					int number_of_epochs,
					float timestep,
					bool save_spikes,

					int numStimuli,
					int* numEntries,
					int** genids,
					float** gentimes,
					bool randomPresentation
					){

	GeneratorSpikingNeurons * temp_test_generator = new GeneratorSpikingNeurons();

	
	RecordingElectrodes * recording_electrodes = new RecordingElectrodes(neurons);
	RecordingElectrodes * input_recording_electrodes = new RecordingElectrodes(input_neurons);

	neurons->initialise_device_pointers();
	connections->initialise_device_pointers();
	input_neurons->initialise_device_pointersNew();

	recording_electrodes->initialise_device_pointers();
	recording_electrodes->initialise_host_pointers();

	input_recording_electrodes->initialise_device_pointers();
	input_recording_electrodes->initialise_host_pointers();


	// THREADS&BLOCKS
	// The number of threads per block I shall keep held at 128
	int threads = 128;
	connections->set_threads_per_block_and_blocks_per_grid(threads);
	neurons->set_threads_per_block_and_blocks_per_grid(threads);
	input_neurons->set_threads_per_block_and_blocks_per_grid(threads);


	input_neurons->generate_random_states_wrapper();

	hiprandState_t* states;
	hipMalloc((void**) &states, neurons->total_number_of_neurons*sizeof(hiprandState_t));
	// Initialise the random states
	init<<<neurons->threads_per_block, neurons->number_of_neuron_blocks_per_grid>>>(42, states, neurons->total_number_of_neurons);
	CudaCheckError();
	// Keep space for the random numbers
	float* gpu_randfloats;
	CudaSafeCall(hipMalloc((void**) &gpu_randfloats, neurons->total_number_of_neurons*sizeof(float)));

	// Variables necessary
	clock_t begin = clock();

	// Poisson number
	int numPoisson = 0;
	for (int i = 0; i < neurons->total_number_of_neurons; i++){
		if (neurons->neuron_variables[i].rate != 0.0f){
			++numPoisson;
		}
	}

	// STIMULUS ORDER
	int presentorder[numStimuli];
	for (int i = 0; i < numStimuli; i++){
		presentorder[i] = i;
	}

	// SEEDING
	srand(42);

	recording_electrodes->write_initial_synaptic_weights_to_file(connections);

	// Running through all of the Epochs
	for (int epoch_number = 0; epoch_number < number_of_epochs; epoch_number++) {
		// If we want a random presentation, create the set of numbers:
		if (randomPresentation) {
			random_shuffle(&presentorder[0], &presentorder[numStimuli]);
		}
		// Running through every Stimulus
		for (int j = 0; j < numStimuli; j++){
			// Get the presentation position:
			int present = presentorder[j];
			// Get the number of entries for this specific stimulus
			size_t numEnts = numEntries[present];
			if (numEnts > 0){

				temp_test_generator->initialise_device_pointers_for_ents(numEnts, present);
				temp_test_generator->set_threads_per_block_and_blocks_per_grid(threads);
				
			}
			// Reset the variables necessary
			neurons->reset_neuron_variables_and_spikes();
			connections->reset_connection_spikes();

			// Running the Simulation!
			// Variables as Necessary
			int number_of_timesteps_per_epoch = total_time_per_epoch / timestep;
			float current_time_in_seconds = 0.0f;
			// GO!
			for (int timestep_index = 0; timestep_index < number_of_timesteps_per_epoch; timestep_index++){
				// SIMULATION
				// Current simulation timestep
				current_time_in_seconds = float(timestep_index)*float(timestep);
				// Start by resetting all the things
				neurons->reset_device_current_injections();
				// If there are poisson populations
				if (numPoisson > 0) {
					// First create the set of random numbers of poisson neurons
					randoms<<<neurons->threads_per_block, neurons->number_of_neuron_blocks_per_grid>>>(states, gpu_randfloats, neurons->total_number_of_neurons);
					CudaCheckError();

					// Update Poisson neuron states
					neurons->poisupdate_wrapper(gpu_randfloats, timestep);
					input_neurons->update_poisson_state_wrapper(timestep);
					
				}
				// If there are any spike generators
				if (numEnts > 0) {
					// Update those neurons corresponding to the Spike Generators
					temp_test_generator->generupdate2_wrapper(current_time_in_seconds, timestep);
				} 
				
				connections->calculate_postsynaptic_current_injection_for_connection_wrapper(neurons->d_current_injections, current_time_in_seconds);

				// Carry out LTD on appropriate synapses
				connections->ltdweights_wrapper(neurons->d_lastspiketime, current_time_in_seconds);

				// Update States of neurons
				neurons->stateupdate_wrapper(timestep);

				// Check which neurons are spiking and deal with them
				neurons->spikingneurons_wrapper(current_time_in_seconds);
				input_neurons->spikingneurons_wrapper(current_time_in_seconds);
								
				// Check which synapses to send spikes down and do it
				connections->synapsespikes_wrapper(neurons->d_lastspiketime, current_time_in_seconds);

				// // Carry out the last step, LTP!
				connections->synapseLTP_wrapper(neurons->d_lastspiketime, current_time_in_seconds);
				

				// Only save the spikes if necessary
				if (save_spikes){
					recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch, true);
					input_recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch, false);

				}
			}
			if (numEnts > 0){
				// CudaSafeCall(hipFree(d_genids));
				// CudaSafeCall(hipFree(d_gentimes));
			}
		}
		#ifndef QUIETSTART
		clock_t mid = clock();
		if (save_spikes) {
			printf("Epoch %d, Complete.\n Running Time: %f\n Number of Spikes: %d\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC), recording_electrodes->h_total_number_of_spikes);
			printf("Number of Input Spikes: %d\n\n", input_recording_electrodes->h_total_number_of_spikes);
		
		} else {
			printf("Epoch %d, Complete.\n Running Time: %f\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC));
		}
		#endif
		// Output Spikes list after each epoch:
		// Only save the spikes if necessary
		if (save_spikes){
			recording_electrodes->write_spikes_to_file(neurons, epoch_number);
		}
	}
	
	// SIMULATION COMPLETE!
	#ifndef QUIETSTART
	// Finish the simulation and check time
	clock_t end = clock();
	float timed = float(end-begin) / CLOCKS_PER_SEC;
	printf("Simulation Complete! Time Elapsed: %f\n\n", timed);
	#endif

	recording_electrodes->save_network_state(connections);


	delete neurons;
	delete connections;
	delete recording_electrodes;

	CudaSafeCall(hipFree(states));
	CudaSafeCall(hipFree(gpu_randfloats));
	// CudaSafeCall(hipFree(currentinjection));
	// Free Memory on CPU
	free(recording_electrodes->h_spikestoretimes);
	free(recording_electrodes->h_spikestoreID);

}


// Random Number Getter
__global__ void randoms(hiprandState_t* states, float* numbers, size_t total_number_of_neurons) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < total_number_of_neurons) {
		/* hiprand works like rand - except that it takes a state as a parameter */
		numbers[idx] = hiprand_uniform(&states[idx]);
	}
}

__global__ void init(unsigned int seed, hiprandState_t* states, size_t total_number_of_neurons) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < total_number_of_neurons) {
		hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
					idx, /* the sequence number should be different for each core (unless you want all
							cores to get the same sequence of numbers for some reason - use thread id! */
 					0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
					&states[idx]);
	}
}


