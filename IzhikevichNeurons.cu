#include "hip/hip_runtime.h"
#include "IzhikevichNeurons.h"
#include <stdlib.h>
#include <stdio.h>
#include "CUDAErrorCheckHelpers.h"


// IzhikevichNeurons Constructor
IzhikevichNeurons::IzhikevichNeurons() {

	izhikevich_neuron_variables = NULL;

}


// IzhikevichNeurons Destructor
IzhikevichNeurons::~IzhikevichNeurons() {

}


int IzhikevichNeurons::AddGroupNew(neuron_struct *params, int group_shape[2]){

	int new_group_id = Neurons::AddGroupNew(params, group_shape);

	izhikevich_neuron_variables = (izhikevich_neuron_struct*)realloc(izhikevich_neuron_variables, (total_number_of_neurons*sizeof(izhikevich_neuron_struct)));
	for (int i = (total_number_of_neurons - number_of_neurons_in_new_group); i < total_number_of_neurons; i++){
		izhikevich_neuron_variables[i] = *((izhikevich_neuron_struct*)params);
	}

	return new_group_id;
}


void IzhikevichNeurons::initialise_device_pointersNew() {
 	
 	Neurons::initialise_device_pointersNew();

	CudaSafeCall(hipMalloc((void **)&d_izhikevich_neuron_variables, sizeof(struct izhikevich_neuron_struct)*total_number_of_neurons));
	
	reset_neuron_variables_and_spikesNew();
}

void IzhikevichNeurons::reset_neuron_variables_and_spikesNew() {

	Neurons::reset_neuron_variables_and_spikesNew();	

	CudaSafeCall(hipMemcpy(d_izhikevich_neuron_variables, izhikevich_neuron_variables, sizeof(struct izhikevich_neuron_struct)*total_number_of_neurons, hipMemcpyHostToDevice));

}



// // CUDA __global__ function declarations
// // NOTE: these are NOT MEMBER FUNCTIONS
// // They are called by their corresponding wrapper member function

// __global__ void poisupdate(float* d_randoms, 
// 							struct neuron_struct* d_neuronpop_variables,
// 							float timestep,
// 							size_t numNeurons);

// __global__ void genupdate(struct neuron_struct* neuronpop_variables,
// 							int* genids,
// 							float* gentimes,
// 							float currtime,
// 							float timestep,
// 							size_t numEntries);

// __global__ void spikingneurons(struct neuron_struct* neuronpop_variables,
// 							float* d_lastspiketime,
// 							float currtime,
// 							size_t numNeurons);

// __global__ void stateupdate(struct neuron_struct* neuronpop_variables,
// 							float* currentinj,
// 							float timestep,
// 							size_t numNeurons);



// // Wrapper member function definitions
// // See NOTE above
// void IzhikevichNeurons::poisupdate_wrapper(float* d_randoms, float timestep) {

// 	poisupdate<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(d_randoms,
// 														d_neuron_variables,
// 														timestep,
// 														total_number_of_neurons);
// 	CudaCheckError();
// }


// void IzhikevichNeurons::genupdate_wrapper(int* genids,
// 							float* gentimes,
// 							float currtime,
// 							float timestep,
// 							size_t numEntries,
// 							int genblocknum, 
// 							dim3 threadsPerBlock) {

// 	genupdate<<<genblocknum, threadsPerBlock>>> (d_neuron_variables,
// 												genids,
// 												gentimes,
// 												currtime,
// 												timestep,
// 												numEntries);

// 	CudaCheckError();
// }


// void IzhikevichNeurons::spikingneurons_wrapper(float currtime) {

// 	spikingneurons<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(d_neuron_variables,
// 																		d_lastspiketime,
// 																		currtime,
// 																		total_number_of_neurons);

// 	CudaCheckError();
// }


// void IzhikevichNeurons::stateupdate_wrapper(float* current_injection,
// 							float timestep) {

// 	stateupdate<<<number_of_neuron_blocks_per_grid, threads_per_block>>>(d_neuron_variables,
// 																	current_injection,
// 																	timestep,
// 																	total_number_of_neurons);

// 	CudaCheckError();
// }




// // CUDA __global__ function definitions
// // These are called by the Neurons class member functions
// // May have to vary names if 'including' more than one subclass

// // Poisson Updating Kernal
// __global__ void poisupdate(float* d_randoms, 
// 							struct neuron_struct* d_neuronpop_variables,
// 							float timestep,
// 							size_t numNeurons){
// 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (idx < numNeurons){
// 		// if the randomnumber is LT the rate
// 		if (d_randoms[idx] < (d_neuronpop_variables[idx].rate*timestep)){
// 			d_neuronpop_variables[idx].state_u = 0.0f;
// 			d_neuronpop_variables[idx].state_v = 35.0f;
// 		} else if (d_neuronpop_variables[idx].rate != 0.0f) {
// 			d_neuronpop_variables[idx].state_u = 0.0f;
// 			d_neuronpop_variables[idx].state_v = -70.0f;
// 		}
// 	}
// 	__syncthreads();
// }


// // Spike Generator Updating Kernel
// __global__ void genupdate(struct neuron_struct* d_neuronpop_variables,
// 							int* genids,
// 							float* gentimes,
// 							float currtime,
// 							float timestep,
// 							size_t numEntries){
// 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (idx < numEntries){
// 		// Check if the current time is one of the gen times
// 		if (fabs(currtime - gentimes[idx]) > 0.5*timestep) {
// 			// This sync seems absolutely necessary for when I spike inputs ... weird.
// 			d_neuronpop_variables[genids[idx]].state_u = 0.0f;
// 			d_neuronpop_variables[genids[idx]].state_v = -70.0f;
// 		} else {
// 			__syncthreads();
// 			d_neuronpop_variables[genids[idx]].state_u = 0.0f;
// 			d_neuronpop_variables[genids[idx]].state_v = 35.0f;
// 		}
// 	}
// }


// // Spiking Neurons
// __global__ void spikingneurons(struct neuron_struct* d_neuronpop_variables,
// 								float* d_lastspiketime,
// 								float currtime,
// 								size_t numNeurons){
// 	// Get thread IDs
// 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (idx < numNeurons) {
// 		// First checking if neuron has spiked:
// 		if (d_neuronpop_variables[idx].state_v >= 30.0f){
// 			// Reset the values of these neurons
// 			d_neuronpop_variables[idx].state_v = d_neuronpop_variables[idx].paramc;
// 			d_neuronpop_variables[idx].state_u += d_neuronpop_variables[idx].paramd;
// 			// Update the last spike times of these neurons
// 			d_lastspiketime[idx] = currtime;
// 		}
// 	}
// 	__syncthreads();
// }


// // State Update
// __global__ void stateupdate(struct neuron_struct* d_neuronpop_variables,
// 							float* currentinj,
// 							float timestep,
// 							size_t numNeurons){
// 	// We require the equation timestep in ms:
// 	float eqtimestep = timestep*1000.0f;
// 	// Get thread IDs
// 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (idx < numNeurons) {
// 		// Update the neuron states according to the Izhikevich equations
// 		float v_update = 0.04f*d_neuronpop_variables[idx].state_v*d_neuronpop_variables[idx].state_v + 
// 							5.0f*d_neuronpop_variables[idx].state_v + 140 - d_neuronpop_variables[idx].state_u + currentinj[idx];
// 		d_neuronpop_variables[idx].state_v += eqtimestep*v_update;
// 		d_neuronpop_variables[idx].state_u += eqtimestep*(d_neuronpop_variables[idx].parama * (d_neuronpop_variables[idx].paramb*d_neuronpop_variables[idx].state_v - 
// 							d_neuronpop_variables[idx].state_u));
// 	}
// 	__syncthreads();
// }

