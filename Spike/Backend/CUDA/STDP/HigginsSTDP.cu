#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/STDP/HigginsSTDP.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, HigginsSTDP);

namespace Backend {
  namespace CUDA {
    void HigginsSTDP::prepare() {
      STDP::prepare();
      // allocate_device_pointers();
    }

    void HigginsSTDP::reset_state() {
      STDP::reset_state();
    }

    void HigginsSTDP::apply_ltd_to_synapse_weights(float current_time_in_seconds) {
      izhikevich_apply_ltd_to_synapse_weights_kernel<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
        (synapses_backend->time_of_last_spike_to_reach_synapse,
         synapses_backend->synaptic_efficacies_or_weights,
         synapses_backend->stdp,
         neurons_backend->last_spike_time_of_each_neuron,
         synapses_backend->postsynaptic_neuron_indices,
         current_time_in_seconds,
         *(frontend()->stdp_params), // Should make device copy?
         frontend()->syns->total_number_of_synapses);

      CudaCheckError();
    }

    void HigginsSTDP::apply_ltp_to_synapse_weights(float current_time_in_seconds) {
      izhikevich_apply_ltp_to_synapse_weights_kernel<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
        (synapses_backend->postsynaptic_neuron_indices,
         neurons_backend->last_spike_time_of_each_neuron,
         synapses_backend->stdp,
         synapses_backend->time_of_last_spike_to_reach_synapse,
         synapses_backend->synaptic_efficacies_or_weights,
         *(frontend()->stdp_params),
         current_time_in_seconds,
         frontend()->syns->total_number_of_synapses);

      CudaCheckError();
    }

    // LTP on synapses
    __global__ void izhikevich_apply_ltp_to_synapse_weights_kernel
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     struct higgins_stdp_parameters_struct stdp_vars,
     float currtime,
     size_t total_number_of_synapse) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_synapse) {
        // Get the synapses upon which we should do LTP
        // Reversed indexing to check post->pre synapses
        if ((d_last_spike_time_of_each_neuron[d_postsyns[idx]] == currtime) && (d_stdp[idx] == true)){
          // Get the last active time / weight of the synapse
          // Calc time difference and weight change
          float diff = currtime - d_time_of_last_spike_to_reach_synapse[idx];
          float weightchange = (stdp_vars.w_max - d_synaptic_efficacies_or_weights[idx]) * (stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus));
          // Update weights
          d_synaptic_efficacies_or_weights[idx] += weightchange;
        }
        idx += blockDim.x * gridDim.x;

      }
    }

    // LTD on Synapses
    __global__ void izhikevich_apply_ltd_to_synapse_weights_kernel
    (float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     bool* d_stdp,
     float* d_last_spike_time_of_each_neuron,
     int* d_postsyns,
     float currtime,
     struct higgins_stdp_parameters_struct stdp_vars,
     size_t total_number_of_synapse){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_synapse) {

        // Get the locations for updating
        // Get the synapses that are to be LTD'd
        if ((d_time_of_last_spike_to_reach_synapse[idx] == currtime) && (d_stdp[idx] == 1)) {
          float diff = d_last_spike_time_of_each_neuron[d_postsyns[idx]] - currtime;
          // STDP Update Rule
          float weightscale = stdp_vars.w_max * stdp_vars.a_minus * expf(diff / stdp_vars.tau_minus);
          // Now scale the weight (using an inverted column/row)
          d_synaptic_efficacies_or_weights[idx] += weightscale; 
        }
        idx += blockDim.x * gridDim.x;
      }
    }

  }
}
