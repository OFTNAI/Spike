#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/STDP/HigginsSTDP.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, HigginsSTDP);

namespace Backend {
  namespace CUDA {
    void HigginsSTDP::prepare() {
      STDP::prepare();
      // allocate_device_pointers();
    }

    void HigginsSTDP::reset_state() {
      STDP::reset_state();
    }

    void HigginsSTDP::apply_ltd_to_synapse_weights(float current_time_in_seconds) {
      izhikevich_apply_ltd_to_synapse_weights_kernel<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
        (synapses_backend->time_of_last_spike_to_reach_synapse,
         synapses_backend->synaptic_efficacies_or_weights,
         synapses_backend->stdp,
         neurons_backend->last_spike_time_of_each_neuron,
         synapses_backend->postsynaptic_neuron_indices,
         current_time_in_seconds,
         *(frontend()->stdp_params), // Should make device copy?
         stdp_synapse_indices,
         total_number_of_stdp_synapses);

      CudaCheckError();
    }

    void HigginsSTDP::apply_ltp_to_synapse_weights(float current_time_in_seconds) {
      izhikevich_apply_ltp_to_synapse_weights_kernel<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
        (synapses_backend->postsynaptic_neuron_indices,
         neurons_backend->last_spike_time_of_each_neuron,
         synapses_backend->stdp,
         synapses_backend->time_of_last_spike_to_reach_synapse,
         synapses_backend->synaptic_efficacies_or_weights,
         *(frontend()->stdp_params),
         current_time_in_seconds,
         stdp_synapse_indices,
         total_number_of_stdp_synapses);

      CudaCheckError();
    }

    // LTP on synapses
    __global__ void izhikevich_apply_ltp_to_synapse_weights_kernel
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     struct higgins_stdp_parameters_struct stdp_vars,
     float currtime,
     int* d_stdp_synapse_indices,
     size_t total_number_of_stdp_synapses) {

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < total_number_of_stdp_synapses) {
        idx = d_stdp_synapse_indices[indx];
        // Get the synapses upon which we should do LTP
        // Reversed indexing to check post->pre synapses
        if ((d_last_spike_time_of_each_neuron[d_postsyns[idx]] == currtime) && (d_stdp[idx])){
          // Get the last active time / weight of the synapse
          // Calc time difference and weight change
          float diff = currtime - d_time_of_last_spike_to_reach_synapse[idx];
          float weightchange = (stdp_vars.w_max - d_synaptic_efficacies_or_weights[idx]) * (stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus));
          // Update weights
          d_synaptic_efficacies_or_weights[idx] += weightchange;
        }
        indx += blockDim.x * gridDim.x;

      }
    }

    // LTD on Synapses
    __global__ void izhikevich_apply_ltd_to_synapse_weights_kernel
    (float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     bool* d_stdp,
     float* d_last_spike_time_of_each_neuron,
     int* d_postsyns,
     float currtime,
     struct higgins_stdp_parameters_struct stdp_vars,
     int* d_stdp_synapse_indices,
     size_t total_number_of_stdp_synapses){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < total_number_of_stdp_synapses) {
        idx = d_stdp_synapse_indices[indx];
        // Get the locations for updating
        // Get the synapses that are to be LTD'd
        if ((d_time_of_last_spike_to_reach_synapse[idx] == currtime) && (d_stdp[idx])) {
          float diff = d_last_spike_time_of_each_neuron[d_postsyns[idx]] - currtime;
          // STDP Update Rule
          float weightscale = stdp_vars.w_max * stdp_vars.a_minus * expf(diff / stdp_vars.tau_minus);
          // Now scale the weight (using an inverted column/row)
          d_synaptic_efficacies_or_weights[idx] += weightscale; 
        }
        indx += blockDim.x * gridDim.x;
      }
    }

  }
}
