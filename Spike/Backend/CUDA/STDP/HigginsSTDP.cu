#include "hip/hip_runtime.h"
#include "Spike/Backend/CUDA/STDP/HigginsSTDP.hpp"

namespace Backend {
  namespace CUDA {
    // LTP on synapses
    __global__ void izhikevich_apply_ltp_to_synapse_weights_kernel
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     struct higgins_stdp_parameters_struct stdp_vars,
     float currtime,
     size_t total_number_of_synapse) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_synapse) {
        // Get the synapses upon which we should do LTP
        // Reversed indexing to check post->pre synapses
        if ((d_last_spike_time_of_each_neuron[d_postsyns[idx]] == currtime) && (d_stdp[idx] == true)){
          // Get the last active time / weight of the synapse
          // Calc time difference and weight change
          float diff = currtime - d_time_of_last_spike_to_reach_synapse[idx];
          float weightchange = (stdp_vars.w_max - d_synaptic_efficacies_or_weights[idx]) * (stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus));
          // Update weights
          d_synaptic_efficacies_or_weights[idx] += weightchange;
        }
        idx += blockDim.x * gridDim.x;

      }
    }

    // LTD on Synapses
    __global__ void izhikevich_apply_ltd_to_synapse_weights_kernel
    (float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     bool* d_stdp,
     float* d_last_spike_time_of_each_neuron,
     int* d_postsyns,
     float currtime,
     struct higgins_stdp_parameters_struct stdp_vars,
     size_t total_number_of_synapse){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_synapse) {

        // Get the locations for updating
        // Get the synapses that are to be LTD'd
        if ((d_time_of_last_spike_to_reach_synapse[idx] == currtime) && (d_stdp[idx] == 1)) {
          float diff = d_last_spike_time_of_each_neuron[d_postsyns[idx]] - currtime;
          // STDP Update Rule
          float weightscale = stdp_vars.w_max * stdp_vars.a_minus * expf(diff / stdp_vars.tau_minus);
          // Now scale the weight (using an inverted column/row)
          d_synaptic_efficacies_or_weights[idx] += weightscale; 
        }
        idx += blockDim.x * gridDim.x;
      }
    }

  }
}
