#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/STDP/vanRossumSTDP.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, vanRossumSTDP);

namespace Backend {
  namespace CUDA {
    vanRossumSTDP::~vanRossumSTDP() {
      CudaSafeCall(hipFree(index_of_last_afferent_synapse_to_spike));
      CudaSafeCall(hipFree(isindexed_ltd_synapse_spike));
      CudaSafeCall(hipFree(index_of_first_synapse_spiked_after_postneuron));
    }

    void vanRossumSTDP::reset_state() {
      STDP::reset_state();

      CudaSafeCall(hipMemcpy((void*)index_of_last_afferent_synapse_to_spike,
                              (void*)frontend()->index_of_last_afferent_synapse_to_spike,
                              sizeof(int)*frontend()->neurs->total_number_of_neurons,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)isindexed_ltd_synapse_spike,
                              (void*)frontend()->isindexed_ltd_synapse_spike,
                              sizeof(bool)*frontend()->neurs->total_number_of_neurons,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)index_of_first_synapse_spiked_after_postneuron,
                              (void*)frontend()->index_of_first_synapse_spiked_after_postneuron,
                              sizeof(int)*frontend()->neurs->total_number_of_neurons,
                              hipMemcpyHostToDevice));
    }

    void vanRossumSTDP::prepare() {
      STDP::prepare();

      allocate_device_pointers();
    }

    void vanRossumSTDP::allocate_device_pointers() {
      // The following doesn't do anything in original code...
      // ::Backend::CUDA::STDP::allocate_device_pointers();

      CudaSafeCall(hipMalloc((void **)&index_of_last_afferent_synapse_to_spike, sizeof(int)*frontend()->neurs->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&isindexed_ltd_synapse_spike, sizeof(int)*frontend()->neurs->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&index_of_first_synapse_spiked_after_postneuron, sizeof(int)*frontend()->neurs->total_number_of_neurons));
    }

    void vanRossumSTDP::apply_stdp_to_synapse_weights(float current_time_in_seconds) {
      // First reset the indices array
      // In order to carry out nearest spike potentiation only, we must find the spike arriving at each neuron which has the smallest time diff
    vanrossum_get_indices_to_apply_stdp<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
      (synapses_backend->postsynaptic_neuron_indices,
       neurons_backend->last_spike_time_of_each_neuron,
       synapses_backend->time_of_last_spike_to_reach_synapse,
       index_of_last_afferent_synapse_to_spike,
       isindexed_ltd_synapse_spike,
       index_of_first_synapse_spiked_after_postneuron,
       current_time_in_seconds,
       stdp_synapse_indices,
       total_number_of_stdp_synapses);
    CudaCheckError();

    vanrossum_apply_stdp_to_synapse_weights_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
      (synapses_backend->postsynaptic_neuron_indices,
       neurons_backend->last_spike_time_of_each_neuron,
       synapses_backend->stdp,
       synapses_backend->time_of_last_spike_to_reach_synapse,
       synapses_backend->synaptic_efficacies_or_weights,
       index_of_last_afferent_synapse_to_spike,
       isindexed_ltd_synapse_spike,
       index_of_first_synapse_spiked_after_postneuron,
       *(frontend()->stdp_params),
       current_time_in_seconds,
       frontend()->neurs->total_number_of_neurons);
    CudaCheckError();
    }

    // Find nearest spike
    __global__ void vanrossum_apply_stdp_to_synapse_weights_kernel
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     int* d_index_of_last_afferent_synapse_to_spike,
     bool* d_isindexed_ltd_synapse_spike,
     int* d_index_of_first_synapse_spiked_after_postneuron,
     struct vanrossum_stdp_parameters_struct stdp_vars,
     float currtime,
     size_t total_number_of_post_neurons){
      // Global Index
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all neurons
      while (idx < total_number_of_post_neurons) {
        // Check whether a neuron has fired, if so: reset flag
        if (d_last_spike_time_of_each_neuron[idx] == currtime){
          d_isindexed_ltd_synapse_spike[idx] = false;
        }

        // Get the synapse on which to do LTP/LTD
        int index_of_LTP_synapse = d_index_of_last_afferent_synapse_to_spike[idx];
        int index_of_LTD_synapse = d_index_of_first_synapse_spiked_after_postneuron[idx];

        // If we are to carry out STDP on LTP synapse
        if (index_of_LTP_synapse >= 0){
          if(d_stdp[index_of_LTP_synapse]){
            float last_syn_spike_time = d_time_of_last_spike_to_reach_synapse[index_of_LTP_synapse];
            float last_neuron_spike_time = d_last_spike_time_of_each_neuron[idx];
            float new_syn_weight = d_synaptic_efficacies_or_weights[index_of_LTP_synapse];

            if (last_neuron_spike_time == currtime){
              float diff = currtime - last_syn_spike_time;
              // Only carry out LTP if the difference is greater than some range
              if (diff < 7*stdp_vars.tau_plus && diff > 0){
                float weightchange = stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus);
                // Update weights
                new_syn_weight += weightchange;
                // Ensure that the weights are clipped to 1.0f
                new_syn_weight = min(new_syn_weight, 1.0f);
              }
            }
            // Update the synaptic weight as required
            d_synaptic_efficacies_or_weights[index_of_LTP_synapse] = new_syn_weight;
          }
        }

        // Get the synapse for LTD
        if (d_isindexed_ltd_synapse_spike[idx]){
          if (index_of_LTD_synapse >= 0){
            if (d_stdp[index_of_LTD_synapse]){

              float last_syn_spike_time = d_time_of_last_spike_to_reach_synapse[index_of_LTD_synapse];
              float last_neuron_spike_time = d_last_spike_time_of_each_neuron[idx];
              float new_syn_weight = d_synaptic_efficacies_or_weights[index_of_LTD_synapse];

              // Set the index to negative (i.e. Reset it)
              d_index_of_first_synapse_spiked_after_postneuron[idx] = -1;

              float diff = last_syn_spike_time - last_neuron_spike_time;
              // Only carry out LTD if the difference is in some range
              if (diff < 7*stdp_vars.tau_minus && diff > 0){
                float weightchange = new_syn_weight * stdp_vars.a_minus * expf(-diff / stdp_vars.tau_minus);
                // Update the weights
                new_syn_weight -= weightchange;
              }
              d_synaptic_efficacies_or_weights[index_of_LTD_synapse] = new_syn_weight;
            }
          }
        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }

    __global__ void vanrossum_get_indices_to_apply_stdp
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     float* d_time_of_last_spike_to_reach_synapse,
     int* d_index_of_last_afferent_synapse_to_spike,
     bool* d_isindexed_ltd_synapse_spike,
     int* d_index_of_first_synapse_spiked_after_postneuron,
     float currtime,
     int* d_stdp_synapse_indices,
     size_t total_number_of_stdp_synapses){
      int indx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running through all synapses:
      while (indx < total_number_of_stdp_synapses){
        int idx = d_stdp_synapse_indices[indx];
        int postsynaptic_neuron = d_postsyns[idx];

        // Check whether a synapse reached a neuron this timestep
        if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
          // Atomic Exchange the new synapse index
          atomicExch(&d_index_of_last_afferent_synapse_to_spike[postsynaptic_neuron], idx);
        }

        // Check (if we need to) whether a synapse has fired
        if (!d_isindexed_ltd_synapse_spike[postsynaptic_neuron]){
          if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
            d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = true;
            atomicExch(&d_index_of_first_synapse_spiked_after_postneuron[postsynaptic_neuron], idx);
          }
        }
        // Increment index
        idnx += blockDim.x * gridDim.x;
      }
    }
}
