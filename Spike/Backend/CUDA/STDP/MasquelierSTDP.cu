#include "hip/hip_runtime.h"
#include "Spike/Backend/CUDA/STDP/MasquelierSTDP.hpp"

namespace Backend {
  namespace CUDA {
    MasquelierSTDP::~MasquelierSTDP() {
      // TODO
    }

    void MasquelierSTDP::reset_state() {
      CudaSafeCall(hipMemcpy((void*)d_index_of_last_afferent_synapse_to_spike, (void*)index_of_last_afferent_synapse_to_spike, sizeof(int)*neurs->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)d_isindexed_ltd_synapse_spike, (void*)isindexed_ltd_synapse_spike, sizeof(bool)*neurs->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)d_index_of_first_synapse_spiked_after_postneuron, (void*)index_of_first_synapse_spiked_after_postneuron, sizeof(int)*neurs->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void MasquelierSTDP::allocate_device_pointers() {
      // TODO: Check the following (doesn't do anything in original code?...)
      STDP::allocate_device_pointers();

      // **** TODO: The following appears to be host code ... ****
      // Add the correct space for last synapse
      index_of_last_afferent_synapse_to_spike = (int*)malloc(sizeof(int)*neurs->total_number_of_neurons);
      isindexed_ltd_synapse_spike = (bool*)malloc(sizeof(bool)*neurs->total_number_of_neurons);
      index_of_first_synapse_spiked_after_postneuron = (int*)malloc(sizeof(int)*neurs->total_number_of_neurons);
      // **** END apparent host code ****

      CudaSafeCall(hipMalloc((void **)&d_index_of_last_afferent_synapse_to_spike, sizeof(int)*neurs->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_isindexed_ltd_synapse_spike, sizeof(int)*neurs->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_index_of_first_synapse_spiked_after_postneuron, sizeof(int)*neurs->total_number_of_neurons));

      // **** TODO: The following appears to be host code ... ****
      // Initialize indices
      for (int i=0; i < neurs->total_number_of_neurons; i++){
        index_of_last_afferent_synapse_to_spike[i] = -1;
        isindexed_ltd_synapse_spike[i] = false;
        index_of_first_synapse_spiked_after_postneuron[i] = -1;
      }
      // **** END apparent host code ****
    }

    // Find nearest spike
    __global__ void apply_stdp_to_synapse_weights_kernel
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     float* d_synaptic_efficacies_or_weights,
     int* d_index_of_last_afferent_synapse_to_spike,
     bool* d_isindexed_ltd_synapse_spike,
     int* d_index_of_first_synapse_spiked_after_postneuron,
     struct masquelier_stdp_parameters_struct stdp_vars,
     float currtime,
     size_t total_number_of_post_neurons){
      // Global Index
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all synapses
      while (idx < total_number_of_post_neurons) {
        // Get the synapse on which to do LTP/LTD
        int index_of_LTP_synapse = d_index_of_last_afferent_synapse_to_spike[idx];
        int index_of_LTD_synapse = d_index_of_first_synapse_spiked_after_postneuron[idx];

        // If we are to carry out STDP on LTP synapse
        if(d_stdp[index_of_LTP_synapse]){
          float last_syn_spike_time = d_time_of_last_spike_to_reach_synapse[index_of_LTP_synapse];
          float last_neuron_spike_time = d_last_spike_time_of_each_neuron[idx];
          float new_syn_weight = d_synaptic_efficacies_or_weights[index_of_LTP_synapse];

          if (last_neuron_spike_time == currtime){
            float diff = currtime - last_syn_spike_time;
            // Only carry out LTP if the difference is greater than some range
            if (diff < 7*stdp_vars.tau_plus && diff > 0){
              float weightchange = stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus);
              // Update weights
              new_syn_weight += weightchange;
              // Ensure that the weights are clipped to 1.0f
              new_syn_weight = min(new_syn_weight, 1.0f);
            }
          }
          // Update the synaptic weight as required
          d_synaptic_efficacies_or_weights[index_of_LTP_synapse] = new_syn_weight;
        }

        // Get the synapse for LTD
        if (d_isindexed_ltd_synapse_spike[idx]){
          if (index_of_LTD_synapse >= 0){
            if (d_stdp[index_of_LTD_synapse]){

              float last_syn_spike_time = d_time_of_last_spike_to_reach_synapse[index_of_LTD_synapse];
              float last_neuron_spike_time = d_last_spike_time_of_each_neuron[idx];
              float new_syn_weight = d_synaptic_efficacies_or_weights[index_of_LTD_synapse];

              // Set the index to negative (i.e. Reset it)
              d_index_of_first_synapse_spiked_after_postneuron[idx] = -1;

              float diff = last_syn_spike_time - last_neuron_spike_time;
              // Only carry out LTD if the difference is in some range
              if (diff < 7*stdp_vars.tau_minus && diff > 0){
                float weightchange = stdp_vars.a_minus * expf(-diff / stdp_vars.tau_minus);
                // Update the weights
                new_syn_weight -= weightchange;
                // Ensure that the weights are clipped to 0.0f
                new_syn_weight = max(new_syn_weight, 0.0f);
              }
              d_synaptic_efficacies_or_weights[index_of_LTD_synapse] = new_syn_weight;
            }
          }	
        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }

    __global__ void get_indices_to_apply_stdp
    (int* d_postsyns,
     float* d_last_spike_time_of_each_neuron,
     bool* d_stdp,
     float* d_time_of_last_spike_to_reach_synapse,
     int* d_index_of_last_afferent_synapse_to_spike,
     bool* d_isindexed_ltd_synapse_spike,
     int* d_index_of_first_synapse_spiked_after_postneuron,
     float currtime,
     size_t total_number_of_synapse){
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running through all neurons:
      while (idx < total_number_of_synapse){
        int postsynaptic_neuron = d_postsyns[idx];

        // Check whether a synapse reached a neuron this timestep
        if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
          // Atomic Exchange the new synapse index
          atomicExch(&d_index_of_last_afferent_synapse_to_spike[postsynaptic_neuron], idx);
        }
		
        // Check (if we need to) whether a synapse has fired
        if (d_isindexed_ltd_synapse_spike[postsynaptic_neuron]){
          if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
            d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = true;
            atomicExch(&d_index_of_first_synapse_spiked_after_postneuron[postsynaptic_neuron], idx);
          }
        }
        // Check whether a neuron has fired
        if (d_last_spike_time_of_each_neuron[postsynaptic_neuron] == currtime){
          d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = false;
        }
        // Increment index
        idx += blockDim.x * gridDim.x;
      }
    }
  }
}
