#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/ActivityMonitor/SpikingActivityMonitor.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingActivityMonitor);

namespace Backend {
  namespace CUDA {
    SpikingActivityMonitor::~SpikingActivityMonitor() {
      CudaSafeCall(hipFree(neuron_ids_of_stored_spikes_on_device));
      CudaSafeCall(hipFree(total_number_of_spikes_stored_on_device));
      CudaSafeCall(hipFree(time_in_seconds_of_stored_spikes_on_device));
    }

    void SpikingActivityMonitor::reset_state() {
      ActivityMonitor::reset_state();

      CudaSafeCall(hipMemset(&(total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
      CudaSafeCall(hipMemcpy(neuron_ids_of_stored_spikes_on_device, frontend()->reset_neuron_ids, sizeof(int)*frontend()->size_of_device_spike_store, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(time_in_seconds_of_stored_spikes_on_device, frontend()->reset_neuron_times, sizeof(float)*frontend()->size_of_device_spike_store, hipMemcpyHostToDevice));
    }

    void SpikingActivityMonitor::prepare() {
      neurons_frontend = frontend()->neurons;
      neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons_frontend->backend());
      ActivityMonitor::prepare();

      CudaSafeCall(hipMalloc((void **)&neuron_ids_of_stored_spikes_on_device, sizeof(int)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&time_in_seconds_of_stored_spikes_on_device, sizeof(float)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&total_number_of_spikes_stored_on_device, sizeof(int)));
    
      reset_state();
    }
   
    void SpikingActivityMonitor::copy_spikecount_to_front(){
      CudaSafeCall(hipMemcpy((void*)&(frontend()->total_number_of_spikes_stored_on_device[0]), 
                              total_number_of_spikes_stored_on_device, 
                              sizeof(int), hipMemcpyDeviceToHost));
    }

    void SpikingActivityMonitor::copy_spikes_to_front() {
      CudaSafeCall(hipMemcpy((void*)&frontend()->neuron_ids_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              neuron_ids_of_stored_spikes_on_device, 
                              (sizeof(int)*frontend()->total_number_of_spikes_stored_on_device[0]), 
                              hipMemcpyDeviceToHost));
      CudaSafeCall(hipMemcpy((void*)&frontend()->spike_times_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              time_in_seconds_of_stored_spikes_on_device, 
                              sizeof(float)*frontend()->total_number_of_spikes_stored_on_device[0], 
                              hipMemcpyDeviceToHost));
    }

    void SpikingActivityMonitor::collect_spikes_for_timestep
    (float current_time_in_seconds, float timestep) {
      collect_spikes_for_timestep_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
        (neurons_backend->d_neuron_data,
         total_number_of_spikes_stored_on_device,
         neuron_ids_of_stored_spikes_on_device,
         time_in_seconds_of_stored_spikes_on_device,
         frontend()->model->timestep_grouping,
         current_time_in_seconds,
         timestep,
         neurons_frontend->total_number_of_neurons);

      CudaCheckError();
    }


    // Collect Spikes
    __global__ void collect_spikes_for_timestep_kernel
    (spiking_neurons_data_struct* neuron_data,
     int* d_total_number_of_spikes_stored_on_device,
     int* d_neuron_ids_of_stored_spikes_on_device,
     float* d_time_in_seconds_of_stored_spikes_on_device,
     int timestep_grouping,
     float current_time_in_seconds,
     float timestep,
     size_t total_number_of_neurons){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      int bufsize = neuron_data->neuron_spike_time_bitbuffer_bytesize[0];
      while (idx < total_number_of_neurons) {
        for (int g=0; g < timestep_grouping; g++){
          int bitloc = ((int)roundf(current_time_in_seconds / timestep) + g) % (8*bufsize);
          // If a neuron has fired
          if (neuron_data->neuron_spike_time_bitbuffer[idx*bufsize + (bitloc / 8)] & (1 << (bitloc % 8))){
            // Increase the number of spikes stored
            // NOTE: atomicAdd return value is actually original (atomic) value BEFORE incrementation!
            //    - So first value is actually 0 not 1!!!
            int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);

            // In the location, add the id and the time
            d_neuron_ids_of_stored_spikes_on_device[i] = idx;
            d_time_in_seconds_of_stored_spikes_on_device[i] = current_time_in_seconds + g*timestep;
          }
        }
        idx += blockDim.x * gridDim.x;
      }
    }
  }
}
