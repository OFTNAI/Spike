#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/ActivityMonitor/RateActivityMonitor.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, RateActivityMonitor);

namespace Backend {
  namespace CUDA {
    RateActivityMonitor::~RateActivityMonitor() {
      CudaSafeCall(hipFree(per_neuron_spike_counts));
    }

    void RateActivityMonitor::reset_state() {
      ActivityMonitor::reset_state();

      CudaSafeCall(hipMemset(per_neuron_spike_counts, 0, sizeof(int) * frontend()->neurons->total_number_of_neurons));
    }

    void RateActivityMonitor::prepare() {
      ActivityMonitor::prepare();
      allocate_pointers_for_spike_count();
    }

    void RateActivityMonitor::allocate_pointers_for_spike_count() {
      CudaSafeCall(hipMalloc((void **)&per_neuron_spike_counts,
                              sizeof(int) * frontend()->neurons->total_number_of_neurons));
    }

    void RateActivityMonitor::copy_spike_count_to_host(){
      CudaSafeCall(hipMemcpy((void*)&(frontend()->per_neuron_spike_counts[0]), 
                              per_neuron_spike_counts, 
                              sizeof(int)*frontend()->neurons->total_number_of_neurons,
                              hipMemcpyDeviceToHost));
    }

    void RateActivityMonitor::add_spikes_to_per_neuron_spike_count
    (float current_time_in_seconds, float timestep) {
      add_spikes_to_per_neuron_spike_count_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
        (neurons_backend->d_neuron_data,
         per_neuron_spike_counts,
         current_time_in_seconds,
         timestep,
         frontend()->model->timestep_grouping,
         frontend()->neurons->total_number_of_neurons);
         CudaCheckError();
    }

    __global__ void add_spikes_to_per_neuron_spike_count_kernel
    (spiking_neurons_data_struct* neuron_data,
     int* d_per_neuron_spike_counts,
     float current_time_in_seconds,
     float timestep,
     int timestep_grouping,
     size_t total_number_of_neurons) {
      
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      int bufsize = neuron_data->neuron_spike_time_bitbuffer_bytesize[0];
      while (idx < total_number_of_neurons) {
        for (int g=0; g < timestep_grouping; g++){
          int bitloc = ((int)roundf(current_time_in_seconds / timestep) + g) % (8*bufsize);
          // If a neuron has fired
          if (neuron_data->neuron_spike_time_bitbuffer[idx*bufsize + (bitloc / 8)] & (1 << (bitloc % 8))){
            atomicAdd(&d_per_neuron_spike_counts[idx], 1);
          }
        }

        idx += blockDim.x * gridDim.x;
      }
    }

  }
}

