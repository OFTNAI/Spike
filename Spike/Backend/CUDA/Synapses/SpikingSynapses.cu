#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/SpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingSynapses);

namespace Backend {
  namespace CUDA {

    __device__ injection_kernel spiking_device_kernel = spiking_current_injection_kernel;

    SpikingSynapses::SpikingSynapses() {
    }

    SpikingSynapses::~SpikingSynapses() {
      CudaSafeCall(hipFree(delays));
      CudaSafeCall(hipFree(d_syn_labels));
      CudaSafeCall(hipFree(time_of_last_spike_to_reach_synapse));
      CudaSafeCall(hipFree(group_indices));
      CudaSafeCall(hipFree(num_active_synapses));
      CudaSafeCall(hipFree(num_activated_neurons));
      CudaSafeCall(hipFree(active_synapse_counts));
      CudaSafeCall(hipFree(presynaptic_neuron_indices));
      CudaSafeCall(hipFree(neuron_inputs.circular_input_buffer));
      CudaSafeCall(hipFree(d_synaptic_data));
    }

    void SpikingSynapses::reset_state() {
      Synapses::reset_state();

      // Set last spike times to -1000 so that the times do not affect current simulation.
      float *last_spike_to_reach_synapse = (float*)malloc(frontend()->total_number_of_synapses*sizeof(float));
      for (int i=0; i < frontend()->total_number_of_synapses; i++)
        last_spike_to_reach_synapse[i] = -1000.0f;

      CudaSafeCall(hipMemcpy(time_of_last_spike_to_reach_synapse,
        last_spike_to_reach_synapse,
        frontend()->total_number_of_synapses*sizeof(float),
        hipMemcpyHostToDevice));
      free(last_spike_to_reach_synapse);
      
      // Spike Buffer Resetting
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_activated_neurons, 0, sizeof(int)));
      CudaSafeCall(hipMemset(neuron_inputs.circular_input_buffer, 0.0f, sizeof(float)*neuron_inputs.temporal_buffersize*neuron_inputs.input_buffersize));
      CudaSafeCall(hipMemset(neuron_inputs.bufferloc, 0, sizeof(int)));
    }

    void SpikingSynapses::copy_weights_to_host() {
      CudaSafeCall(hipMemcpy(frontend()->synaptic_efficacies_or_weights,
        synaptic_efficacies_or_weights,
        sizeof(float)*frontend()->total_number_of_synapses,
        hipMemcpyDeviceToHost));
    }

    void SpikingSynapses::prepare() {
      Synapses::prepare();
     
      // Extra buffer size for current time and extra to reset before last
      buffersize = frontend()->maximum_axonal_delay_in_timesteps + 2*frontend()->model->timestep_grouping + 1;
      neuron_inputs.input_buffersize = frontend()->neuron_pop_size*frontend()->num_syn_labels;
      neuron_inputs.temporal_buffersize = buffersize;
      
      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();

      synaptic_data = new spiking_synapses_data_struct();
      synaptic_data->num_syn_labels = frontend()->num_syn_labels;
      synaptic_data->neuron_inputs = neuron_inputs;
      CudaSafeCall(hipMemcpy(d_synaptic_data,
        synaptic_data,
        sizeof(spiking_synapses_data_struct),
        hipMemcpyHostToDevice));
         
    }

    void SpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&delays, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&d_syn_labels, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&time_of_last_spike_to_reach_synapse, sizeof(float)*frontend()->total_number_of_synapses));
      // Device pointers for spike buffer and active synapse/neuron storage
      CudaSafeCall(hipMalloc((void **)&group_indices, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&num_active_synapses, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&num_activated_neurons, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&active_synapse_counts, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&presynaptic_neuron_indices, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&d_synaptic_data, sizeof(spiking_synapses_data_struct)));
      // Setting injection kernel
      CudaSafeCall(hipMemcpyFromSymbol(
        &host_injection_kernel,
        HIP_SYMBOL(spiking_device_kernel),
        sizeof(injection_kernel)));
      CudaSafeCall(hipMalloc((void **)&neuron_inputs.circular_input_buffer, sizeof(float)*neuron_inputs.temporal_buffersize*neuron_inputs.input_buffersize));
      CudaSafeCall(hipMalloc((void **)&neuron_inputs.bufferloc, sizeof(int)));
    }

    void SpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
      CudaSafeCall(hipMemcpy(delays, frontend()->delays,
        sizeof(int)*frontend()->total_number_of_synapses,
        hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        d_syn_labels,
        frontend()->syn_labels,
        sizeof(int)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
    }

    void SpikingSynapses::state_update
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {
      
      // Calculate buffer location
      int bufferloc = (int)(std::round(current_time_in_seconds / timestep)) % buffersize;
      //synaptic_data->neuron_inputs = neuron_inputs;


      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);
      ::Backend::CUDA::SpikingNeurons* input_neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(input_neurons->backend());
      assert(input_neurons_backend);

      // Steps for the synapses to cary out:
      // - Get the active synapses
      // - Update the delay sets based upon these
      // - Add any current where necessary (atomically)
      // - Deliver current to destination
      get_active_synapses_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, threads_per_block>>>(
        neurons_backend->per_neuron_efferent_synapse_count,
        input_neurons_backend->per_neuron_efferent_synapse_count,
        neurons_backend->last_spike_time_of_each_neuron,
        input_neurons_backend->last_spike_time_of_each_neuron,
        current_time_in_seconds,
        timestep,
        input_neurons_backend->frontend()->total_number_of_neurons,
        group_indices,
        num_active_synapses,
        num_activated_neurons,
        active_synapse_counts,
        presynaptic_neuron_indices,
        (neurons_backend->frontend()->total_number_of_neurons + input_neurons_backend->frontend()->total_number_of_neurons));
      CudaCheckError();
      /*
      CudaSafeCall(hipMemcpy(
          &h_num_active_synapses,
          num_active_synapses,
          sizeof(int), hipMemcpyDeviceToHost));
      int blocks_per_grid = ((h_num_active_synapses / threads_per_block.x) + 1);
      if (blocks_per_grid > max_num_blocks_per_grid) blocks_per_grid = max_num_blocks_per_grid;
      */
      //activate_synapses<<<blocks_per_grid, threads_per_block>>>(
      activate_synapses<<<neurons_backend->number_of_neuron_blocks_per_grid, threads_per_block>>>(
        neurons_backend->per_neuron_efferent_synapse_total,
        neurons_backend->per_neuron_efferent_synapse_indices,
        input_neurons_backend->per_neuron_efferent_synapse_total,
        input_neurons_backend->per_neuron_efferent_synapse_indices,
        bufferloc,
        buffersize,
        synaptic_data->neuron_inputs,
        postsynaptic_neuron_indices,
        synaptic_efficacies_or_weights,
        weight_scaling_constants,
        time_of_last_spike_to_reach_synapse,
        delays,
        d_syn_labels,
        timestep,
        current_time_in_seconds,
        frontend()->total_number_of_synapses,
        neurons_backend->frontend()->total_number_of_neurons,
        group_indices,
        frontend()->model->timestep_grouping,
        presynaptic_neuron_indices,
        active_synapse_counts,
        num_active_synapses);
      CudaCheckError();
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_activated_neurons, 0, sizeof(int)));
      
    }
      
    
    __global__ void get_active_synapses_kernel(
      int* d_per_neuron_efferent_synapse_count,
      int* d_per_input_neuron_efferent_synapse_count,
      float* d_last_spike_time_of_each_neuron,
      float* d_last_spike_time_of_each_input_neuron,
      float current_time_in_seconds,
      float timestep,
      int num_input_neurons,
      int* group_indices,
      int* num_active_synapses,
      int* num_activated_neurons,
      int* active_synapse_counts,
      int* presynaptic_neuron_indices,
      size_t total_number_of_neurons) {

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < total_number_of_neurons) {
    
    int idx = indx - (num_input_neurons); 
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(idx);
  int corr_idx = CORRECTED_PRESYNAPTIC_ID(idx, presynaptic_is_input);
        float effecttime = presynaptic_is_input ? d_last_spike_time_of_each_input_neuron[corr_idx] : d_last_spike_time_of_each_neuron[corr_idx];
         
  int groupindex = (int)lroundf((effecttime - current_time_in_seconds) / timestep);
        if (groupindex >= 0){
            int synapse_count = presynaptic_is_input ? d_per_input_neuron_efferent_synapse_count[corr_idx] : d_per_neuron_efferent_synapse_count[corr_idx];
            int pos = atomicAdd(&num_activated_neurons[0], 1);
      atomicAdd(&num_active_synapses[0], synapse_count);
      active_synapse_counts[pos] = synapse_count;
      presynaptic_neuron_indices[pos] = idx;
      group_indices[pos] = groupindex;
        }
        indx += blockDim.x * gridDim.x;
      }
    }

    __global__ void activate_synapses(
        int* d_per_neuron_efferent_synapse_total,
        int* d_per_neuron_efferent_synapse_indices,
        int* d_per_input_neuron_efferent_synapse_total,
        int* d_per_input_neuron_efferent_synapse_indices,
        int bufferloc,
        int buffersize,
        neuron_inputs_struct neuron_inputs,
        int* postsynaptic_neuron_indices,
        float* synaptic_efficacies_or_weights,
        float* weight_scaling_constants,
        float* d_time_of_last_spike_to_reach_synapse,
        int* d_delays,
        int * d_syn_labels,
        float timestep,
        float current_time_in_seconds,
        int total_number_of_synapses,
        int total_number_of_neurons,
        int* group_indices,
        int timestep_grouping,
        int* presynaptic_neuron_indices,
        int* active_synapse_counts,
        int* num_active_synapses)
    {
      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      if (indx == 0){
        neuron_inputs.bufferloc[0] = (bufferloc + timestep_grouping) % buffersize;
      }

      int pos = 0;
      int idx = indx;
      while (indx < num_active_synapses[0]) {
  
        int synapse_count = active_synapse_counts[pos];

        while(idx >= synapse_count){
          idx -= synapse_count;
          pos += 1;
          synapse_count = active_synapse_counts[pos];
        }

        int neuron = presynaptic_neuron_indices[pos];
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(neuron);
        int corr_idx = CORRECTED_PRESYNAPTIC_ID(neuron, presynaptic_is_input);

        int synapse_id = presynaptic_is_input ? d_per_input_neuron_efferent_synapse_indices[d_per_input_neuron_efferent_synapse_total[corr_idx] - idx - 1] : d_per_neuron_efferent_synapse_indices[d_per_neuron_efferent_synapse_total[corr_idx] - idx - 1];
        int postneuron = postsynaptic_neuron_indices[synapse_id];

        int targetloc = (bufferloc + d_delays[synapse_id] + group_indices[pos]) % buffersize;

        int syn_label = d_syn_labels[synapse_id];
        float weightinput = weight_scaling_constants[synapse_id]*synaptic_efficacies_or_weights[synapse_id];
        atomicAdd(&neuron_inputs.circular_input_buffer[targetloc*neuron_inputs.input_buffersize + syn_label*total_number_of_neurons + postneuron], weightinput);
        d_time_of_last_spike_to_reach_synapse[synapse_id] = current_time_in_seconds + (d_delays[synapse_id] + group_indices[pos])*timestep;

        indx += blockDim.x * gridDim.x;
        idx += blockDim.x * gridDim.x;
      }

    }

      __device__ float spiking_current_injection_kernel(
  spiking_synapses_data_struct* synaptic_data,
  spiking_neurons_data_struct* neuron_data,
  float current_membrane_voltage,
  float current_time_in_seconds,
  float timestep,
  float multiplication_to_volts,
  int timestep_grouping,
  int idx,
  int g){
        return 0.0f;
      };

  }
}
