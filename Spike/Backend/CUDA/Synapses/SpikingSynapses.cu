#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/SpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingSynapses);

namespace Backend {
  namespace CUDA {

    __device__ injection_kernel spiking_device_kernel = spiking_current_injection_kernel;
    __device__ synaptic_activation_kernel spiking_syn_activation_kernel = get_active_synapses;

    SpikingSynapses::SpikingSynapses() {
    }

    SpikingSynapses::~SpikingSynapses() {
      CudaSafeCall(hipFree(delays));
      CudaSafeCall(hipFree(d_syn_labels));
      CudaSafeCall(hipFree(group_indices));
      CudaSafeCall(hipFree(num_active_synapses));
      CudaSafeCall(hipFree(num_activated_neurons));
      CudaSafeCall(hipFree(active_synapse_counts));
      CudaSafeCall(hipFree(active_synapse_starts));
      CudaSafeCall(hipFree(active_presynaptic_neuron_indices));
      CudaSafeCall(hipFree(neuron_inputs.circular_input_buffer));
      CudaSafeCall(hipFree(d_synaptic_data));
    }

    void SpikingSynapses::reset_state() {
      Synapses::reset_state();

      // Spike Buffer Resetting
      //CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_activated_neurons, 0, 2*sizeof(int)));
      CudaSafeCall(hipMemset(neuron_inputs.circular_input_buffer, 0.0f, sizeof(float)*neuron_inputs.temporal_buffersize*neuron_inputs.input_buffersize));
    }

    void SpikingSynapses::copy_weights_to_host() {
      CudaSafeCall(hipMemcpy(frontend()->synaptic_efficacies_or_weights,
        synaptic_efficacies_or_weights,
        sizeof(float)*frontend()->total_number_of_synapses,
        hipMemcpyDeviceToHost));
    }

    void SpikingSynapses::prepare() {
      Synapses::prepare();
     
      // Extra buffer size for current time and extra to reset before last
      buffersize = frontend()->maximum_axonal_delay_in_timesteps + 2*frontend()->model->timestep_grouping + 1;
      neuron_inputs.input_buffersize = frontend()->neuron_pop_size*frontend()->num_syn_labels;
      neuron_inputs.temporal_buffersize = buffersize;
      
      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();

      synaptic_data = new spiking_synapses_data_struct();
      synaptic_data->synapse_type = EMPTY;
      synaptic_data->syn_labels = d_syn_labels;
      synaptic_data->num_syn_labels = frontend()->num_syn_labels;
      synaptic_data->neuron_inputs = neuron_inputs;
      synaptic_data->num_activated_neurons = num_activated_neurons;
      synaptic_data->num_active_synapses = num_active_synapses;
      synaptic_data->active_presynaptic_neuron_indices = active_presynaptic_neuron_indices;
      synaptic_data->active_synapse_counts = active_synapse_counts;
      synaptic_data->active_synapse_starts = active_synapse_starts;
      synaptic_data->group_indices = group_indices;
      synaptic_data->postsynaptic_neuron_indices = postsynaptic_neuron_indices;
      synaptic_data->delays = delays;
      synaptic_data->synaptic_efficacies_or_weights = synaptic_efficacies_or_weights;
      synaptic_data->weight_scaling_constants = weight_scaling_constants;


      CudaSafeCall(hipMemcpy(d_synaptic_data,
        synaptic_data,
        sizeof(spiking_synapses_data_struct),
        hipMemcpyHostToDevice));
         
    }

    void SpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&delays, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&d_syn_labels, sizeof(int)*frontend()->total_number_of_synapses));
      // Device pointers for spike buffer and active synapse/neuron storage
      CudaSafeCall(hipMalloc((void **)&group_indices, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&num_active_synapses, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&num_activated_neurons, 2*sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&active_synapse_counts, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&active_synapse_starts, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&active_presynaptic_neuron_indices, sizeof(int)*(frontend()->total_number_of_synapses)));
      CudaSafeCall(hipMalloc((void **)&d_synaptic_data, sizeof(spiking_synapses_data_struct)));
      // Setting injection kernel
      CudaSafeCall(hipMemcpyFromSymbol(
        &host_injection_kernel,
        HIP_SYMBOL(spiking_device_kernel),
        sizeof(injection_kernel)));
      // Setting injection kernel
      CudaSafeCall(hipMemcpyFromSymbol(
        &host_syn_activation_kernel,
        HIP_SYMBOL(spiking_syn_activation_kernel),
        sizeof(synaptic_activation_kernel)));

      CudaSafeCall(hipMalloc((void **)&neuron_inputs.circular_input_buffer, sizeof(float)*neuron_inputs.temporal_buffersize*neuron_inputs.input_buffersize));
    }

    void SpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
      CudaSafeCall(hipMemcpy(delays, frontend()->delays,
        sizeof(int)*frontend()->total_number_of_synapses,
        hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        d_syn_labels,
        frontend()->syn_labels,
        sizeof(int)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
      int max_efferents = max(frontend()->model->spiking_neurons->max_num_efferent_synapses, frontend()->model->input_spiking_neurons->max_num_efferent_synapses);
      CudaSafeCall(hipMemcpy(
        num_active_synapses,
        &max_efferents,
        sizeof(int), hipMemcpyHostToDevice));
    }

    void SpikingSynapses::state_update
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {

      if (frontend()->total_number_of_synapses > 0){
      
      // Calculate buffer location
      int bufferloc = (int)(std::round(current_time_in_seconds / timestep)) % buffersize;


      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);
      ::Backend::CUDA::SpikingNeurons* input_neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(input_neurons->backend());
      assert(input_neurons_backend);

      activate_synapses<<<neurons_backend->number_of_neuron_blocks_per_grid, threads_per_block>>>(
          d_synaptic_data,
          neurons_backend->d_neuron_data,
          input_neurons_backend->d_neuron_data,
          bufferloc,
          timestep,
          current_time_in_seconds,
          ((int)roundf(current_time_in_seconds / timestep)),
          frontend()->model->timestep_grouping);
      CudaCheckError();
      }
      
    }
      
    __device__ void get_active_synapses(
      spiking_synapses_data_struct* synaptic_data,
      spiking_neurons_data_struct* neuron_data,
      int timestep_group_index,
      int preneuron_idx,
      int timestep_index,
      bool is_input)
    {
      int pos = atomicAdd(&synaptic_data->num_activated_neurons[timestep_index % 2], 1);
      int synapse_count = neuron_data->per_neuron_efferent_synapse_count[preneuron_idx];
      int synapse_start = neuron_data->per_neuron_efferent_synapse_start[preneuron_idx];
      synaptic_data->active_synapse_counts[pos] = synapse_count;
      synaptic_data->active_synapse_starts[pos] = synapse_start;
      synaptic_data->group_indices[pos] = timestep_group_index;
    };
      

    __global__ void activate_synapses(
        spiking_synapses_data_struct* synaptic_data,
        spiking_neurons_data_struct* neurons_data,
        spiking_neurons_data_struct* in_neurons_data,
        int bufferloc,
        float timestep,
        float current_time_in_seconds,
        int timestep_index,
        int timestep_grouping)
    {
      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      if (indx == 0){
        synaptic_data->num_activated_neurons[((timestep_index / timestep_grouping) + 1) % 2] = 0;
      }
      while (indx < (synaptic_data->num_active_synapses[0]*synaptic_data->num_activated_neurons[((timestep_index / timestep_grouping) % 2)])) {
 
        int pos = indx / synaptic_data->num_active_synapses[0]; 
        int idx = indx % synaptic_data->num_active_synapses[0]; 
        int synapse_count = synaptic_data->active_synapse_counts[pos];

        if (idx >= synapse_count){
          indx += blockDim.x * gridDim.x;
          continue;
        }

        int synapse_id = synaptic_data->active_synapse_starts[pos] + idx;
        int postneuron = synaptic_data->postsynaptic_neuron_indices[synapse_id];
        
        int targetloc = (bufferloc + synaptic_data->delays[synapse_id] + synaptic_data->group_indices[pos]) % synaptic_data->neuron_inputs.temporal_buffersize;
        int syn_label = synaptic_data->syn_labels[synapse_id];
        float weightinput = synaptic_data->synaptic_efficacies_or_weights[synapse_id]*synaptic_data->weight_scaling_constants[synapse_id];
        atomicAdd(&synaptic_data->neuron_inputs.circular_input_buffer[targetloc*synaptic_data->neuron_inputs.input_buffersize + syn_label + postneuron*synaptic_data->num_syn_labels], weightinput);
        indx += blockDim.x * gridDim.x;
      }
    }

      __device__ float spiking_current_injection_kernel(
        spiking_synapses_data_struct* synaptic_data,
        spiking_neurons_data_struct* neuron_data,
        float current_membrane_voltage,
        float current_time_in_seconds,
        float timestep,
        float multiplication_to_volts,
        int idx,
        int g){
        return 0.0f;
      };

  }
}
