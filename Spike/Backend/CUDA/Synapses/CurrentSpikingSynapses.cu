#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/CurrentSpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, CurrentSpikingSynapses);

namespace Backend {
  namespace CUDA {
    __device__ injection_kernel current_device_kernel = current_spiking_current_injection_kernel;
    
    CurrentSpikingSynapses::~CurrentSpikingSynapses(){
      CudaSafeCall(hipFree(neuron_wise_current_trace));
      CudaSafeCall(hipFree(d_decay_terms_tau));
      free(h_neuron_wise_current_trace);
    }
    void CurrentSpikingSynapses::prepare() {
      SpikingSynapses::prepare();

      // Carry out remaining device actions
      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();

      current_spiking_synapses_data_struct temp_synaptic_data;
      memcpy(&temp_synaptic_data, synaptic_data, sizeof(spiking_synapses_data_struct));
      free(synaptic_data);
      synaptic_data = new current_spiking_synapses_data_struct();
      memcpy(synaptic_data, &temp_synaptic_data, sizeof(spiking_synapses_data_struct));
      current_spiking_synapses_data_struct* this_synaptic_data = static_cast<current_spiking_synapses_data_struct*>(synaptic_data);
      this_synaptic_data->neuron_wise_current_trace = neuron_wise_current_trace;
      CudaSafeCall(hipMemcpy(
        d_synaptic_data,
        synaptic_data,
        sizeof(current_spiking_synapses_data_struct), hipMemcpyHostToDevice));
    }
    
    void CurrentSpikingSynapses::allocate_device_pointers() {
      // Set up per neuron current
      current_array_length = frontend()->neuron_pop_size*frontend()->num_syn_labels;
      h_neuron_wise_current_trace = (float*)realloc(h_neuron_wise_current_trace, current_array_length*sizeof(float));
      for (int id = 0; id < current_array_length; id++)
        h_neuron_wise_current_trace[id] = 0.0f;

      CudaSafeCall(hipMalloc((void **)&d_decay_terms_tau, sizeof(float)*frontend()->num_syn_labels));
      CudaSafeCall(hipFree(d_synaptic_data));
      CudaSafeCall(hipMalloc((void **)&d_synaptic_data, sizeof(current_spiking_synapses_data_struct)));
      CudaSafeCall(hipMemcpyFromSymbol(
            &host_injection_kernel,
            HIP_SYMBOL(current_device_kernel),
            sizeof(injection_kernel)));
    }
    
    void CurrentSpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
      CudaSafeCall(hipMemcpy(
        d_decay_terms_tau,
        &(frontend()->decay_terms_tau[0]),
        sizeof(float)*frontend()->num_syn_labels, hipMemcpyHostToDevice));
    }

    void CurrentSpikingSynapses::reset_state() {
      SpikingSynapses::reset_state();
      CudaSafeCall(hipMemcpy(
        neuron_wise_current_trace,
        h_neuron_wise_current_trace,
        sizeof(float)*current_array_length, hipMemcpyHostToDevice));
    }

    void CurrentSpikingSynapses::state_update(::SpikingNeurons * neurons, ::SpikingNeurons* input_neurons, float current_time_in_seconds, float timestep) {
    }
    
    /* KERNELS BELOW */
    __device__ float current_spiking_current_injection_kernel(
        spiking_synapses_data_struct* in_synaptic_data,
	      spiking_neurons_data_struct* neuron_data,
        float multiplication_to_volts,
        float current_membrane_voltage,
        float current_time_in_seconds,
        float timestep,
        int timestep_grouping,
	      int idx,
	      int g){
      
      current_spiking_synapses_data_struct* synaptic_data = (current_spiking_synapses_data_struct*) in_synaptic_data;
        
	    int total_number_of_neurons =  neuron_data->total_number_of_neurons;
      float total_current = 0.0f;
        for (int syn_label = 0; syn_label < synaptic_data->num_syn_labels; syn_label++){
          float decay_term_value = synaptic_data->decay_terms_tau[syn_label];
	        float decay_factor = expf(- timestep / decay_term_value);
          float synaptic_current = synaptic_data->neuron_wise_current_trace[total_number_of_neurons*syn_label + idx];
          // Update the synaptic conductance
	        synaptic_current *= decay_factor;
          int bufferloc = ((synaptic_data->neuron_inputs.bufferloc[0] + g) % synaptic_data->neuron_inputs.temporal_buffersize)*synaptic_data->neuron_inputs.input_buffersize;
          synaptic_current += synaptic_data->neuron_inputs.circular_input_buffer[bufferloc + syn_label*total_number_of_neurons + idx];
	        // Reset the conductance update
          synaptic_data->neuron_inputs.circular_input_buffer[bufferloc + syn_label*total_number_of_neurons + idx] = 0.0f;
          total_current += synaptic_current;
          synaptic_data->neuron_wise_current_trace[total_number_of_neurons*syn_label + idx] = synaptic_current;

	      }
	      
        return total_current*multiplication_to_volts;
    }

  }
}
  
