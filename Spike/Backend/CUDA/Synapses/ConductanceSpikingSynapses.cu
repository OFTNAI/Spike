#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/ConductanceSpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, ConductanceSpikingSynapses);

namespace Backend {
  namespace CUDA {
    // ConductanceSpikingSynapses Destructor
    ConductanceSpikingSynapses::~ConductanceSpikingSynapses() {
      CudaSafeCall(hipFree(synaptic_conductances_g));
      CudaSafeCall(hipFree(biological_conductance_scaling_constants_lambda));
      CudaSafeCall(hipFree(reversal_potentials_Vhat));
      CudaSafeCall(hipFree(decay_terms_tau_g));
      CudaSafeCall(hipFree(num_active_synapses));
      CudaSafeCall(hipHostFree(h_num_active_synapses));
      CudaSafeCall(hipFree(active_synapse_indices));
      CudaSafeCall(hipFree(num_after_deactivation));
      CudaSafeCall(hipFree(synapse_switches));
    }

    void ConductanceSpikingSynapses::prepare() {
      SpikingSynapses::prepare();
      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();
    }

    void ConductanceSpikingSynapses::reset_state() {
      SpikingSynapses::reset_state();
      CudaSafeCall(hipMemcpy(synaptic_conductances_g,
                              frontend()->synaptic_conductances_g,
                              sizeof(float)*frontend()->total_number_of_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
    }


    void ConductanceSpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&biological_conductance_scaling_constants_lambda, sizeof(float)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&reversal_potentials_Vhat, sizeof(float)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&decay_terms_tau_g, sizeof(float)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&synaptic_conductances_g, sizeof(float)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&active_synapse_indices, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&num_active_synapses, sizeof(int)));
      CudaSafeCall(hipHostMalloc((void **)&h_num_active_synapses, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&num_after_deactivation, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&synapse_switches, sizeof(int)*frontend()->total_number_of_synapses));
    }

    void ConductanceSpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
      CudaSafeCall(hipMemcpy(biological_conductance_scaling_constants_lambda,
        frontend()->biological_conductance_scaling_constants_lambda,
        sizeof(float)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        reversal_potentials_Vhat,
        frontend()->reversal_potentials_Vhat,
        sizeof(float)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        decay_terms_tau_g,
        frontend()->decay_terms_tau_g,
        sizeof(float)*frontend()->total_number_of_synapses,
        hipMemcpyHostToDevice));
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
    }


    void ConductanceSpikingSynapses::calculate_postsynaptic_current_injection(::SpikingNeurons * neurons, float current_time_in_seconds, float timestep) {

      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);

      conductance_calculate_postsynaptic_current_injection_kernel<<<active_syn_blocks_per_grid, threads_per_block>>>(
        presynaptic_neuron_indices,
        postsynaptic_neuron_indices,
        reversal_potentials_Vhat,
        neurons_backend->current_injections,
        num_active_synapses,
        active_synapse_indices,
        neurons_backend->membrane_potentials_v, 
        synaptic_conductances_g);

      CudaCheckError();
    }

    void ConductanceSpikingSynapses::update_synaptic_conductances(float timestep, float current_time_in_seconds) {
      conductance_update_synaptic_conductances_kernel<<<active_syn_blocks_per_grid, threads_per_block>>>(
        timestep, 
        synaptic_conductances_g, 
        synaptic_efficacies_or_weights, 
        time_of_last_spike_to_reach_synapse,
        biological_conductance_scaling_constants_lambda,
        num_active_synapses,
        active_synapse_indices,
        current_time_in_seconds,
        decay_terms_tau_g);

      CudaCheckError();
    }

    void ConductanceSpikingSynapses::interact_spikes_with_synapses
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {

      // Setting up access to neuron backends
      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);
      ::Backend::CUDA::SpikingNeurons* input_neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(input_neurons->backend());
      assert(input_neurons_backend);
      
      std::vector<::Backend::CUDA::SpikingNeurons*> neuron_backends_vec;
      neuron_backends_vec.push_back(neurons_backend);
      neuron_backends_vec.push_back(input_neurons_backend);
      // Get Active Synapses
      for (int neuron_pop = 0; neuron_pop < neuron_backends_vec.size(); neuron_pop++){      
        get_active_synapses_kernel<<<neuron_backends_vec[neuron_pop]->number_of_neuron_blocks_per_grid, threads_per_block>>>(
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_count,
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_total,
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_indices,
                delays,
                spikes_travelling_to_synapse,
                neuron_backends_vec[neuron_pop]->last_spike_time_of_each_neuron,
                decay_terms_tau_g,
                current_time_in_seconds,
                num_active_synapses,
                active_synapse_indices,
                num_after_deactivation,
                synapse_switches,
                timestep,
                neuron_backends_vec[neuron_pop]->frontend()->total_number_of_neurons);
        CudaCheckError();
      }

      // Setting up the custom block size for active synapses only:
      // Carry out the update every 10 timesteps. This timescale affects speed of the kernels not which syns
      if (fmod(current_time_in_seconds, 100.0*timestep) < timestep){
        // Copying to a pinned memory location (h_num_active_synapses) is much faster
        CudaSafeCall(hipMemcpy(h_num_active_synapses, num_active_synapses, sizeof(int), hipMemcpyDeviceToHost));
        active_syn_blocks_per_grid = dim3((h_num_active_synapses[0] + threads_per_block.x) /  threads_per_block.x);
        // Ensure we do not exceed the maximum number of efficient blocks
        if (active_syn_blocks_per_grid.x > number_of_synapse_blocks_per_grid.x)
          active_syn_blocks_per_grid = number_of_synapse_blocks_per_grid;
      }

      // Option for high fidelity. Ensures that a synapse can support multiple spikes
      if (neurons_backend->frontend()->high_fidelity_spike_flag){
        conductance_check_bitarray_for_presynaptic_neuron_spikes<<<active_syn_blocks_per_grid, threads_per_block>>>(
                  presynaptic_neuron_indices,
                  delays,
                  neurons_backend->bitarray_of_neuron_spikes,
                  input_neurons_backend->bitarray_of_neuron_spikes,
                  neurons_backend->frontend()->bitarray_length,
                  neurons_backend->frontend()->bitarray_maximum_axonal_delay_in_timesteps,
                  current_time_in_seconds,
                  timestep,
                  num_active_synapses,
                  active_synapse_indices,
                  time_of_last_spike_to_reach_synapse);
        CudaCheckError();
      } else {
        //CudaSafeCall(hipMemcpy(num_after_deactivation, num_active_synapses, sizeof(int), hipMemcpyDeviceToDevice));
        CudaSafeCall(hipMemcpy(h_num_active_synapses, num_after_deactivation, sizeof(int), hipMemcpyDeviceToHost));
        if (h_num_active_synapses[0] < 0)
         CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
        conductance_move_spikes_towards_synapses_kernel<<<active_syn_blocks_per_grid, threads_per_block>>>(
                  presynaptic_neuron_indices,
                  delays,
                  spikes_travelling_to_synapse,
                  decay_terms_tau_g,
                  current_time_in_seconds,
                  num_active_synapses,
                  active_synapse_indices,
                  num_after_deactivation,
                  synapse_switches,
                  time_of_last_spike_to_reach_synapse,
                  timestep);
        CudaCheckError();
        //CudaSafeCall(hipMemcpy(num_active_synapses, num_after_deactivation, sizeof(int), hipMemcpyDeviceToDevice));
        //CudaSafeCall(hipMemcpy(active_synapse_indices, synapse_switches, sizeof(int)*frontend()->total_number_of_synapses, hipMemcpyDeviceToDevice));
      }
    }


    /* KERNELS BELOW */
    __global__ void get_active_synapses_kernel(int* d_per_neuron_efferent_synapse_count,
        int* d_per_neuron_efferent_synapse_total,
                int* d_per_neuron_efferent_synapse_indices,
                int* d_delays,
                int* d_spikes_travelling_to_synapse,
                float* d_last_spike_time_of_each_neuron,
                float * d_decay_terms_tau_g,
                float current_time_in_seconds,
                int* d_num_active_synapses,
                int* d_active_synapses,
                int* num_after_deactivation,
                int* synapse_switches,
                float timestep,
                size_t total_number_of_neurons) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
    
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(idx);
        float effecttime = d_last_spike_time_of_each_neuron[CORRECTED_PRESYNAPTIC_ID(idx, presynaptic_is_input)];

        // Check if spike occurred within the last timestep    
        if (fabs(effecttime - current_time_in_seconds) < 0.5*timestep){
          // For each of this neuron's efferent synapses
          for (int i = 0; i < d_per_neuron_efferent_synapse_count[idx]; i++){
            int synapse_id = d_per_neuron_efferent_synapse_indices[d_per_neuron_efferent_synapse_total[idx] - i - 1];
            // If this synapse is not active, make it active
            if (d_spikes_travelling_to_synapse[synapse_id] == 0) {
              int pos = atomicAdd(&num_after_deactivation[0], -1);
              pos -= 1;
              if (pos >= 0){
                d_active_synapses[synapse_switches[pos]] = synapse_id;
                d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id];
              } else {
                // SET SOME FLAG TO TRUE!
                pos = atomicAdd(&d_num_active_synapses[0], 1);
                d_active_synapses[pos] = synapse_id;  
                d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id];
              }
            } else if (d_spikes_travelling_to_synapse[synapse_id] < 0) {
              // TO BE REPLACED WITH DECAYING TRACE
              // If the synapses is active and the spike has already reached the post-syn, reset the delay for next spike
              d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id];
            }
          }
        }

        __syncthreads();
        idx += blockDim.x * gridDim.x;
      }
    }

    __global__ void conductance_calculate_postsynaptic_current_injection_kernel(int * d_presynaptic_neuron_indices,
                  int* d_postsynaptic_neuron_indices,
                  float* d_reversal_potentials_Vhat,
                  float* d_neurons_current_injections,
                  int* d_num_active_synapses,
                  int* d_active_synapses,
                  float * d_membrane_potentials_v,
                  float * d_synaptic_conductances_g){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];
        if (idx < 0){
          indx += blockDim.x * gridDim.x;
          continue;
        }

        float reversal_potential_Vhat = d_reversal_potentials_Vhat[idx];
        int postsynaptic_neuron_index = d_postsynaptic_neuron_indices[idx];
        float membrane_potential_v = d_membrane_potentials_v[postsynaptic_neuron_index];
        float synaptic_conductance_g = d_synaptic_conductances_g[idx];

        float component_for_sum = synaptic_conductance_g * (reversal_potential_Vhat - membrane_potential_v);
        if (component_for_sum != 0.0) {
          atomicAdd(&d_neurons_current_injections[postsynaptic_neuron_index], component_for_sum);
        }

        __syncthreads();
        indx += blockDim.x * gridDim.x;


      }
    }

    __global__ void conductance_update_synaptic_conductances_kernel(float timestep,
                                float * d_synaptic_conductances_g,
                                float * d_synaptic_efficacies_or_weights,
                                float * d_time_of_last_spike_to_reach_synapse,
                                float * d_biological_conductance_scaling_constants_lambda,
                                int* d_num_active_synapses,
                                int* d_active_synapses,
                                float current_time_in_seconds,
                                float * d_decay_terms_tau_g) {

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];
        if (idx < 0){
          indx += blockDim.x * gridDim.x;
          continue;
        }

        float synaptic_conductance_g = d_synaptic_conductances_g[idx];

        float new_conductance = (1.0 - (timestep/d_decay_terms_tau_g[idx])) * synaptic_conductance_g;

        if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
          float synaptic_efficacy = d_synaptic_efficacies_or_weights[idx];
          float biological_conductance_scaling_constant_lambda = d_biological_conductance_scaling_constants_lambda[idx];
          float synaptic_efficacy_times_scaling_constant = synaptic_efficacy * biological_conductance_scaling_constant_lambda;
          new_conductance += synaptic_efficacy_times_scaling_constant;
        }

        d_synaptic_conductances_g[idx] = new_conductance;

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();

    }

    __global__ void conductance_move_spikes_towards_synapses_kernel(int* d_presynaptic_neuron_indices,
                    int* d_delays,
                    int* d_spikes_travelling_to_synapse,
                    float* d_decay_terms_tau_g,
                    float current_time_in_seconds,
                    int* d_num_active_synapses,
                    int* d_active_synapses,
                    int* num_after_deactivation,
                    int* synapse_switches,
                    float* d_time_of_last_spike_to_reach_synapse,
            float timestep){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];
        if (idx < 0){
          indx += blockDim.x * gridDim.x;
          continue;
        }
        //synapse_switches[indx] = d_active_synapses[indx];
        int timesteps_until_spike_reaches_synapse = d_spikes_travelling_to_synapse[idx];

        // If the spike is about to reach the synapse, set the spike time to next timestep
        if (timesteps_until_spike_reaches_synapse == 1){
              d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds + timestep;
          timesteps_until_spike_reaches_synapse = -1;
        }
        // If the spike is anything but zero (reset value) decrement it
        if (timesteps_until_spike_reaches_synapse != 0) 
          timesteps_until_spike_reaches_synapse -= 1;
        // Given a spike injection window, check if the synapse is within it. If not, remove the synapse from active
        if (-timesteps_until_spike_reaches_synapse*timestep > 10.0f*d_decay_terms_tau_g[idx]){
          int pos = atomicAdd(&num_after_deactivation[0], 1);
          synapse_switches[pos] = indx;
          d_active_synapses[indx] = -1;
          timesteps_until_spike_reaches_synapse = 0;
//          while (true){
//            int pos = atomicAdd(&num_after_deactivation[0], -1);
//            if (pos <= 0){
//              atomicAdd(&num_after_deactivation[0], 1);
//              break;
//            }
//            // Check if the position for the index swap is appropriate
//            if (((pos - 1) <= indx))
//              break;
//            int pos_timestep = d_spikes_travelling_to_synapse[d_active_synapses[pos - 1]];
//            if (-pos_timestep*timestep < 10.0f*d_decay_terms_tau_g[d_active_synapses[pos -1]]){
//              synapse_switches[indx] = d_active_synapses[pos - 1]; 
//              break;
//            }
//          }
          //atomicExch(&d_active_synapses[indx], d_active_synapses[atomicAdd(&d_num_active_synapses[0], -1) - 1]);;
          //d_active_synapses[indx] = d_active_synapses[pos - 1];
        }

        d_spikes_travelling_to_synapse[idx] = timesteps_until_spike_reaches_synapse;

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }

    __global__ void conductance_check_bitarray_for_presynaptic_neuron_spikes(int* d_presynaptic_neuron_indices,
                    int* d_delays,
                    unsigned char* d_bitarray_of_neuron_spikes,
                    unsigned char* d_input_neuron_bitarray_of_neuron_spikes,
                    int bitarray_length,
                    int bitarray_maximum_axonal_delay_in_timesteps,
                    float current_time_in_seconds,
                    float timestep,
                    int * d_num_active_synapses,
                    int * d_active_synapses,
                    float* d_time_of_last_spike_to_reach_synapse){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];

        int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
        int delay = d_delays[idx];

        // Get offset depending upon the current timestep
        int offset_index = ((int)(round(current_time_in_seconds / timestep)) % bitarray_maximum_axonal_delay_in_timesteps) - delay;
        offset_index = (offset_index < 0) ? (offset_index + bitarray_maximum_axonal_delay_in_timesteps) : offset_index;
        int offset_byte = offset_index / 8;
        int offset_bit_pos = offset_index - (8 * offset_byte);

        // Get the correct neuron index
        int neuron_index = CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input);

        // Check the spike
        int neuron_id_spike_store_start = neuron_index * bitarray_length;
        int check = 0;
        if (presynaptic_is_input){
          unsigned char byte = d_input_neuron_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
          check = ((byte >> offset_bit_pos) & 1);
          if (check == 1){
            d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
          }
        } else {
          unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
          check = ((byte >> offset_bit_pos) & 1);
          if (check == 1){
            d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
          }
        }

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
