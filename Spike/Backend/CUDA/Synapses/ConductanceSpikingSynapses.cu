#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/ConductanceSpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, ConductanceSpikingSynapses);

namespace Backend {
  namespace CUDA {
    // ConductanceSpikingSynapses Destructor
    ConductanceSpikingSynapses::~ConductanceSpikingSynapses() {
      CudaSafeCall(hipFree(biological_conductance_scaling_constants_lambda));
      CudaSafeCall(hipFree(num_active_synapses));
      CudaSafeCall(hipHostFree(h_num_active_synapses));
      CudaSafeCall(hipFree(active_synapse_indices));
      CudaSafeCall(hipFree(num_after_deactivation));
      CudaSafeCall(hipFree(synapse_switches));
      CudaSafeCall(hipFree(synapse_decay_id));
      CudaSafeCall(hipFree(neuron_wise_conductance_trace));
      CudaSafeCall(hipFree(decay_term_values));
      free(h_decay_term_values);
      free(h_synapse_decay_id);
      free(h_neuron_wise_conductance_trace);
    }

    void ConductanceSpikingSynapses::prepare() {
      SpikingSynapses::prepare();
      // Set up tau values and ids (Host-Side)
      h_synapse_decay_id = (int*)realloc(h_synapse_decay_id, frontend()->total_number_of_synapses*sizeof(int));
      // Prepare the tau synaptic conductance host-side vars
      for (int syn_id = 0; syn_id < frontend()->total_number_of_synapses; syn_id++){
        float tau_g = frontend()->decay_terms_tau_g[syn_id];
        float reversal_pot = frontend()->reversal_potentials_Vhat[syn_id];
        int id = -1;
        // Get Tau ID
        for (int i = 0; i < num_decay_terms; i++){
          // If this combination exists already, find it and assign the correct id
          if ((tau_g == h_decay_term_values[i]) && (reversal_pot == h_reversal_values[i]))
            id = i;       
        }
        if (id < 0){
          num_decay_terms += 1;
          h_decay_term_values = (float*)realloc(h_decay_term_values, (num_decay_terms)*sizeof(float));
          h_reversal_values = (float*)realloc(h_reversal_values, (num_decay_terms)*sizeof(float));
          id = num_decay_terms - 1;
          h_decay_term_values[id] = tau_g;
          h_reversal_values[id] = reversal_pot;
        } 
        // Set this tau id
        h_synapse_decay_id[syn_id] = id;
      }
      // Set up per neuron conductances
      conductance_trace_length = frontend()->neuron_pop_size*num_decay_terms;
      h_neuron_wise_conductance_trace = (float*)realloc(h_neuron_wise_conductance_trace, conductance_trace_length*sizeof(float));
      for (int id = 0; id < conductance_trace_length; id++)
        h_neuron_wise_conductance_trace[id] = 0.0f;
      // Set up block size for conductances
      // conductance_trace_blocks_per_grid = dim3((conductance_trace_length + threads_per_block.x) / threads_per_block.x);
      // if (conductance_trace_blocks_per_grid.x > number_of_synapse_blocks_per_grid.x)
      //   conductance_trace_blocks_per_grid = number_of_synapse_blocks_per_grid;

      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();
    }

    void ConductanceSpikingSynapses::reset_state() {
      SpikingSynapses::reset_state();
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
      CudaSafeCall(hipMemcpy(
        neuron_wise_conductance_trace,
        h_neuron_wise_conductance_trace,
        sizeof(float)*conductance_trace_length, hipMemcpyHostToDevice));
    }


    void ConductanceSpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&biological_conductance_scaling_constants_lambda, sizeof(float)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&active_synapse_indices, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&num_active_synapses, sizeof(int)));
      CudaSafeCall(hipHostMalloc((void **)&h_num_active_synapses, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&num_after_deactivation, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&synapse_switches, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&synapse_decay_id, sizeof(int)*frontend()->total_number_of_synapses));
      CudaSafeCall(hipMalloc((void **)&neuron_wise_conductance_trace, sizeof(float)*conductance_trace_length));
      CudaSafeCall(hipMalloc((void **)&decay_term_values, sizeof(float)*num_decay_terms));
      CudaSafeCall(hipMalloc((void **)&reversal_values, sizeof(float)*num_decay_terms));
    }

    void ConductanceSpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
      CudaSafeCall(hipMemcpy(biological_conductance_scaling_constants_lambda,
        frontend()->biological_conductance_scaling_constants_lambda,
        sizeof(float)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        synapse_decay_id,
        h_synapse_decay_id,
        sizeof(int)*frontend()->total_number_of_synapses, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        neuron_wise_conductance_trace,
        h_neuron_wise_conductance_trace,
        sizeof(float)*conductance_trace_length, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        decay_term_values,
        h_decay_term_values,
        sizeof(float)*num_decay_terms, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(
        reversal_values,
        h_reversal_values,
        sizeof(float)*num_decay_terms, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemset(num_active_synapses, 0, sizeof(int)));
      CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
    }


    void ConductanceSpikingSynapses::calculate_postsynaptic_current_injection(::SpikingNeurons * neurons, float current_time_in_seconds, float timestep) {

      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);

      conductance_calculate_postsynaptic_current_injection_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, threads_per_block>>>(
        decay_term_values,
        reversal_values,
        num_decay_terms,
        synapse_decay_id,
        neuron_wise_conductance_trace,
        neurons_backend->current_injections,
        num_active_synapses,
        active_synapse_indices,
        neurons_backend->membrane_potentials_v, 
        timestep,
        neurons_backend->frontend()->total_number_of_neurons);

      CudaCheckError();
    }

    void ConductanceSpikingSynapses::update_synaptic_conductances(float timestep, float current_time_in_seconds) {
      conductance_update_synaptic_conductances_kernel<<<active_syn_blocks_per_grid, threads_per_block>>>(
        postsynaptic_neuron_indices,
        neuron_wise_conductance_trace,
        synapse_decay_id,
        frontend()->neuron_pop_size,
        synaptic_efficacies_or_weights, 
        time_of_last_spike_to_reach_synapse,
        biological_conductance_scaling_constants_lambda,
        num_active_synapses,
        active_synapse_indices,
        current_time_in_seconds);

      CudaCheckError();
    }

    void ConductanceSpikingSynapses::interact_spikes_with_synapses
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {

      // Setting up access to neuron backends
      ::Backend::CUDA::SpikingNeurons* neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(neurons->backend());
      assert(neurons_backend);
      ::Backend::CUDA::SpikingNeurons* input_neurons_backend =
        dynamic_cast<::Backend::CUDA::SpikingNeurons*>(input_neurons->backend());
      assert(input_neurons_backend);
      
      std::vector<::Backend::CUDA::SpikingNeurons*> neuron_backends_vec;
      neuron_backends_vec.push_back(neurons_backend);
      neuron_backends_vec.push_back(input_neurons_backend);
      // Get Active Synapses
      for (int neuron_pop = 0; neuron_pop < neuron_backends_vec.size(); neuron_pop++){      
        get_active_synapses_kernel<<<neuron_backends_vec[neuron_pop]->number_of_neuron_blocks_per_grid, threads_per_block>>>(
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_count,
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_total,
                neuron_backends_vec[neuron_pop]->per_neuron_efferent_synapse_indices,
                delays,
                spikes_travelling_to_synapse,
                neuron_backends_vec[neuron_pop]->last_spike_time_of_each_neuron,
                current_time_in_seconds,
                num_active_synapses,
                active_synapse_indices,
                num_after_deactivation,
                synapse_switches,
                timestep,
                neuron_backends_vec[neuron_pop]->frontend()->total_number_of_neurons);
        CudaCheckError();
      }

      // Setting up the custom block size for active synapses only:
      // Carry out the update every 10 timesteps. This timescale affects speed of the kernels not which syns
      if (fmod(current_time_in_seconds, 100.0*timestep) < timestep){
        // Copying to a pinned memory location (h_num_active_synapses) is much faster
        CudaSafeCall(hipMemcpy(h_num_active_synapses, num_active_synapses, sizeof(int), hipMemcpyDeviceToHost));
        active_syn_blocks_per_grid = dim3((h_num_active_synapses[0] + threads_per_block.x) /  threads_per_block.x);
        // Ensure we do not exceed the maximum number of efficient blocks
        if (active_syn_blocks_per_grid.x > number_of_synapse_blocks_per_grid.x)
          active_syn_blocks_per_grid = number_of_synapse_blocks_per_grid;
      }

      // Option for high fidelity. Ensures that a synapse can support multiple spikes
      if (neurons_backend->frontend()->high_fidelity_spike_flag){
        conductance_check_bitarray_for_presynaptic_neuron_spikes<<<active_syn_blocks_per_grid, threads_per_block>>>(
                  presynaptic_neuron_indices,
                  delays,
                  neurons_backend->bitarray_of_neuron_spikes,
                  input_neurons_backend->bitarray_of_neuron_spikes,
                  neurons_backend->frontend()->bitarray_length,
                  neurons_backend->frontend()->bitarray_maximum_axonal_delay_in_timesteps,
                  current_time_in_seconds,
                  timestep,
                  num_active_synapses,
                  active_synapse_indices,
                  time_of_last_spike_to_reach_synapse);
        CudaCheckError();
      } else {
        //CudaSafeCall(hipMemcpy(num_after_deactivation, num_active_synapses, sizeof(int), hipMemcpyDeviceToDevice));
        CudaSafeCall(hipMemcpy(h_num_active_synapses, num_after_deactivation, sizeof(int), hipMemcpyDeviceToHost));
        if (h_num_active_synapses[0] < 0)
         CudaSafeCall(hipMemset(num_after_deactivation, 0, sizeof(int)));
        conductance_move_spikes_towards_synapses_kernel<<<active_syn_blocks_per_grid, threads_per_block>>>(
                  spikes_travelling_to_synapse,
                  current_time_in_seconds,
                  num_active_synapses,
                  active_synapse_indices,
                  num_after_deactivation,
                  synapse_switches,
                  time_of_last_spike_to_reach_synapse,
                  timestep);
        //CudaSafeCall(hipMemcpy(num_active_synapses, num_after_deactivation, sizeof(int), hipMemcpyDeviceToDevice));
        //CudaSafeCall(hipMemcpy(active_synapse_indices, synapse_switches, sizeof(int)*frontend()->total_number_of_synapses, hipMemcpyDeviceToDevice));
      }
    }


    /* KERNELS BELOW */
    __global__ void get_active_synapses_kernel(int* d_per_neuron_efferent_synapse_count,
        int* d_per_neuron_efferent_synapse_total,
                int* d_per_neuron_efferent_synapse_indices,
                int* d_delays,
                int* d_spikes_travelling_to_synapse,
                float* d_last_spike_time_of_each_neuron,
                float current_time_in_seconds,
                int* d_num_active_synapses,
                int* d_active_synapses,
                int* num_after_deactivation,
                int* synapse_switches,
                float timestep,
                size_t total_number_of_neurons) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
    
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(idx);
        float effecttime = d_last_spike_time_of_each_neuron[CORRECTED_PRESYNAPTIC_ID(idx, presynaptic_is_input)];

        // Check if spike occurred within the last timestep    
        if (fabs(effecttime - current_time_in_seconds) < 0.5*timestep){
          // For each of this neuron's efferent synapses
          for (int i = 0; i < d_per_neuron_efferent_synapse_count[idx]; i++){
            int synapse_id = d_per_neuron_efferent_synapse_indices[d_per_neuron_efferent_synapse_total[idx] - i - 1];
            // If this synapse is not active, make it active
            if (d_spikes_travelling_to_synapse[synapse_id] == 0) {
              int pos = atomicAdd(&num_after_deactivation[0], -1);
              pos -= 1;
              if (pos >= 0){
                d_active_synapses[synapse_switches[pos]] = synapse_id;
                d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id] + 1;
              } else {
                // SET SOME FLAG TO TRUE!
                pos = atomicAdd(&d_num_active_synapses[0], 1);
                d_active_synapses[pos] = synapse_id;  
                d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id] + 1;
              }
            } else if (d_spikes_travelling_to_synapse[synapse_id] < 0) {
              // TO BE REPLACED WITH DECAYING TRACE
              // If the synapses is active and the spike has already reached the post-syn, reset the delay for next spike
              d_spikes_travelling_to_synapse[synapse_id] = d_delays[synapse_id];
            }
          }
        }

        __syncthreads();
        idx += blockDim.x * gridDim.x;
      }
    }

    __global__ void conductance_calculate_postsynaptic_current_injection_kernel(
                  float* decay_term_values,
                  float* reversal_values,
                  int num_decay_terms,
                  int* synapse_decay_values,
                  float* neuron_wise_conductance_traces,
                  float* d_neurons_current_injections,
                  int* d_num_active_synapses,
                  int* d_active_synapses,
                  float * d_membrane_potentials_v,
                  float timestep,
                  size_t total_number_of_neurons){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        float membrane_potential_v = d_membrane_potentials_v[idx];

        for (int decay_id = 0; decay_id < num_decay_terms; decay_id++){
          float synaptic_conductance_g = neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons];
          // First decay the conductance values as required
          synaptic_conductance_g *= expf(- timestep / decay_term_values[decay_id]);
          neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons] = synaptic_conductance_g;
          d_neurons_current_injections[idx] += synaptic_conductance_g * (reversal_values[decay_id] - membrane_potential_v);
        }

        //if (d_neurons_current_injections[idx] > 0.0001)
        //printf("%f in here!\n", d_neurons_current_injections[idx]);
        idx += blockDim.x * gridDim.x;

      }
    }


    __global__ void conductance_update_synaptic_conductances_kernel(
                                int* postsynaptic_neuron_indices,
                                float * neuron_wise_conductance_trace,
                                int * synaptic_decay_id,
                                int total_number_of_neurons,
                                float * d_synaptic_efficacies_or_weights,
                                float * d_time_of_last_spike_to_reach_synapse,
                                float * d_biological_conductance_scaling_constants_lambda,
                                int* d_num_active_synapses,
                                int* d_active_synapses,
                                float current_time_in_seconds) {

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];
        if (idx < 0){
          indx += blockDim.x * gridDim.x;
          continue;
        }

        if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
          int postsynaptic_neuron_id = postsynaptic_neuron_indices[idx];
          int trace_id = synaptic_decay_id[idx];
          float synaptic_efficacy = d_biological_conductance_scaling_constants_lambda[idx] * d_synaptic_efficacies_or_weights[idx];
          atomicAdd(&neuron_wise_conductance_trace[total_number_of_neurons*trace_id + postsynaptic_neuron_id], synaptic_efficacy);
        }

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();

    }

    __global__ void conductance_move_spikes_towards_synapses_kernel(
                    int* d_spikes_travelling_to_synapse,
                    float current_time_in_seconds,
                    int* d_num_active_synapses,
                    int* d_active_synapses,
                    int* num_after_deactivation,
                    int* synapse_switches,
                    float* d_time_of_last_spike_to_reach_synapse,
                    float timestep){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];
        if (idx < 0){
          indx += blockDim.x * gridDim.x;
          continue;
        }
        //synapse_switches[indx] = d_active_synapses[indx];
        int timesteps_until_spike_reaches_synapse = d_spikes_travelling_to_synapse[idx];

        // If the spike is anything but zero (reset value) decrement it
        if (timesteps_until_spike_reaches_synapse != 0) 
          timesteps_until_spike_reaches_synapse -= 1;
        // If the spike is about to reach the synapse, set the spike time to next timestep
        if (timesteps_until_spike_reaches_synapse == 1){
            d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
        }
        // Given a spike injection window, check if the synapse is within it. If not, remove the synapse from active
        if (timesteps_until_spike_reaches_synapse == 0){
          int pos = atomicAdd(&num_after_deactivation[0], 1);
          synapse_switches[pos] = indx;
          d_active_synapses[indx] = -1;
        }

        d_spikes_travelling_to_synapse[idx] = timesteps_until_spike_reaches_synapse;

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }

    __global__ void conductance_check_bitarray_for_presynaptic_neuron_spikes(int* d_presynaptic_neuron_indices,
                    int* d_delays,
                    unsigned char* d_bitarray_of_neuron_spikes,
                    unsigned char* d_input_neuron_bitarray_of_neuron_spikes,
                    int bitarray_length,
                    int bitarray_maximum_axonal_delay_in_timesteps,
                    float current_time_in_seconds,
                    float timestep,
                    int * d_num_active_synapses,
                    int * d_active_synapses,
                    float* d_time_of_last_spike_to_reach_synapse){

      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      while (indx < d_num_active_synapses[0]) {
        int idx = d_active_synapses[indx];

        int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
        bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
        int delay = d_delays[idx];

        // Get offset depending upon the current timestep
        int offset_index = ((int)(round(current_time_in_seconds / timestep)) % bitarray_maximum_axonal_delay_in_timesteps) - delay;
        offset_index = (offset_index < 0) ? (offset_index + bitarray_maximum_axonal_delay_in_timesteps) : offset_index;
        int offset_byte = offset_index / 8;
        int offset_bit_pos = offset_index - (8 * offset_byte);

        // Get the correct neuron index
        int neuron_index = CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input);

        // Check the spike
        int neuron_id_spike_store_start = neuron_index * bitarray_length;
        int check = 0;
        if (presynaptic_is_input){
          unsigned char byte = d_input_neuron_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
          check = ((byte >> offset_bit_pos) & 1);
          if (check == 1){
            d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
          }
        } else {
          unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
          check = ((byte >> offset_bit_pos) & 1);
          if (check == 1){
            d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
          }
        }

        indx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
