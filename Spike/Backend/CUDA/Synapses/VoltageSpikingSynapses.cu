#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/VoltageSpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, VoltageSpikingSynapses);

namespace Backend {
  namespace CUDA {
    __device__ injection_kernel voltage_device_kernel = voltage_spiking_current_injection_kernel;

    // VoltageSpikingSynapses Destructor
    VoltageSpikingSynapses::~VoltageSpikingSynapses() {
    }

    void VoltageSpikingSynapses::prepare() {
      SpikingSynapses::prepare();

      synaptic_data->synapse_type = VOLTAGE;
      CudaSafeCall(hipMemcpy(
        d_synaptic_data,
        synaptic_data,
        sizeof(spiking_synapses_data_struct), hipMemcpyHostToDevice));

      allocate_device_pointers();
      copy_constants_and_initial_efficacies_to_device();
    }

    void VoltageSpikingSynapses::reset_state() {
      SpikingSynapses::reset_state();
    }


    void VoltageSpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMemcpyFromSymbol(
            &host_injection_kernel,
            HIP_SYMBOL(voltage_device_kernel),
            sizeof(injection_kernel)));
    }

    void VoltageSpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
    }



    /* STATE UPDATE */
    void VoltageSpikingSynapses::state_update
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {
      SpikingSynapses::state_update(neurons, input_neurons, current_time_in_seconds, timestep);
    }


    /* KERNELS BELOW */
    __device__ float voltage_spiking_current_injection_kernel(
        spiking_synapses_data_struct* in_synaptic_data,
        spiking_neurons_data_struct* neuron_data,
        float multiplication_to_volts,
        float current_membrane_voltage,
        float current_time_in_seconds,
        float timestep,
        int idx,
        int g){
      
      spiking_synapses_data_struct* synaptic_data = (spiking_synapses_data_struct*) in_synaptic_data;
        
      int bufferloc = (((int)roundf(current_time_in_seconds / timestep) + g) % synaptic_data->neuron_inputs.temporal_buffersize)*synaptic_data->neuron_inputs.input_buffersize;
    
      float total_current = 0.0f;
      for (int syn_label = 0; syn_label < synaptic_data->num_syn_labels; syn_label++){
        total_current += synaptic_data->neuron_inputs.circular_input_buffer[bufferloc + syn_label + idx*synaptic_data->num_syn_labels];
        
        synaptic_data->neuron_inputs.circular_input_buffer[bufferloc + syn_label + idx*synaptic_data->num_syn_labels] = 0.0f;
        
      }
    
    
      // This is already in volts, no conversion necessary
      return total_current;
    }

  }
}
