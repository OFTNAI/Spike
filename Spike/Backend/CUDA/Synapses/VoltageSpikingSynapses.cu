// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Synapses/VoltageSpikingSynapses.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, VoltageSpikingSynapses);

namespace Backend {
  namespace CUDA {
    __device__ injection_kernel voltage_device_kernel = voltage_spiking_current_injection_kernel;

    // VoltageSpikingSynapses Destructor
    VoltageSpikingSynapses::~VoltageSpikingSynapses() {
    }

    void VoltageSpikingSynapses::prepare() {
      SpikingSynapses::prepare();
    }

    void VoltageSpikingSynapses::reset_state() {
      SpikingSynapses::reset_state();
    }


    void VoltageSpikingSynapses::allocate_device_pointers() {
      CudaSafeCall(hipMemcpyFromSymbol(
            &host_injection_kernel,
            HIP_SYMBOL(voltage_device_kernel),
            sizeof(injection_kernel)));
    }

    void VoltageSpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
    }



    /* STATE UPDATE */
    void VoltageSpikingSynapses::state_update
    (::SpikingNeurons* neurons,
     ::SpikingNeurons* input_neurons,
     float current_time_in_seconds, float timestep) {
      SpikingSynapses::state_update(neurons, input_neurons, current_time_in_seconds, timestep);
    }


    /* KERNELS BELOW */
    __device__ float voltage_spiking_current_injection_kernel(
        spiking_synapses_data_struct* in_synaptic_data,
	      spiking_neurons_data_struct* neuron_data,
        float multiplication_to_volts,
        float current_membrane_voltage,
        float timestep,
        int timestep_grouping,
	      int idx,
	      int g){
      
      voltage_spiking_synapses_data_struct* synaptic_data = (voltage_spiking_synapses_data_struct*) in_synaptic_data;
        
	    int total_number_of_neurons =  neuron_data->total_number_of_neurons;
      float total_current = 0.0f;
      for (int syn_label = 0; syn_label < synaptic_data->num_syn_labels; syn_label++){
	      total_current += synaptic_data->neuron_wise_input_update[total_number_of_neurons*timestep_grouping*syn_label + g*total_number_of_neurons + idx];
	    }

      // This is already in volts, no conversion necessary
      total_current *= (1.0f / timestep);
      return total_current;
    }

  }
}
