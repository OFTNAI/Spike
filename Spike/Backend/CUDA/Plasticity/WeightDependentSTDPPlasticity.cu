#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/WeightDependentSTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, WeightDependentSTDPPlasticity);

namespace Backend {
  namespace CUDA {
    WeightDependentSTDPPlasticity::~WeightDependentSTDPPlasticity() {
      CudaSafeCall(hipFree(stdp_pre_memory_trace));
      CudaSafeCall(hipFree(stdp_post_memory_trace));
      if (h_stdp_trace)
        free(h_stdp_trace);
    }

    void WeightDependentSTDPPlasticity::reset_state() {
      STDPPlasticity::reset_state();
      
      CudaSafeCall(hipMemcpy((void*)stdp_pre_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)stdp_post_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
    }

    void WeightDependentSTDPPlasticity::prepare() {
      STDPPlasticity::prepare();
      allocate_device_pointers();
    }

    void WeightDependentSTDPPlasticity::allocate_device_pointers() {
      // The following doesn't do anything in original code...
      // ::Backend::CUDA::STDPPlasticity::allocate_device_pointers();
      CudaSafeCall(hipMalloc((void **)&stdp_pre_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      CudaSafeCall(hipMalloc((void **)&stdp_post_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      // Also allocate and set trace template
      h_stdp_trace = (float *)malloc( sizeof(float) * total_number_of_plastic_synapses);
      for (int i=0; i < total_number_of_plastic_synapses; i++){
        h_stdp_trace[i] = 0.0f;
      }
    }

    void WeightDependentSTDPPlasticity::apply_stdp_to_synapse_weights(float current_time_in_seconds, float timestep) {
        ltp_and_ltd<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
          (synapses_backend->postsynaptic_neuron_indices,
           synapses_backend->presynaptic_neuron_indices,
           synapses_backend->delays,
           neurons_backend->d_neuron_data,
           input_neurons_backend->d_neuron_data,
           synapses_backend->synaptic_efficacies_or_weights,
           stdp_pre_memory_trace,
           stdp_post_memory_trace,
           frontend()->stdp_params->tau_minus,//expf(- timestep / frontend()->stdp_params->tau_minus),
           frontend()->stdp_params->tau_plus,//expf(- timestep / frontend()->stdp_params->tau_plus),
           *(frontend()->stdp_params),
           timestep,
           frontend()->model->timestep_grouping,
           current_time_in_seconds,
           plastic_synapse_indices,
           total_number_of_plastic_synapses);
          CudaCheckError();
    }


    __global__ void ltp_and_ltd
          (int* d_postsyns,
           int* d_presyns,
           int* d_syndelays,
           spiking_neurons_data_struct* neuron_data,
           spiking_neurons_data_struct* input_neuron_data,
           float* d_synaptic_efficacies_or_weights,
           float* stdp_pre_memory_trace,
           float* stdp_post_memory_trace,
           float post_decay,
           float pre_decay,
           weightdependent_stdp_plasticity_parameters_struct stdp_vars,
           float timestep,
           int timestep_grouping,
           float current_time_in_seconds,
           int* d_plastic_synapse_indices,
           size_t total_number_of_plastic_synapses){
      // Global Index
      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      int bufsize = input_neuron_data->neuron_spike_time_bitbuffer_bytesize[0];

      // Running though all neurons
      while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];

        // Getting synapse details
        float stdp_pre_memory_trace_val = stdp_pre_memory_trace[indx];
        float stdp_post_memory_trace_val = stdp_post_memory_trace[indx];
        int postid = d_postsyns[idx];
        int preid = d_presyns[idx];
        float old_synaptic_weight = d_synaptic_efficacies_or_weights[idx];
        float new_synaptic_weight = old_synaptic_weight;

        // Correcting for input vs output neuron types
        bool is_input = PRESYNAPTIC_IS_INPUT(preid);
        int corr_preid = CORRECTED_PRESYNAPTIC_ID(preid, is_input);
        uint8_t* pre_bitbuffer = is_input ? input_neuron_data->neuron_spike_time_bitbuffer : neuron_data->neuron_spike_time_bitbuffer;
        float* pre_last_spike_times = is_input ? input_neuron_data->last_spike_time_of_each_neuron : neuron_data->last_spike_time_of_each_neuron;



        //int pre_spike_g = -1;
        int pre_spike_g = ((int)roundf((pre_last_spike_times[corr_preid] - current_time_in_seconds) / timestep));
        int post_spike_g = ((int)roundf((neuron_data->last_spike_time_of_each_neuron[postid] - current_time_in_seconds) / timestep));
        if (pre_spike_g >= timestep_grouping)
          pre_spike_g *= -1;
        /*
        for (int g=0; g < timestep_grouping; g++){
        // Looping over timesteps
        int postbitloc = ((int)roundf(current_time_in_seconds / timestep) + g) % (bufsize*8);
        int prebitloc = postbitloc - d_syndelays[idx];
        prebitloc = (prebitloc < 0) ? (bufsize*8 + prebitloc) : prebitloc;
          if (pre_bitbuffer[corr_preid*bufsize + (prebitloc / 8)] & (1 << (prebitloc % 8)))
            pre_spike_g = g;
          //if (neuron_data->neuron_spike_time_bitbuffer[postid*bufsize + (postbitloc / 8)] & (1 << (postbitloc % 8)))
            //post_spike_g = g;
        }*/

        stdp_post_memory_trace_val *= expf(-(timestep_grouping*timestep) / post_decay);
        stdp_pre_memory_trace_val *= expf(-(timestep_grouping*timestep) / pre_decay);

        // Change this if nearest only
        stdp_post_memory_trace_val += (post_spike_g >= 0) ? stdp_vars.a_minus*expf(-((timestep_grouping - post_spike_g)*timestep) / post_decay) : 0.0f;
        stdp_pre_memory_trace_val += (pre_spike_g >= 0) ? stdp_vars.a_plus*expf(-((timestep_grouping - pre_spike_g)*timestep) / pre_decay) : 0.0f;
          
        float syn_update_val = 0.0f; 
        //old_synaptic_weight = new_synaptic_weight;
        // OnPre Weight Update
        if (pre_spike_g >= 0){
          float temp_post_trace = stdp_post_memory_trace_val;
          temp_post_trace += (post_spike_g > pre_spike_g) ? -stdp_vars.a_minus*expf(-((timestep_grouping - post_spike_g)*timestep) / post_decay): 0.0f;
          temp_post_trace *= (1.0f / (expf(-(timestep_grouping - pre_spike_g)*timestep / post_decay))); 
          syn_update_val -= stdp_vars.lambda * stdp_vars.alpha * old_synaptic_weight * temp_post_trace;
        }
        // OnPost Weight Update
        if (post_spike_g >= 0){
          float temp_pre_trace = stdp_pre_memory_trace_val;
          temp_pre_trace += (pre_spike_g > post_spike_g) ? -stdp_vars.a_plus*expf(-((timestep_grouping - pre_spike_g)*timestep) / pre_decay): 0.0f;
          temp_pre_trace *= (1.0f / (expf(-(timestep_grouping - post_spike_g)*timestep / pre_decay))); 
          syn_update_val += stdp_vars.lambda * (stdp_vars.w_max - old_synaptic_weight) * temp_pre_trace;
        }

        new_synaptic_weight = old_synaptic_weight + syn_update_val;
        if (new_synaptic_weight < 0.0f)
          new_synaptic_weight = 0.0f;
        
        // Weight Update
        d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;

        // Correctly set the trace values
        stdp_pre_memory_trace[indx] = stdp_pre_memory_trace_val;
        stdp_post_memory_trace[indx] = stdp_post_memory_trace_val;

        indx += blockDim.x * gridDim.x;
      }

    }
    
  }
}
