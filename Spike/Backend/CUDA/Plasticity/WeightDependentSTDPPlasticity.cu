#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/WeightDependentSTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, WeightDependentSTDPPlasticity);

namespace Backend {
  namespace CUDA {
    WeightDependentSTDPPlasticity::~WeightDependentSTDPPlasticity() {
      CudaSafeCall(hipFree(stdp_pre_memory_trace));
      CudaSafeCall(hipFree(stdp_post_memory_trace));
      if (h_stdp_trace)
        free(h_stdp_trace);
    }

    void WeightDependentSTDPPlasticity::reset_state() {
      STDPPlasticity::reset_state();
      
      CudaSafeCall(hipMemcpy((void*)stdp_pre_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)stdp_post_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
    }

    void WeightDependentSTDPPlasticity::prepare() {
      STDPPlasticity::prepare();
      allocate_device_pointers();
    }

    void WeightDependentSTDPPlasticity::allocate_device_pointers() {
      // The following doesn't do anything in original code...
      // ::Backend::CUDA::STDPPlasticity::allocate_device_pointers();
      CudaSafeCall(hipMalloc((void **)&stdp_pre_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      CudaSafeCall(hipMalloc((void **)&stdp_post_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      // Also allocate and set trace template
      h_stdp_trace = (float *)malloc( sizeof(float) * total_number_of_plastic_synapses);
      for (int i=0; i < total_number_of_plastic_synapses; i++){
        h_stdp_trace[i] = 0.0f;
      }
    }

    void WeightDependentSTDPPlasticity::apply_stdp_to_synapse_weights(float current_time_in_seconds, float timestep) {
      if (total_number_of_plastic_synapses > 0){ 
        ltp_and_ltd<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
          (synapses_backend->postsynaptic_neuron_indices,
           synapses_backend->time_of_last_spike_to_reach_synapse,
           neurons_backend->last_spike_time_of_each_neuron,
           synapses_backend->synaptic_efficacies_or_weights,
           stdp_pre_memory_trace,
           stdp_post_memory_trace,
           *(frontend()->stdp_params),
           timestep,
           frontend()->model->timestep_grouping,
           current_time_in_seconds,
           plastic_synapse_indices,
           total_number_of_plastic_synapses);
        
      }
    }

    __global__ void ltp_and_ltd
          (int* d_postsyns,
           float* d_time_of_last_spike_to_reach_synapse,
           float* d_last_spike_time_of_each_neuron,
           float* d_synaptic_efficacies_or_weights,
           float* stdp_pre_memory_trace,
           float* stdp_post_memory_trace,
           struct weightdependent_stdp_plasticity_parameters_struct stdp_vars,
           float timestep,
           int timestep_grouping,
           float current_time_in_seconds,
           int* d_plastic_synapse_indices,
           size_t total_number_of_plastic_synapses){
      // Global Index
      int indx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all neurons
      while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];
        // First decay the memory trace (USING INDX FOR TRACE HERE AND BELOW)
        float stdp_pre_memory_trace_val = stdp_pre_memory_trace[indx];
        float stdp_post_memory_trace_val = stdp_post_memory_trace[indx];
        int postid = d_postsyns[idx];

        for (int g=0; g < timestep_grouping; g++){	
          stdp_post_memory_trace_val *= expf( - timestep / stdp_vars.tau_minus);
          stdp_pre_memory_trace_val *= expf(- timestep / stdp_vars.tau_plus);

          // First update the memory trace for every pre and post neuron
          if (fabs(d_time_of_last_spike_to_reach_synapse[idx] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
            // Update the presynaptic memory trace
            stdp_pre_memory_trace_val += stdp_vars.a_plus;
            if (stdp_vars.nearest_spike_only)
              stdp_pre_memory_trace_val = stdp_vars.a_plus;
            // Carry out the necessary LTD
            float old_synaptic_weight = d_synaptic_efficacies_or_weights[idx];
            float new_synaptic_weight = old_synaptic_weight - stdp_vars.lambda * stdp_vars.alpha * old_synaptic_weight * stdp_post_memory_trace_val;
            if (new_synaptic_weight >= 0.0f)
              d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;
           }
          // Dealing with LTP
          if (fabs(d_last_spike_time_of_each_neuron[postid] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
            stdp_post_memory_trace_val += stdp_vars.a_minus;
            if (stdp_vars.nearest_spike_only)
              stdp_post_memory_trace_val = stdp_vars.a_minus;
              // If output neuron just fired, do LTP
              d_synaptic_efficacies_or_weights[idx] += stdp_vars.lambda * (stdp_vars.w_max - d_synaptic_efficacies_or_weights[idx]) *stdp_pre_memory_trace_val;
          }
        }

        // Correctly set the trace values
        stdp_pre_memory_trace[indx] = stdp_pre_memory_trace_val;
        stdp_post_memory_trace[indx] = stdp_post_memory_trace_val;

        indx += blockDim.x * gridDim.x;
      }

    }
    
  }
}
