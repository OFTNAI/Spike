#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/WeightNormSTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, WeightNormSTDPPlasticity);

namespace Backend {
  namespace CUDA {
    WeightNormSTDPPlasticity::~WeightNormSTDPPlasticity() {    
        CudaSafeCall(hipFree(plastic_synapse_indices));
        CudaSafeCall(hipFree(sum_squared_afferent_values));
        CudaSafeCall(hipFree(neuron_in_plasticity_set));
        CudaSafeCall(hipFree(initial_weights));
        CudaSafeCall(hipFree(weight_divisor));
    }

    void WeightNormSTDPPlasticity::reset_state() {
	if (total_number_of_plastic_synapses > 0) {

		// Now load values into device memory
		CudaSafeCall(hipMemcpy((void*)initial_weights, &(frontend()->plastic_synapses[0]), sizeof(float)*total_number_of_plastic_synapses, hipMemcpyHostToDevice));

		CudaSafeCall(hipMemcpy((void*)afferent_weight_change_updater,
				(void*)frontend()->afferent_weight_change_updater,
				sizeof(float)*frontend()->neurs->total_number_of_neurons, hipMemcpyHostToDevice));
    	}
    }

    void WeightNormSTDPPlasticity::prepare() {
      
      // Set up synapses backend and synaptic details
      synapses_backend = dynamic_cast<::Backend::CUDA::Synapses*>
	(frontend()->syns->backend());
      total_number_of_plastic_synapses = frontend()->total_number_of_plastic_synapses;

      // This learning rule requires a device side storage of a number of variables

      allocate_device_pointers();
    }

    void WeightNormSTDPPlasticity::allocate_device_pointers() {
	if (total_number_of_plastic_synapses > 0){
	  CudaSafeCall(hipMalloc((void **)&plastic_synapse_indices, sizeof(int)*total_number_of_plastic_synapses));
	  CudaSafeCall(hipMemcpy((void*)plastic_synapse_indices,
				  (void*)&(frontend()->plastic_synapses[0]),
				  sizeof(int)*total_number_of_plastic_synapses,
				  hipMemcpyHostToDevice));
	  // Loading vectors from front-end
	  CudaSafeCall(hipMalloc((void **)&sum_squared_afferent_values, sizeof(float)*frontend()->neurs->total_number_of_neurons));
	  CudaSafeCall(hipMalloc((void **)&afferent_weight_change_updater, sizeof(float)*frontend()->neurs->total_number_of_neurons));
	  CudaSafeCall(hipMalloc((void **)&neuron_in_plasticity_set, sizeof(bool)*frontend()->neurs->total_number_of_neurons));
	  // Copy values
	  CudaSafeCall(hipMemcpy((void*)sum_squared_afferent_values,
				(void*)frontend()->sum_squared_afferent_values,
				sizeof(float)*frontend()->neurs->total_number_of_neurons, hipMemcpyHostToDevice));
	  CudaSafeCall(hipMemcpy((void*)afferent_weight_change_updater,
				(void*)frontend()->afferent_weight_change_updater,
				sizeof(float)*frontend()->neurs->total_number_of_neurons, hipMemcpyHostToDevice));
	  CudaSafeCall(hipMemcpy((void*)neuron_in_plasticity_set,
				(void*)frontend()->neuron_in_plasticity_set,
				sizeof(bool)*frontend()->neurs->total_number_of_neurons, hipMemcpyHostToDevice));

	  // Loading initial weights and setting weight changes to zero
	  CudaSafeCall(hipMalloc((void **)&initial_weights, sizeof(float)*total_number_of_plastic_synapses));
	  CudaSafeCall(hipMalloc((void **)&weight_divisor, sizeof(float)*frontend()->neurs->total_number_of_neurons));
	}
    }

    void WeightNormSTDPPlasticity::weight_normalization(){
	if (total_number_of_plastic_synapses > 0) {
	CudaSafeCall(hipMemcpy((void*)afferent_weight_change_updater,
			(void*)frontend()->afferent_weight_change_updater,
			sizeof(float)*frontend()->neurs->total_number_of_neurons, hipMemcpyHostToDevice));

	// First calculate the weight change
	weight_change_calculations<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>(
		synapses_backend->postsynaptic_neuron_indices,
		synapses_backend->synaptic_efficacies_or_weights,
		initial_weights,
		afferent_weight_change_updater,
		plastic_synapse_indices,
		total_number_of_plastic_synapses);
	CudaCheckError();
	weight_division_calc<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>(
		sum_squared_afferent_values,
		afferent_weight_change_updater,
		weight_divisor,
		neuron_in_plasticity_set,
		frontend()->neurs->total_number_of_neurons);
	CudaCheckError();
	weight_update<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>(
		synapses_backend->postsynaptic_neuron_indices,
		neuron_in_plasticity_set,
		synapses_backend->synaptic_efficacies_or_weights,
		weight_divisor,
		plastic_synapse_indices,
		total_number_of_plastic_synapses);
	CudaCheckError();
	}	
    }

    __global__ void weight_change_calculations(
		int* postsyn_ids,
		float* current_weight,
		float* initial_weights,
		float* afferent_weight_change_updater,
		int* d_plastic_synapse_indices,
		size_t total_number_of_plastic_synapses)
	{
		// Global Index
		int indx = threadIdx.x + blockIdx.x * blockDim.x;
		
		while (indx < total_number_of_plastic_synapses) {
			// Get the current synapse index
			int idx = d_plastic_synapse_indices[indx];
			int post_id = postsyn_ids[idx];
			float weight_change = current_weight[idx] - initial_weights[indx];
			if (weight_change != 0.0){
				float update_value = weight_change*weight_change + 2.0f*initial_weights[indx]*weight_change;
				atomicAdd(&afferent_weight_change_updater[post_id], update_value);
			}
			indx += blockDim.x * gridDim.x;
		}
		__syncthreads();
	}

	  __global__ void weight_division_calc(
		float* sum_squared_afferent_values,
		float* afferent_weight_change_updater,
		float* weight_divisor,
		bool* neuron_in_plasticity_set,
		size_t total_number_of_neurons)
	{
		// Global Index
		int idx = threadIdx.x + blockIdx.x * blockDim.x;

		while (idx < total_number_of_neurons) {
			if (neuron_in_plasticity_set[idx])
			{
				if ((sum_squared_afferent_values[idx] - afferent_weight_change_updater[idx] < 0.01))
					printf("NORMALIZATION DIFF VERY LARGE. DANGER OF SYNAPSES ALL -> ZERO");
				weight_divisor[idx] = sqrtf(sum_squared_afferent_values[idx] + afferent_weight_change_updater[idx]) / sqrtf(sum_squared_afferent_values[idx]);
			}
			idx += blockDim.x * gridDim.x;		
		}
		__syncthreads();
	}


	__global__ void weight_update(
		int* postsyn_neuron,
		bool* neuron_in_plasticity_set,
		float* current_weight,
		float* weight_divisor,
		int* d_plastic_synapse_indices,
		size_t total_number_of_plastic_synapses){
	
		// Global Index
		int indx = threadIdx.x + blockIdx.x * blockDim.x;
		
		while (indx < total_number_of_plastic_synapses) {
			int idx = d_plastic_synapse_indices[indx];
			int postneuron = postsyn_neuron[idx];
			if (neuron_in_plasticity_set[postneuron]){
				float division_value = weight_divisor[postneuron];
				//if (division_value != 1.0)
				//printf("%f, %f, %f wat \n", division_value, current_weight[idx], (current_weight[idx] / division_value));
				if (division_value != 1.0)
					current_weight[idx] /= division_value;
			}
			indx += blockDim.x * gridDim.x;
		}
	}


  }
}
