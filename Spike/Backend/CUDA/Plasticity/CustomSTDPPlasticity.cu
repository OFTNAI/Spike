#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/CustomSTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, CustomSTDPPlasticity);

namespace Backend {
  namespace CUDA {
    CustomSTDPPlasticity::~CustomSTDPPlasticity() {
      CudaSafeCall(hipFree(stdp_pre_memory_trace));
      CudaSafeCall(hipFree(stdp_post_memory_trace));
      if (h_stdp_trace)
        free(h_stdp_trace);
    }

    void CustomSTDPPlasticity::reset_state() {
      STDPPlasticity::reset_state();
      
      CudaSafeCall(hipMemcpy((void*)stdp_pre_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy((void*)stdp_post_memory_trace,
                              (void*)h_stdp_trace,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
    }

    void CustomSTDPPlasticity::prepare() {
      STDPPlasticity::prepare();
      allocate_device_pointers();
    }

    void CustomSTDPPlasticity::allocate_device_pointers() {
      // The following doesn't do anything in original code...
      // ::Backend::CUDA::STDPPlasticity::allocate_device_pointers();
      CudaSafeCall(hipMalloc((void **)&stdp_pre_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      CudaSafeCall(hipMalloc((void **)&stdp_post_memory_trace, sizeof(float)*total_number_of_plastic_synapses));
      // Also allocate and set trace template
      h_stdp_trace = (float *)malloc( sizeof(float) * total_number_of_plastic_synapses);
      for (int i=0; i < total_number_of_plastic_synapses; i++){
        h_stdp_trace[i] = 0.0f;
      }
    }

    void CustomSTDPPlasticity::apply_stdp_to_synapse_weights(float current_time_in_seconds, float timestep) {
        ltp_and_ltd<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
          (synapses_backend->postsynaptic_neuron_indices,
           synapses_backend->time_of_last_spike_to_reach_synapse,
           neurons_backend->last_spike_time_of_each_neuron,
           synapses_backend->synaptic_efficacies_or_weights,
           stdp_pre_memory_trace,
           stdp_post_memory_trace,
           *(frontend()->stdp_params),
           timestep,
           frontend()->model->timestep_grouping,
           current_time_in_seconds,
           plastic_synapse_indices,
           total_number_of_plastic_synapses);
          CudaCheckError();
    }


    __global__ void ltp_and_ltd
          (int* d_postsyns,
           float* d_time_of_last_spike_to_reach_synapse,
           float* d_last_spike_time_of_each_neuron,
           float* d_synaptic_efficacies_or_weights,
           float* stdp_pre_memory_trace,
           float* stdp_post_memory_trace,
           custom_stdp_plasticity_parameters_struct stdp_vars,
           float timestep,
           int timestep_grouping,
           float current_time_in_seconds,
           int* d_plastic_synapse_indices,
           size_t total_number_of_plastic_synapses){
      // Global Index
      int indx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all neurons
      while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];
        // First decay the memory trace (USING INDX FOR TRACE HERE AND BELOW)
        float stdp_pre_memory_trace_val = stdp_pre_memory_trace[indx];
        float stdp_post_memory_trace_val = stdp_post_memory_trace[indx];
        int postid = d_postsyns[idx];

        for (int g=0; g < timestep_grouping; g++){	
          stdp_post_memory_trace_val *= expf( - timestep / stdp_vars.tau_minus);
          stdp_pre_memory_trace_val *= expf(- timestep / stdp_vars.tau_plus);
          // First update the memory trace for every pre and post neuron
          if (fabs(d_time_of_last_spike_to_reach_synapse[idx] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
            // Update the presynaptic memory trace
            stdp_pre_memory_trace_val += stdp_vars.a_plus;
            if (stdp_vars.nearest_spike_only)
              stdp_pre_memory_trace_val = stdp_vars.a_plus;
          }
          // Dealing with LTP
          if (fabs(d_last_spike_time_of_each_neuron[postid] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
            stdp_post_memory_trace_val += stdp_vars.a_minus;
            if (stdp_vars.nearest_spike_only)
              stdp_post_memory_trace_val = stdp_vars.a_minus;
          }
          
          float syn_update_val = 0.0f; 
          float old_synaptic_weight = d_synaptic_efficacies_or_weights[idx];
          if (fabs(d_time_of_last_spike_to_reach_synapse[idx] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
            // Carry out the necessary LTD
            syn_update_val -= stdp_post_memory_trace_val;
          }
          if (fabs(d_last_spike_time_of_each_neuron[postid] - (current_time_in_seconds + g*timestep)) < 0.5f*timestep){
              // If output neuron just fired, do LTP
            syn_update_val += stdp_pre_memory_trace_val;
          }
          float new_synaptic_weight = old_synaptic_weight + syn_update_val;
          if ((new_synaptic_weight >= 0.0f) && (new_synaptic_weight <= stdp_vars.w_max)) {
              d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;
          } else if (new_synaptic_weight < 0.0f) {
              d_synaptic_efficacies_or_weights[idx] = 0.0f;
          } else {
              d_synaptic_efficacies_or_weights[idx] = stdp_vars.w_max;
          }
        }

        // Correctly set the trace values
        stdp_pre_memory_trace[indx] = stdp_pre_memory_trace_val;
        stdp_post_memory_trace[indx] = stdp_post_memory_trace_val;

        indx += blockDim.x * gridDim.x;
      }

    }
    
  }
}
