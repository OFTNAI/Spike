// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/STDPPlasticity.hpp"
#include <iostream>

// SPIKE_EXPORT_BACKEND_TYPE(CUDA, STDPPlasticity);

namespace Backend {
  namespace CUDA {
    STDPPlasticity::~STDPPlasticity() {
      CudaSafeCall(hipFree(plastic_synapse_indices));
    }

    void STDPPlasticity::prepare() {

      neurons_backend = dynamic_cast<::Backend::CUDA::SpikingNeurons*>
        (frontend()->neurs->backend());
      synapses_backend = dynamic_cast<::Backend::CUDA::SpikingSynapses*>
        (frontend()->syns->backend());

      // Get the correct ID
      int plasticity_id = frontend()->plasticity_rule_id;
      if (plasticity_id >= 0){
        total_number_of_plastic_synapses = frontend()->plastic_synapses.size();
      } else {
        total_number_of_plastic_synapses = 0;
      }

      allocate_device_pointers();
    }

    void STDPPlasticity::allocate_device_pointers(){
      if (total_number_of_plastic_synapses > 0){
        CudaSafeCall(hipMalloc((void **)&plastic_synapse_indices, sizeof(int)*total_number_of_plastic_synapses));
        CudaSafeCall(hipMemcpy((void*)plastic_synapse_indices,
                                (void*)&(frontend()->plastic_synapses[0]),
                                sizeof(int)*total_number_of_plastic_synapses,
                                hipMemcpyHostToDevice));
      }
    }

    void STDPPlasticity::reset_state() {
    }
  }
}
