#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Plasticity/EvansSTDPPlasticity.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, EvansSTDPPlasticity);

namespace Backend {
  namespace CUDA {
    EvansSTDPPlasticity::~EvansSTDPPlasticity() {
      CudaSafeCall(hipFree(recent_postsynaptic_activities_D));
      CudaSafeCall(hipFree(recent_presynaptic_activities_C));
    }

    void EvansSTDPPlasticity::prepare() {
      STDPPlasticity::prepare();

      allocate_device_pointers();
    }

    void EvansSTDPPlasticity::reset_state() {
      STDPPlasticity::reset_state();


      CudaSafeCall(hipMemcpy(recent_presynaptic_activities_C,
                              frontend()->recent_presynaptic_activities_C,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(recent_postsynaptic_activities_D,
                              frontend()->recent_postsynaptic_activities_D,
                              sizeof(float)*total_number_of_plastic_synapses,
                              hipMemcpyHostToDevice));
    }

    void EvansSTDPPlasticity::allocate_device_pointers(){
      // RUN AFTER NETWORK HAS BEEN STARTED
      // (eg, see prepare_backend() call at end of
      //  FourLayerVisionSpikingModel::finalise_model)
      CudaSafeCall(hipMalloc((void **)&recent_postsynaptic_activities_D, sizeof(float)*frontend()->neurs->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&recent_presynaptic_activities_C, sizeof(float)*frontend()->syns->total_number_of_synapses));

    }

    void EvansSTDPPlasticity::update_synaptic_efficacies_or_weights(float current_time_in_seconds, float timestep) {
        ltp_and_ltd<<<synapses_backend->number_of_synapse_blocks_per_grid, synapses_backend->threads_per_block>>>
          (synapses_backend->postsynaptic_neuron_indices,
           synapses_backend->presynaptic_neuron_indices,
           synapses_backend->delays,
           neurons_backend->d_neuron_data,
           input_neurons_backend->d_neuron_data,
           synapses_backend->synaptic_efficacies_or_weights,
           recent_presynaptic_activities_C,
           recent_postsynaptic_activities_D,
           *(frontend()->stdp_params),
           timestep,
           frontend()->model->timestep_grouping,
           current_time_in_seconds,
           plastic_synapse_indices,
           total_number_of_plastic_synapses);
          CudaCheckError();
    }
    
    __global__ void ltp_and_ltd
          (int* d_postsyns,
           int* d_presyns,
           int* d_syndelays,
           spiking_neurons_data_struct* neuron_data,
           spiking_neurons_data_struct* input_neuron_data,
           float* d_synaptic_efficacies_or_weights,
           float* recent_presynaptic_activities_C,
           float* recent_postsynaptic_activities_D,
           evans_stdp_plasticity_parameters_struct stdp_vars,
           float timestep,
           int timestep_grouping,
           float current_time_in_seconds,
           int* d_plastic_synapse_indices,
           size_t total_number_of_plastic_synapses){
      // Global Index
      int indx = threadIdx.x + blockIdx.x * blockDim.x;

      // Running though all neurons
      while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];
        
        // Getting synapse details
        float recent_presynaptic_activity_C = recent_presynaptic_activities_C[indx];
        float recent_postsynaptic_activity_D = recent_postsynaptic_activities_D[indx];
        int postid = d_postsyns[idx];
        int preid = d_presyns[idx];
        int bufsize = input_neuron_data->neuron_spike_time_bitbuffer_bytesize[0];
        float old_synaptic_weight = d_synaptic_efficacies_or_weights[idx];
        float new_synaptic_weight = old_synaptic_weight;

        // Correcting for input vs output neuron types
        bool is_input = PRESYNAPTIC_IS_INPUT(preid);
        int corr_preid = CORRECTED_PRESYNAPTIC_ID(preid, is_input);
        uint8_t* pre_bitbuffer = is_input ? input_neuron_data->neuron_spike_time_bitbuffer : neuron_data->neuron_spike_time_bitbuffer;

        // Looping over timesteps
        for (int g=0; g < timestep_grouping; g++){
          // Decaying STDP traces
          recent_presynaptic_activity_C = (1 - (timestep/stdp_vars.decay_term_tau_C)) * recent_presynaptic_activity_C;
          recent_postsynaptic_activity_D = (1 - (timestep/stdp_vars.decay_term_tau_D)) * recent_postsynaptic_activity_D;

          // Bit Indexing to detect spikes
          int postbitloc = ((int)roundf(current_time_in_seconds / timestep) + g) % (bufsize*8);
          int prebitloc = postbitloc - d_syndelays[idx];
          prebitloc = (prebitloc < 0) ? (bufsize*8 + prebitloc) : prebitloc;

          // OnPre Trace Update
          if (pre_bitbuffer[corr_preid*bufsize + (prebitloc / 8)] & (1 << (prebitloc % 8))){
            recent_presynaptic_activity_C += timestep * stdp_vars.synaptic_neurotransmitter_concentration_alpha_C * (1 - recent_presynaptic_activity_C);
          }
          // OnPost Trace Update
          if (neuron_data->neuron_spike_time_bitbuffer[postid*bufsize + (postbitloc / 8)] & (1 << (postbitloc % 8))){
            recent_postsynaptic_activity_D += timestep * stdp_vars.model_parameter_alpha_D * (1 - recent_postsynaptic_activity_D);
          }
          
          float syn_update_val = 0.0f; 
          old_synaptic_weight = new_synaptic_weight;
          // OnPre Weight Update
          if (pre_bitbuffer[corr_preid*bufsize + (prebitloc / 8)] & (1 << (prebitloc % 8))){
            syn_update_val -= (old_synaptic_weight * recent_postsynaptic_activity_D);
          }
          // OnPost Weight Update
          if (neuron_data->neuron_spike_time_bitbuffer[postid*bufsize + (postbitloc / 8)] & (1 << (postbitloc % 8))){
            syn_update_val += ((1 - old_synaptic_weight) * recent_presynaptic_activity_C);
          }

          new_synaptic_weight = old_synaptic_weight + syn_update_val;
          if (new_synaptic_weight < 0.0f)
            new_synaptic_weight = 0.0f;
          if (new_synaptic_weight > 1.0f)
            new_synaptic_weight = 1.0f;
        }
        
        // Weight Update
        d_synaptic_efficacies_or_weights[idx] = new_synaptic_weight;

        // Correctly set the trace values
        recent_presynaptic_activities_C[indx] = recent_presynaptic_activity_C;
        recent_postsynaptic_activities_D[indx] = recent_postsynaptic_activity_D;

        indx += blockDim.x * gridDim.x;
      }

    }


  }
}
