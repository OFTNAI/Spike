#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/LIFSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, LIFSpikingNeurons);

namespace Backend {
  namespace CUDA {
    namespace INLINE_LIF {
      #include "Spike/Backend/CUDA/InlineDeviceFunctions.hpp"
    }
    LIFSpikingNeurons::~LIFSpikingNeurons() {
      CudaSafeCall(hipFree(membrane_time_constants_tau_m));
      CudaSafeCall(hipFree(membrane_resistances_R));
    }

    void LIFSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&membrane_time_constants_tau_m, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&membrane_resistances_R, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipFree(d_neuron_data));
      CudaSafeCall(hipMalloc((void **)&d_neuron_data, sizeof(lif_spiking_neurons_data_struct)));
    }

    void LIFSpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(membrane_time_constants_tau_m,
                              frontend()->membrane_time_constants_tau_m,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(membrane_resistances_R,
                              frontend()->membrane_resistances_R,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));
    }

    void LIFSpikingNeurons::prepare() {
      SpikingNeurons::prepare();
      allocate_device_pointers();
      copy_constants_to_device();

      lif_spiking_neurons_data_struct temp_neuron_data;
      memcpy(&temp_neuron_data, neuron_data, sizeof(spiking_neurons_data_struct));
      free(neuron_data);
      neuron_data = new lif_spiking_neurons_data_struct();
      memcpy(neuron_data, &temp_neuron_data, sizeof(spiking_neurons_data_struct));
      lif_spiking_neurons_data_struct* this_neuron_data = static_cast<lif_spiking_neurons_data_struct*>(neuron_data);
      this_neuron_data->membrane_time_constants_tau_m = membrane_time_constants_tau_m;
      this_neuron_data->membrane_resistances_R = membrane_resistances_R;
      CudaSafeCall(hipMemcpy(d_neuron_data,
                              neuron_data,
                              sizeof(lif_spiking_neurons_data_struct),
                              hipMemcpyHostToDevice));
    }

    void LIFSpikingNeurons::reset_state() {
      SpikingNeurons::reset_state();
    }

    void LIFSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      ::Backend::CUDA::SpikingSynapses* synapses_backend =
        dynamic_cast<::Backend::CUDA::SpikingSynapses*>(frontend()->model->spiking_synapses->backend());
      lif_update_membrane_potentials<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (synapses_backend->host_injection_kernel,
         synapses_backend->host_syn_activation_kernel,
         synapses_backend->d_synaptic_data,
         d_neuron_data,
         frontend()->background_current,
         timestep,
         frontend()->model->timestep_grouping,
         current_time_in_seconds,
         frontend()->refractory_period_in_seconds,
         frontend()->total_number_of_neurons);

      CudaCheckError();
    }
    /* KERNELS BELOW */
    __global__ void lif_update_membrane_potentials(
        injection_kernel current_injection_kernel,
        synaptic_activation_kernel syn_activation_kernel,
        spiking_synapses_data_struct* synaptic_data,
        spiking_neurons_data_struct* in_neuron_data,
        float background_current,
        float timestep,
        int timestep_grouping,
        float current_time_in_seconds,
        float refractory_period_in_seconds,
        size_t total_number_of_neurons) {
      // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        lif_spiking_neurons_data_struct* neuron_data = (lif_spiking_neurons_data_struct*) in_neuron_data;
        float equation_constant = timestep / neuron_data->membrane_time_constants_tau_m[idx];
        float resting_potential_V0 = neuron_data->resting_potentials_v0[idx];
        float temp_membrane_resistance_R = neuron_data->membrane_resistances_R[idx];
        float membrane_potential_Vi = neuron_data->membrane_potentials_v[idx];
        float voltage_input_for_timestep = 0.0f;

        for (int g=0; g < timestep_grouping; g++){
          #ifndef INLINEDEVICEFUNCS
            voltage_input_for_timestep = current_injection_kernel(
                  synaptic_data,
                  in_neuron_data,
                  temp_membrane_resistance_R*equation_constant,
                  membrane_potential_Vi,
                  current_time_in_seconds,
                  timestep,
                  timestep_grouping,
                  idx,
                  g);
          #else
            switch (synaptic_data->synapse_type)
            {
              case CONDUCTANCE: 
                voltage_input_for_timestep = INLINE_LIF::my_conductance_spiking_injection_kernel(
                  synaptic_data,
                  in_neuron_data,
                  temp_membrane_resistance_R*equation_constant,
                  membrane_potential_Vi,
                  current_time_in_seconds,
                  timestep,
                  timestep_grouping,
                  idx,
                  g);
                break;
              case CURRENT: 
                voltage_input_for_timestep = INLINE_LIF::my_current_spiking_injection_kernel(
                  synaptic_data,
                  in_neuron_data,
                  temp_membrane_resistance_R*equation_constant,
                  membrane_potential_Vi,
                  current_time_in_seconds,
                  timestep,
                  timestep_grouping,
                  idx,
                  g);
                break;
              case VOLTAGE: 
                voltage_input_for_timestep = INLINE_LIF::my_voltage_spiking_injection_kernel(
                  synaptic_data,
                  in_neuron_data,
                  temp_membrane_resistance_R*equation_constant,
                  membrane_potential_Vi,
                  current_time_in_seconds,
                  timestep,
                  timestep_grouping,
                  idx,
                  g);
                break;
              default:
                break;
            }
          #endif
          if (((current_time_in_seconds + g*timestep) - neuron_data->last_spike_time_of_each_neuron[idx]) > refractory_period_in_seconds){
            
            membrane_potential_Vi = equation_constant * resting_potential_V0 + (1 - equation_constant) * membrane_potential_Vi + equation_constant * background_current + voltage_input_for_timestep;
            
    
            // Finally check for a spike
            if (membrane_potential_Vi >= neuron_data->thresholds_for_action_potential_spikes[idx]){
              neuron_data->last_spike_time_of_each_neuron[idx] = current_time_in_seconds + (g*timestep);
              membrane_potential_Vi = neuron_data->after_spike_reset_potentials_vreset[idx];
              #ifndef INLINEDEVICEFUNCS
                syn_activation_kernel(
              #else
                INLINE_LIF::my_activate_synapses(
              #endif
                  synaptic_data,
                  in_neuron_data,
                  g,
                  idx,
                  false);
              //break;
              continue;
            }
          }
      }
      neuron_data->membrane_potentials_v[idx] = membrane_potential_Vi;
      idx += blockDim.x * gridDim.x;
      }
    } 


  } // namespace CUDA
} // namespace Backend
