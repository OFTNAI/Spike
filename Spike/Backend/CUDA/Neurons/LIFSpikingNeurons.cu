#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/LIFSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, LIFSpikingNeurons);

namespace Backend {
  namespace CUDA {
    LIFSpikingNeurons::~LIFSpikingNeurons() {
      CudaSafeCall(hipFree(membrane_time_constants_tau_m));
      CudaSafeCall(hipFree(membrane_resistances_R));
    }

    void LIFSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&membrane_time_constants_tau_m, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&membrane_resistances_R, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_neuron_data, sizeof(lif_spiking_neurons_data_struct)));
    }

    void LIFSpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(membrane_time_constants_tau_m,
                              frontend()->membrane_time_constants_tau_m,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(membrane_resistances_R,
                              frontend()->membrane_resistances_R,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));
    }

    void LIFSpikingNeurons::prepare() {
      SpikingNeurons::prepare();
      allocate_device_pointers();
      copy_constants_to_device();

      neuron_data = new lif_spiking_neurons_data_struct();
      memcpy(neuron_data, (static_cast<LIFSpikingNeurons*>(this)->SpikingNeurons::neuron_data), sizeof(spiking_neurons_data_struct));
      neuron_data->membrane_time_constants_tau_m = membrane_time_constants_tau_m;
      neuron_data->membrane_resistances_R = membrane_resistances_R;
      neuron_data->total_number_of_neurons = frontend()->total_number_of_neurons;
      CudaSafeCall(hipMemcpy(d_neuron_data,
                              neuron_data,
                              sizeof(lif_spiking_neurons_data_struct),
                              hipMemcpyHostToDevice));
    }

    void LIFSpikingNeurons::reset_state() {
      SpikingNeurons::reset_state();
    }

    void LIFSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      ::Backend::CUDA::ConductanceSpikingSynapses* synapses_backend =
        dynamic_cast<::Backend::CUDA::ConductanceSpikingSynapses*>(frontend()->model->spiking_synapses->backend());
      lif_update_membrane_potentials<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (synapses_backend->host_injection_kernel,
         synapses_backend->d_synaptic_data,
         d_neuron_data,
         frontend()->background_current,
         timestep,
	       frontend()->model->timestep_grouping,
         current_time_in_seconds,
         frontend()->refractory_period_in_seconds,
         frontend()->total_number_of_neurons);

      CudaCheckError();
    }
    /* KERNELS BELOW */
    __global__ void lif_update_membrane_potentials(
        injection_kernel current_injection_kernel,
        spiking_synapses_data_struct* synaptic_data,
	      spiking_neurons_data_struct* in_neuron_data,
        float background_current,
        float timestep,
				int timestep_grouping,
        float current_time_in_seconds,
        float refractory_period_in_seconds,
        size_t total_number_of_neurons) {
      // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        lif_spiking_neurons_data_struct* neuron_data = (lif_spiking_neurons_data_struct*) in_neuron_data;
        float equation_constant = timestep / neuron_data->membrane_time_constants_tau_m[idx];
        float resting_potential_V0 = neuron_data->resting_potentials[idx];
        float temp_membrane_resistance_R = neuron_data->membrane_resistances_R[idx];
        float membrane_potential_Vi = neuron_data->membrane_potentials_v[idx];

    	  for (int g=0; g < timestep_grouping; g++){	  
            float current_injection_I = current_injection_kernel(
                  synaptic_data,
                  in_neuron_data,
                  membrane_potential_Vi,
                  timestep,
                  timestep_grouping,
                  idx,
                  g);
            if (((current_time_in_seconds + g*timestep) - neuron_data->last_spike_time_of_each_neuron[idx]) >= refractory_period_in_seconds){
              
              membrane_potential_Vi = equation_constant * (resting_potential_V0 + temp_membrane_resistance_R * current_injection_I) + (1 - equation_constant) * membrane_potential_Vi + equation_constant * background_current;
              
	  
	            // Finally check for a spike
	            if (membrane_potential_Vi >= neuron_data->thresholds_for_action_potential_spikes[idx]){
	  	          neuron_data->last_spike_time_of_each_neuron[idx] = current_time_in_seconds + (g*timestep);
		            membrane_potential_Vi = neuron_data->resting_potentials[idx];
                //break;
		            continue;
	      }

	    }
	  }
          
	  neuron_data->membrane_potentials_v[idx] = membrane_potential_Vi;
	  
          idx += blockDim.x * gridDim.x;
        }
     } 


  } // namespace CUDA
} // namespace Backend
