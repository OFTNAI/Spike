#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/GeneratorInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, GeneratorInputSpikingNeurons)

namespace Backend {
  namespace CUDA {
    GeneratorInputSpikingNeurons::~GeneratorInputSpikingNeurons() {
      CudaSafeCall(hipFree(neuron_ids_for_stimulus));
      CudaSafeCall(hipFree(spike_times_for_stimulus));
    }
    
    // Allocate device pointers for the longest stimulus so that they do not need to be replaced
    void GeneratorInputSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&neuron_ids_for_stimulus, sizeof(int)*frontend()->length_of_longest_stimulus));
      CudaSafeCall(hipMalloc((void **)&spike_times_for_stimulus, sizeof(float)*frontend()->length_of_longest_stimulus));
    }

    void GeneratorInputSpikingNeurons::prepare() {
      InputSpikingNeurons::prepare();
      CudaSafeCall(hipMalloc((void **)&neuron_ids_for_stimulus, sizeof(int)*frontend()->length_of_longest_stimulus));
      CudaSafeCall(hipMalloc((void **)&spike_times_for_stimulus, sizeof(float)*frontend()->length_of_longest_stimulus));
    }

    void GeneratorInputSpikingNeurons::reset_state() {
      InputSpikingNeurons::reset_state();

      CudaSafeCall(hipMemcpy(neuron_ids_for_stimulus,
                              frontend()->neuron_id_matrix_for_stimuli[frontend()->current_stimulus_index],
                              sizeof(int)*frontend()->number_of_spikes_in_stimuli[frontend()->current_stimulus_index],
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(spike_times_for_stimulus,
                              frontend()->spike_times_matrix_for_stimuli[frontend()->current_stimulus_index],
                              sizeof(float)*frontend()->number_of_spikes_in_stimuli[frontend()->current_stimulus_index],
                              hipMemcpyHostToDevice));
    }

    void GeneratorInputSpikingNeurons::check_for_neuron_spikes(float current_time_in_seconds, float timestep) {
      check_for_generator_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (neuron_ids_for_stimulus,
         spike_times_for_stimulus,
         last_spike_time_of_each_neuron,
         bitarray_of_neuron_spikes,
         frontend()->bitarray_length,
         frontend()->bitarray_maximum_axonal_delay_in_timesteps,
         current_time_in_seconds,
         timestep,
         frontend()->number_of_spikes_in_stimuli[frontend()->current_stimulus_index],
         frontend()->high_fidelity_spike_flag);

      CudaCheckError();
    }

    void GeneratorInputSpikingNeurons::update_membrane_potentials(float timestep, float current_time_in_seconds) {
    }

    __global__ void check_for_generator_spikes_kernel(int *d_neuron_ids_for_stimulus,
                                                      float *d_spike_times_for_stimulus,
                                                      float* d_last_spike_time_of_each_neuron,
                                                      unsigned char* d_bitarray_of_neuron_spikes,
                                                      int bitarray_length,
                                                      int bitarray_maximum_axonal_delay_in_timesteps,
                                                      float current_time_in_seconds,
                                                      float timestep,
                                                      size_t number_of_spikes_in_stimulus,
                                                      bool high_fidelity_spike_flag) {

      // // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < number_of_spikes_in_stimulus) {
        if (fabs(current_time_in_seconds - d_spike_times_for_stimulus[idx]) < 0.5 * timestep) {
          __syncthreads();
          d_last_spike_time_of_each_neuron[d_neuron_ids_for_stimulus[idx]] = current_time_in_seconds;

          if (high_fidelity_spike_flag){
            // Get start of the given neuron's bits
            int neuron_id_spike_store_start = d_neuron_ids_for_stimulus[idx] * bitarray_length;
            // Get offset depending upon the current timestep
            int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
            int offset_byte = offset_index / 8;
            int offset_bit_pos = offset_index - (8 * offset_byte);
            // Get the specific position at which we should be putting the current value
            unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
            // Set the specific bit in the byte to on 
            byte |= (1 << offset_bit_pos);
            // Assign the byte
            d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
          }
        } else {
          // High fidelity spike storage
          if (high_fidelity_spike_flag){
            // Get start of the given neuron's bits
            int neuron_id_spike_store_start = d_neuron_ids_for_stimulus[idx] * bitarray_length;
            // Get offset depending upon the current timestep
            int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
            int offset_byte = offset_index / 8;
            int offset_bit_pos = offset_index - (8 * offset_byte);
            // Get the specific position at which we should be putting the current value
            unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
            // Set the specific bit in the byte to on 
            byte &= ~(1 << offset_bit_pos);
            // Assign the byte
            d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
          }
        }

        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
