#include "hip/hip_runtime.h"
#include "Spike/Backend/CUDA/Neurons/IzhikevichSpikingNeurons.hpp"

namespace Backend {
  namespace CUDA {

    IzhikevichSpikingNeurons::~IzhikevichSpikingNeurons() {
      CudaSafeCall(hipFree(d_param_a));
      CudaSafeCall(hipFree(d_param_b));
      CudaSafeCall(hipFree(d_param_d));
      CudaSafeCall(hipFree(d_states_u));
    }
    
    void IzhikevichSpikingNeurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps, bool high_fidelity_spike_storage) {
 	
      SpikingNeurons::allocate_device_pointers(maximum_axonal_delay_in_timesteps, high_fidelity_spike_storage);

      CudaSafeCall(hipMalloc((void **)&d_param_a, sizeof(float)*total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_param_b, sizeof(float)*total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_param_d, sizeof(float)*total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_states_u, sizeof(float)*total_number_of_neurons));
    }

    void IzhikevichSpikingNeurons::copy_constants_to_device() {
      SpikingNeurons::copy_constants_to_device();

      CudaSafeCall(hipMemcpy(d_param_a, param_a, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(d_param_b, param_b, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(d_param_d, param_d, sizeof(float)*total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void IzhikevichSpikingNeurons::check_for_neuron_spikes(float current_time_in_seconds, float timestep) {
	SpikingNeurons::check_for_neuron_spikes(current_time_in_seconds, timestep);

	reset_states_u_after_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
          (d_states_u,
           d_param_d,
           d_last_spike_time_of_each_neuron,
           current_time_in_seconds,
           total_number_of_neurons);
	CudaCheckError();
    }

    void IzhikevichSpikingNeurons::update_membrane_potentials(float timestep, float current_time_in_seconds) {

      izhikevich_update_membrane_potentials_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (d_membrane_potentials_v,
         d_states_u,
         d_param_a,
         d_param_b,
         d_current_injections,
         timestep,
         total_number_of_neurons);

      CudaCheckError();
    }

    void IzhikevichSpikingNeurons::reset() {
      CudaSafeCall(hipMemset(d_states_u, 0.0f, sizeof(float)*total_number_of_neurons));
    }

    __global__ void reset_states_u_after_spikes_kernel(float *d_states_u,
                                                       float * d_param_d,
                                                       float* d_last_spike_time_of_each_neuron,
                                                       float current_time_in_seconds,
                                                       size_t total_number_of_neurons) {
	
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {

          d_states_u[idx] += d_param_d[idx];

        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }


    __global__ void izhikevich_update_membrane_potentials_kernel(float *d_membrane_potentials_v,
                                                                 float *d_states_u,
                                                                 float *d_param_a,
                                                                 float *d_param_b,
                                                                 float* d_current_injections,
                                                                 float timestep,
                                                                 size_t total_number_of_neurons) {

      // We require the equation timestep in ms:
      float eqtimestep = timestep*1000.0f;
      // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
        // Update the neuron states according to the Izhikevich equations
        float v_update = 0.04f*d_membrane_potentials_v[idx]*d_membrane_potentials_v[idx] 
          + 5.0f*d_membrane_potentials_v[idx]
          + 140 
          - d_states_u[idx]
          + d_current_injections[idx];

        d_membrane_potentials_v[idx] += eqtimestep*v_update;
        d_states_u[idx] += eqtimestep*(d_param_a[idx] * (d_param_b[idx] * d_membrane_potentials_v[idx] - 
                                                         d_states_u[idx]));

        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
