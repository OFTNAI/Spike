#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/IzhikevichSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, IzhikevichSpikingNeurons);

namespace Backend {
  namespace CUDA {

    IzhikevichSpikingNeurons::~IzhikevichSpikingNeurons() {
      CudaSafeCall(hipFree(param_a));
      CudaSafeCall(hipFree(param_b));
      CudaSafeCall(hipFree(param_d));
      CudaSafeCall(hipFree(states_u));
    }
    
    void IzhikevichSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&param_a, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&param_b, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&param_d, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&states_u, sizeof(float)*frontend()->total_number_of_neurons));
    }

    void IzhikevichSpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(param_a, frontend()->param_a, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(param_b, frontend()->param_b, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(param_d, frontend()->param_d, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void IzhikevichSpikingNeurons::prepare() {
      SpikingNeurons::prepare();
      allocate_device_pointers();
      copy_constants_to_device();
    }

    void IzhikevichSpikingNeurons::reset_state() {
      SpikingNeurons::reset_state();
      CudaSafeCall(hipMemset(states_u, 0.0f, sizeof(float)*frontend()->total_number_of_neurons));
    }

    void IzhikevichSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      izhikevich_update_membrane_potentials_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (membrane_potentials_v,
         states_u,
         param_a,
         param_b,
         current_injections,
	 thresholds_for_action_potential_spikes,
	 last_spike_time_of_each_neuron,
	 resting_potentials,
         timestep,
	 current_time_in_seconds,
         frontend()->total_number_of_neurons);

      CudaCheckError();

	reset_states_u_after_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
          (states_u,
           param_d,
           last_spike_time_of_each_neuron,
           current_time_in_seconds,
           frontend()->total_number_of_neurons);
	CudaCheckError();
    }

    __global__ void reset_states_u_after_spikes_kernel(float *d_states_u,
                                                       float * d_param_d,
                                                       float* d_last_spike_time_of_each_neuron,
                                                       float current_time_in_seconds,
                                                       size_t total_number_of_neurons) {
	
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {

          d_states_u[idx] += d_param_d[idx];

        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }


    __global__ void izhikevich_update_membrane_potentials_kernel(float *d_membrane_potentials_v,
                                                                 float *d_states_u,
                                                                 float *d_param_a,
                                                                 float *d_param_b,
                                                                 float* d_current_injections,
								 float* thresholds_for_action_potentials,
								 float* last_spike_time_of_each_neuron,
								 float* resting_potentials,
								 float current_time_in_seconds,
                                                                 float timestep,
                                                                 size_t total_number_of_neurons) {

      // We require the equation timestep in ms:
      float eqtimestep = timestep*1000.0f;
      // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
        // Update the neuron states according to the Izhikevich equations
        float v_update = 0.04f*d_membrane_potentials_v[idx]*d_membrane_potentials_v[idx] 
          + 5.0f*d_membrane_potentials_v[idx]
          + 140 
          - d_states_u[idx]
          + d_current_injections[idx];

        d_membrane_potentials_v[idx] += eqtimestep*v_update;
        d_states_u[idx] += eqtimestep*(d_param_a[idx] * (d_param_b[idx] * d_membrane_potentials_v[idx] - 
                                                         d_states_u[idx]));

	if (d_membrane_potentials_v[idx] >= thresholds_for_action_potentials[idx]){
		d_membrane_potentials_v[idx] = resting_potentials[idx];
		last_spike_time_of_each_neuron[idx] = current_time_in_seconds;
	}

        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
