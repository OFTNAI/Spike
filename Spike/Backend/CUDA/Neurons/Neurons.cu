// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/Neurons.hpp"

// SPIKE_EXPORT_BACKEND_TYPE(CUDA, Neurons);

namespace Backend {
  namespace CUDA {
    Neurons::~Neurons() {
      CudaSafeCall(hipFree(per_neuron_afferent_synapse_count));
      CudaSafeCall(hipFree(per_neuron_efferent_synapse_count));
      CudaSafeCall(hipFree(per_neuron_efferent_synapse_total));
      CudaSafeCall(hipFree(per_neuron_efferent_synapse_indices));
      free(h_per_neuron_efferent_synapse_total);
    }

    void Neurons::allocate_device_pointers() {
      h_per_neuron_efferent_synapse_total = (int*)malloc(sizeof(int)*frontend()->total_number_of_neurons);
      for (int i = 0; i < frontend()->total_number_of_neurons; i++){
        if (i == 0)
          h_per_neuron_efferent_synapse_total[i] = frontend()->per_neuron_efferent_synapse_count[i];
	else
	  h_per_neuron_efferent_synapse_total[i] = h_per_neuron_efferent_synapse_total[i-1] + frontend()->per_neuron_efferent_synapse_count[i];
      }
      CudaSafeCall(hipMalloc((void **)&per_neuron_afferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_efferent_synapse_total, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_efferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_efferent_synapse_indices, sizeof(int)*(h_per_neuron_efferent_synapse_total[frontend()->total_number_of_neurons - 1])));
    }

    void Neurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(per_neuron_afferent_synapse_count, frontend()->per_neuron_afferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(per_neuron_efferent_synapse_total, h_per_neuron_efferent_synapse_total, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(per_neuron_efferent_synapse_count, frontend()->per_neuron_efferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      for (int i = 0; i < frontend()->total_number_of_neurons; i++)
	CudaSafeCall(hipMemcpy(&per_neuron_efferent_synapse_indices[h_per_neuron_efferent_synapse_total[i] - frontend()->per_neuron_efferent_synapse_count[i]], frontend()->per_neuron_efferent_synapse_indices[i], sizeof(int)*frontend()->per_neuron_efferent_synapse_count[i], hipMemcpyHostToDevice));
	
    }

    void Neurons::set_threads_per_block_and_blocks_per_grid(int threads) {
      threads_per_block.x = threads;
      hipDeviceProp_t deviceProp;
      int deviceID;

      hipGetDevice(&deviceID);
      hipGetDeviceProperties(&deviceProp, deviceID);

      int max_num_blocks = deviceProp.multiProcessorCount*(deviceProp.maxThreadsPerMultiProcessor / threads);

      int number_of_neuron_blocks = (frontend()->total_number_of_neurons + threads) / threads;
      number_of_neuron_blocks_per_grid = dim3(number_of_neuron_blocks);
      if (number_of_neuron_blocks > max_num_blocks)
	number_of_neuron_blocks_per_grid = dim3(max_num_blocks);
    }

    void Neurons::prepare() {
      set_threads_per_block_and_blocks_per_grid(context->params.threads_per_block_neurons);
      allocate_device_pointers();
      copy_constants_to_device();

      neuron_data = new neurons_data_struct;
      neuron_data->total_number_of_neurons = frontend()->total_number_of_neurons;
    }

    void Neurons::reset_state() {
    }  
  } // ::Backend::CUDA
} // ::Backend
