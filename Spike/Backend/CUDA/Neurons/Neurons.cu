#include "Spike/Backend/CUDA/Neurons/Neurons.hpp"

namespace Backend {
  namespace CUDA {
    Neurons::~Neurons() {
      CudaSafeCall(hipFree(per_neuron_afferent_synapse_count));
      CudaSafeCall(hipFree(current_injections));
    }

    void Neurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps,  bool high_fidelity_spike_storage) {

      CudaSafeCall(hipMalloc((void **)&current_injections, sizeof(float)*total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_afferent_synapse_count, sizeof(int)*total_number_of_neurons));
    }

    void Neurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(d_per_neuron_afferent_synapse_count, per_neuron_afferent_synapse_count, sizeof(int)*total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void Neurons::set_threads_per_block_and_blocks_per_grid(int threads) {
      threads_per_block.x = threads;

      int number_of_neuron_blocks = (total_number_of_neurons + threads) / threads;
      number_of_neuron_blocks_per_grid.x = number_of_neuron_blocks;
    }

      /**  
       *  A local, non-polymorphic function called by Neurons::reset_neuron_activities to reset Neurons::d_current_injections.
       */
    void Neurons::reset_current_injections() {
      CudaSafeCall(hipMemset(d_current_injections, 0.0f, total_number_of_neurons*sizeof(float)));
    }

    void Neurons::reset_state() {
      reset_current_injections();
    }
  } // ::Backend::CUDA
} // ::Backend
