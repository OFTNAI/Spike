// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/Neurons.hpp"

// SPIKE_EXPORT_BACKEND_TYPE(CUDA, Neurons);

namespace Backend {
  namespace CUDA {
    Neurons::~Neurons() {
      CudaSafeCall(hipFree(per_neuron_afferent_synapse_count));
      CudaSafeCall(hipFree(per_neuron_efferent_synapse_count));
      CudaSafeCall(hipFree(per_neuron_efferent_synapse_start));
      CudaSafeCall(hipFree(d_neuron_data));
    }

    void Neurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&per_neuron_afferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_efferent_synapse_start, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&per_neuron_efferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&d_neuron_data, sizeof(neurons_data_struct)));
    }

    void Neurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(per_neuron_afferent_synapse_count, frontend()->per_neuron_afferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(per_neuron_efferent_synapse_start, frontend()->per_neuron_efferent_synapse_start, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(per_neuron_efferent_synapse_count, frontend()->per_neuron_efferent_synapse_count, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void Neurons::set_threads_per_block_and_blocks_per_grid(int threads) {
      threads_per_block.x = threads;
      hipDeviceProp_t deviceProp;
      int deviceID;

      hipGetDevice(&deviceID);
      hipGetDeviceProperties(&deviceProp, deviceID);

      int max_num_blocks = (deviceProp.multiProcessorCount*(deviceProp.maxThreadsPerMultiProcessor / threads));

      int number_of_neuron_blocks = (frontend()->total_number_of_neurons + threads) / threads;
      number_of_neuron_blocks_per_grid = dim3(number_of_neuron_blocks);
      if (number_of_neuron_blocks > max_num_blocks)
	number_of_neuron_blocks_per_grid = dim3(max_num_blocks);
    }

    void Neurons::prepare() {
      set_threads_per_block_and_blocks_per_grid(context->params.threads_per_block_neurons);
      allocate_device_pointers();
      copy_constants_to_device();

      neuron_data = new neurons_data_struct();
      neuron_data->total_number_of_neurons = frontend()->total_number_of_neurons;
      neuron_data->per_neuron_efferent_synapse_count = per_neuron_efferent_synapse_count;
      neuron_data->per_neuron_efferent_synapse_start = per_neuron_efferent_synapse_start;
      CudaSafeCall(hipMemcpy(
		d_neuron_data,
		neuron_data,
		sizeof(neurons_data_struct), hipMemcpyHostToDevice));

    }

    void Neurons::reset_state() {
      CudaSafeCall(hipMemcpy(per_neuron_efferent_synapse_start, frontend()->per_neuron_efferent_synapse_start, sizeof(int)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }  
  } // ::Backend::CUDA
} // ::Backend
