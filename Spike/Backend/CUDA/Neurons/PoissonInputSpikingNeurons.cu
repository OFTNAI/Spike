#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/PoissonInputSpikingNeurons.hpp"
#include "Spike/Backend/CUDA/Neurons/SpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, PoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    namespace INLINE_POIS {
      #include "Spike/Backend/CUDA/InlineDeviceFunctions.hpp"
    }
    PoissonInputSpikingNeurons::~PoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(rates));
    }

    void PoissonInputSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&rates, sizeof(float)*frontend()->total_number_of_neurons));
    }

    void PoissonInputSpikingNeurons::copy_constants_to_device() {
      if (frontend()->rates) {
        CudaSafeCall(hipMemcpy(rates, frontend()->rates, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      }
    }

    void PoissonInputSpikingNeurons::reset_state() {
      InputSpikingNeurons::reset_state();
    }

    void PoissonInputSpikingNeurons::prepare() {
      InputSpikingNeurons::prepare();

      allocate_device_pointers();
      copy_constants_to_device();

      // Crudely assume that the RandomStateManager backend is also CUDA:
      random_state_manager_backend
        = dynamic_cast<::Backend::CUDA::RandomStateManager*>
        (frontend()->random_state_manager->backend());
      assert(random_state_manager_backend);
    }

    void PoissonInputSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      ::Backend::CUDA::SpikingSynapses* synapses_backend =
        dynamic_cast<::Backend::CUDA::SpikingSynapses*>(frontend()->model->spiking_synapses->backend());
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>(
         synapses_backend->host_syn_activation_kernel,
         synapses_backend->d_synaptic_data,
         d_neuron_data,
         random_state_manager_backend->states,
         rates,
         membrane_potentials_v,
         timestep,
         frontend()->model->timestep_grouping,
         thresholds_for_action_potential_spikes,
         resting_potentials_v0,
         last_spike_time_of_each_neuron,
         current_time_in_seconds,
         (int)roundf(current_time_in_seconds / timestep),
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

  CudaCheckError();
    }

    __global__ void poisson_update_membrane_potentials_kernel(
        synaptic_activation_kernel syn_activation_kernel,
        spiking_synapses_data_struct* synaptic_data,
        spiking_neurons_data_struct* in_neuron_data,
        hiprandState_t* d_states,
       float *d_rates,
       float *d_membrane_potentials_v,
       float timestep,
       int timestep_grouping,
       float * d_thresholds_for_action_potential_spikes,
       float* d_resting_potentials,
       float* d_last_spike_time_of_each_neuron,
       float current_time_in_seconds,
       int timestep_index,
       size_t total_number_of_input_neurons,
       int current_stimulus_index) {

   
      int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
      int idx = t_idx;
      int bufsize = in_neuron_data->neuron_spike_time_bitbuffer_bytesize[0];

      while (idx < total_number_of_input_neurons){

        int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;
        float rate = d_rates[rate_index];

        if (rate > 0.01) {
          for (int g=0; g < timestep_grouping; g++){
            int bitloc = (timestep_index + g) % (8*bufsize);
            in_neuron_data->neuron_spike_time_bitbuffer[idx*bufsize + (bitloc / 8)] &= ~(1 << (bitloc % 8));
            // Creates random float between 0 and 1 from uniform distribution
            // d_states effectively provides a different seed for each thread
            // hiprand_uniform produces different float every time you call it
            float random_float = hiprand_uniform(&d_states[t_idx]);
      
            // if the randomnumber is less than the rate
            if (random_float < (rate * timestep)) {
              in_neuron_data->neuron_spike_time_bitbuffer[idx*bufsize + (bitloc / 8)] |= (1 << (bitloc % 8));
              #ifndef INLINEDEVICEFUNCS
                syn_activation_kernel(
              #else
                INLINE_POIS::my_activate_synapses(
              #endif
                  synaptic_data,
                  in_neuron_data,
                  g,
                  idx,
                  timestep_index / timestep_grouping,
                  true);
            } 
          }
        }

        idx += blockDim.x * gridDim.x;

      }
    }
  }
}
