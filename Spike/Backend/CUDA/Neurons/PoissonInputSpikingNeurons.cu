#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/PoissonInputSpikingNeurons.hpp"

namespace Backend {
  namespace CUDA {
    PoissonInputSpikingNeurons::~PoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(rates));
    }

    void PoissonInputSpikingNeurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps, bool high_fidelity_spike_storage) {
      InputSpikingNeurons::allocate_device_pointers(maximum_axonal_delay_in_timesteps, high_fidelity_spike_storage);
      CudaSafeCall(hipMalloc((void **)&rates, sizeof(float)*frontend()->total_number_of_neurons));
    }

    void PoissonInputSpikingNeurons::copy_constants_to_device() {
      InputSpikingNeurons::copy_constants_to_device();

      if (rates != nullptr && frontend()->rates) {
        // TODO: Above check shouldn't be necessary (esp. the frontend() bit!)
        // So many bugs ...
        printf(";;;;;; %p, %p, %d\n",
               rates, frontend()->rates,
               frontend()->total_number_of_neurons);
        CudaSafeCall(hipMemcpy(rates, frontend()->rates, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      }
    }

    void PoissonInputSpikingNeurons::reset_state() {
      printf("!!! TODO PoissonInputSpikingNeurons::reset_state called\n");
    }

    void PoissonInputSpikingNeurons::prepare() {
      // TODO: Check (call copy_constants, set_up_states etc?)
      // set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);
      // allocate_device_pointers(spiking_synapses->maximum_axonal_delay_in_timesteps, high_fidelity_spike_storage);
      // copy_constants_to_device();

      InputSpikingNeurons::prepare();

      // Crudely assume that the RandomStateManager backend is also CUDA:
      random_state_manager_backend
        = dynamic_cast<::Backend::CUDA::RandomStateManager*>
        (frontend()->random_state_manager->backend());
    }

    void PoissonInputSpikingNeurons::update_membrane_potentials(float timestep, float current_time_in_seconds) {

      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>
        (random_state_manager_backend->states,
         rates,
         membrane_potentials_v,
         timestep,
         thresholds_for_action_potential_spikes,
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

	CudaCheckError();
    }

    __global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states,
                                                              float *d_rates,
                                                              float *d_membrane_potentials_v,
                                                              float timestep,
                                                              float * d_thresholds_for_action_potential_spikes,
                                                              size_t total_number_of_input_neurons,
                                                              int current_stimulus_index) {

	 
      int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
      int idx = t_idx;
      while (idx < total_number_of_input_neurons){

        int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;

        float rate = d_rates[rate_index];

        if (rate > 0.1) {
          // Creates random float between 0 and 1 from uniform distribution
          // d_states effectively provides a different seed for each thread
          // hiprand_uniform produces different float every time you call it
          float random_float = hiprand_uniform(&d_states[t_idx]);
			
          // if the randomnumber is less than the rate
          if (random_float < (rate * timestep)) {
            // Puts membrane potential above default spiking threshold
            d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;
          } 
        }

        idx += blockDim.x * gridDim.x;

      }
      __syncthreads();
    }
  }
}
