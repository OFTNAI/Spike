#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/PoissonInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, PoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    PoissonInputSpikingNeurons::~PoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(rates));
    }

    void PoissonInputSpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&rates, sizeof(float)*frontend()->total_number_of_neurons));
    }

    void PoissonInputSpikingNeurons::copy_constants_to_device() {
      if (frontend()->rates) {
        CudaSafeCall(hipMemcpy(rates, frontend()->rates, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      }
    }

    void PoissonInputSpikingNeurons::reset_state() {
      InputSpikingNeurons::reset_state();
    }

    void PoissonInputSpikingNeurons::prepare() {
      InputSpikingNeurons::prepare();

      allocate_device_pointers();
      copy_constants_to_device();

      // Crudely assume that the RandomStateManager backend is also CUDA:
      random_state_manager_backend
        = dynamic_cast<::Backend::CUDA::RandomStateManager*>
        (frontend()->random_state_manager->backend());
      assert(random_state_manager_backend);
    }

    void PoissonInputSpikingNeurons::update_membrane_potentials(float timestep, float current_time_in_seconds) {
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>
        (random_state_manager_backend->states,
         rates,
         membrane_potentials_v,
         timestep,
         thresholds_for_action_potential_spikes,
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

	CudaCheckError();
    }

    __global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states,
                                                              float *d_rates,
                                                              float *d_membrane_potentials_v,
                                                              float timestep,
                                                              float * d_thresholds_for_action_potential_spikes,
                                                              size_t total_number_of_input_neurons,
                                                              int current_stimulus_index) {

	 
      int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
      int idx = t_idx;
      while (idx < total_number_of_input_neurons){

        int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;

        float rate = d_rates[rate_index];

        if (rate > 0.1) {
          // Creates random float between 0 and 1 from uniform distribution
          // d_states effectively provides a different seed for each thread
          // hiprand_uniform produces different float every time you call it
          float random_float = hiprand_uniform(&d_states[t_idx]);
			
          // if the randomnumber is less than the rate
          if (random_float < (rate * timestep)) {
            // Puts membrane potential above default spiking threshold
            d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;
          } 
        }

        idx += blockDim.x * gridDim.x;

      }
      __syncthreads();
    }
  }
}
