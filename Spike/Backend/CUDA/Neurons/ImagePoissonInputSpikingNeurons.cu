#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/ImagePoissonInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, ImagePoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    ImagePoissonInputSpikingNeurons::~ImagePoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(gabor_input_rates));
    }

    void ImagePoissonInputSpikingNeurons::allocate_device_pointers() {
    }

    void ImagePoissonInputSpikingNeurons::copy_rates_to_device() {
      if (gabor_input_rates)
        CudaSafeCall(hipFree(gabor_input_rates));
      CudaSafeCall(hipMalloc((void **)&gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates));
      CudaSafeCall(hipMemcpy(gabor_input_rates, frontend()->gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates, hipMemcpyHostToDevice));
    }

    void ImagePoissonInputSpikingNeurons::reset_state() {
      PoissonInputSpikingNeurons::reset_state();
    }

    void ImagePoissonInputSpikingNeurons::prepare() {
      PoissonInputSpikingNeurons::prepare();
      allocate_device_pointers();
      copy_rates_to_device();
    }

    void ImagePoissonInputSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      ::Backend::CUDA::SpikingSynapses* synapses_backend =
        dynamic_cast<::Backend::CUDA::SpikingSynapses*>(frontend()->model->spiking_synapses->backend());
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>(
         synapses_backend->host_syn_activation_kernel,
         synapses_backend->d_synaptic_data,
         d_neuron_data,
         random_state_manager_backend->states,
         gabor_input_rates,
         active,
         membrane_potentials_v,
         timestep,
         frontend()->model->timestep_grouping,
         thresholds_for_action_potential_spikes,
         resting_potentials_v0,
         last_spike_time_of_each_neuron,
         current_time_in_seconds,
         (int)roundf(current_time_in_seconds / timestep),
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

      CudaCheckError();
    }
  }
}
