// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/ImagePoissonInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, ImagePoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    ImagePoissonInputSpikingNeurons::~ImagePoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(gabor_input_rates));
    }

    void ImagePoissonInputSpikingNeurons::allocate_device_pointers() {
    }

    void ImagePoissonInputSpikingNeurons::copy_rates_to_device() {
      if (gabor_input_rates)
        CudaSafeCall(hipFree(gabor_input_rates));
      CudaSafeCall(hipMalloc((void **)&gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates));
      CudaSafeCall(hipMemcpy(gabor_input_rates, frontend()->gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates, hipMemcpyHostToDevice));
    }

    void ImagePoissonInputSpikingNeurons::reset_state() {
      PoissonInputSpikingNeurons::reset_state();
    }

    void ImagePoissonInputSpikingNeurons::prepare() {
      PoissonInputSpikingNeurons::prepare();
      allocate_device_pointers();
      copy_rates_to_device();
    }

    void ImagePoissonInputSpikingNeurons::update_membrane_potentials(float timestep,float current_time_in_seconds) {
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>
        (random_state_manager_backend->states,
         gabor_input_rates,
         membrane_potentials_v,
         timestep,
         thresholds_for_action_potential_spikes,
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

	CudaCheckError();
    }
  }
}
