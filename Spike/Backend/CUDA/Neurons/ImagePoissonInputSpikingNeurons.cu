// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/ImagePoissonInputSpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, ImagePoissonInputSpikingNeurons);

namespace Backend {
  namespace CUDA {
    ImagePoissonInputSpikingNeurons::~ImagePoissonInputSpikingNeurons() {
      CudaSafeCall(hipFree(gabor_input_rates));
    }

    void ImagePoissonInputSpikingNeurons::allocate_device_pointers() {
    }

    void ImagePoissonInputSpikingNeurons::copy_rates_to_device() {
      if (gabor_input_rates)
        CudaSafeCall(hipFree(gabor_input_rates));
      CudaSafeCall(hipMalloc((void **)&gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates));
      CudaSafeCall(hipMemcpy(gabor_input_rates, frontend()->gabor_input_rates, sizeof(float)*frontend()->total_number_of_rates, hipMemcpyHostToDevice));
    }

    void ImagePoissonInputSpikingNeurons::reset_state() {
      PoissonInputSpikingNeurons::reset_state();
    }

    void ImagePoissonInputSpikingNeurons::prepare() {
      PoissonInputSpikingNeurons::prepare();
      allocate_device_pointers();
      copy_rates_to_device();
    }

    void ImagePoissonInputSpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
      poisson_update_membrane_potentials_kernel<<<random_state_manager_backend->block_dimensions, random_state_manager_backend->threads_per_block>>>
        (random_state_manager_backend->states,
         gabor_input_rates,
         membrane_potentials_v,
         timestep,
         frontend()->model->timestep_grouping,
         thresholds_for_action_potential_spikes,
         resting_potentials_v0,
         last_spike_time_of_each_neuron,
         current_time_in_seconds,
         frontend()->total_number_of_neurons,
         frontend()->current_stimulus_index);

      CudaCheckError();
    }
  }
}
