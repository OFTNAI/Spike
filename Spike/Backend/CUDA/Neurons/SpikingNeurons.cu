// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/SpikingNeurons.hpp"

// SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingNeurons);

namespace Backend {
  namespace CUDA {
    SpikingNeurons::~SpikingNeurons() {
      CudaSafeCall(hipFree(last_spike_time_of_each_neuron));
      CudaSafeCall(hipFree(membrane_potentials_v));
      CudaSafeCall(hipFree(thresholds_for_action_potential_spikes));
      CudaSafeCall(hipFree(resting_potentials));
      CudaSafeCall(hipFree(d_neuron_data));
    }

    void SpikingNeurons::allocate_device_pointers() {
      CudaSafeCall(hipMalloc((void **)&last_spike_time_of_each_neuron, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&membrane_potentials_v, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&resting_potentials, sizeof(float)*frontend()->total_number_of_neurons));
     
      CudaSafeCall(hipMalloc((void **)&d_neuron_data, sizeof(spiking_neurons_data_struct)));
    }

    void SpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(thresholds_for_action_potential_spikes, frontend()->thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(resting_potentials, frontend()->after_spike_reset_membrane_potentials_c, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void SpikingNeurons::prepare() {
      Neurons::prepare();
      allocate_device_pointers();
      copy_constants_to_device();

      neuron_data = new spiking_neurons_data_struct();
      memcpy(neuron_data, (static_cast<SpikingNeurons*>(this)->Neurons::neuron_data), sizeof(neurons_data_struct));
      neuron_data->last_spike_time_of_each_neuron = last_spike_time_of_each_neuron;
      neuron_data->membrane_potentials_v = membrane_potentials_v;
      neuron_data->thresholds_for_action_potential_spikes = thresholds_for_action_potential_spikes;
      neuron_data->resting_potentials = resting_potentials;
      neuron_data->total_number_of_neurons = frontend()->total_number_of_neurons;
      CudaSafeCall(hipMemcpy(
		d_neuron_data, 
		neuron_data,
		sizeof(spiking_neurons_data_struct), hipMemcpyHostToDevice));
    }

    void SpikingNeurons::reset_state() {
      Neurons::reset_state();

      // Set last spike times to -1000 so that the times do not affect current simulation.
      float* tmp_last_spike_times;
      tmp_last_spike_times = (float*)malloc(sizeof(float)*frontend()->total_number_of_neurons);
      for (int i=0; i < frontend()->total_number_of_neurons; i++){
        tmp_last_spike_times[i] = -1000.0f;
      }

      CudaSafeCall(hipMemcpy(last_spike_time_of_each_neuron,
                              tmp_last_spike_times,
                              frontend()->total_number_of_neurons*sizeof(float),
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(membrane_potentials_v,
                              frontend()->after_spike_reset_membrane_potentials_c,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));

      // Free tmp_last_spike_times
      free (tmp_last_spike_times);
    }
    
    void SpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
    }


  } // ::Backend::CUDA
} // ::Backend
