#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/SpikingNeurons.hpp"

// SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingNeurons);

namespace Backend {
  namespace CUDA {
    SpikingNeurons::~SpikingNeurons() {
      CudaSafeCall(hipFree(last_spike_time_of_each_neuron));
      CudaSafeCall(hipFree(membrane_potentials_v));
      CudaSafeCall(hipFree(thresholds_for_action_potential_spikes));
      CudaSafeCall(hipFree(resting_potentials));
      CudaSafeCall(hipFree(bitarray_of_neuron_spikes));
    }

    void SpikingNeurons::allocate_device_pointers(int maximum_axonal_delay_in_timesteps, bool high_fidelity_spike_storage) {
      CudaSafeCall(hipMalloc((void **)&last_spike_time_of_each_neuron, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&membrane_potentials_v, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&resting_potentials, sizeof(float)*frontend()->total_number_of_neurons));

      if (high_fidelity_spike_storage){
        // Create bit array of correct length
        CudaSafeCall(hipMalloc((void **)&bitarray_of_neuron_spikes, sizeof(unsigned char)*frontend()->bitarray_length*frontend()->total_number_of_neurons));
      }
    }

    void SpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(thresholds_for_action_potential_spikes, frontend()->thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(resting_potentials, frontend()->after_spike_reset_membrane_potentials_c, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void SpikingNeurons::prepare() {
      Neurons::prepare();
      allocate_device_pointers(context->params.maximum_axonal_delay_in_timesteps, context->params.high_fidelity_spike_storage);
      copy_constants_to_device();
    }

    void SpikingNeurons::reset_state() {
      Neurons::reset_state();

      // Set last spike times to -1000 so that the times do not affect current simulation.
      float* tmp_last_spike_times;
      tmp_last_spike_times = (float*)malloc(sizeof(float)*frontend()->total_number_of_neurons);
      for (int i=0; i < frontend()->total_number_of_neurons; i++){
        tmp_last_spike_times[i] = -1000.0f;
      }

      CudaSafeCall(hipMemcpy(last_spike_time_of_each_neuron,
                              tmp_last_spike_times,
                              frontend()->total_number_of_neurons*sizeof(float),
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(membrane_potentials_v,
                              frontend()->after_spike_reset_membrane_potentials_c,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));

      if (frontend()->high_fidelity_spike_flag) {
        CudaSafeCall(hipMemcpy(bitarray_of_neuron_spikes,
                                frontend()->bitarray_of_neuron_spikes,
                                sizeof(unsigned char)*frontend()->bitarray_length*frontend()->total_number_of_neurons,
                                hipMemcpyHostToDevice));
      }
    }

    void SpikingNeurons::check_for_neuron_spikes(float current_time_in_seconds, float timestep) {
      check_for_neuron_spikes_kernel<<<number_of_neuron_blocks_per_grid, threads_per_block>>>
        (membrane_potentials_v,
         thresholds_for_action_potential_spikes,
         resting_potentials,
         last_spike_time_of_each_neuron,
         bitarray_of_neuron_spikes,
         frontend()->bitarray_length,
         frontend()->bitarray_maximum_axonal_delay_in_timesteps,
         current_time_in_seconds,
         timestep,
         frontend()->total_number_of_neurons,
         frontend()->high_fidelity_spike_flag);
  
      CudaCheckError();
    }

    __global__ void check_for_neuron_spikes_kernel(float *membrane_potentials_v,
                                                   float *thresholds_for_action_potential_spikes,
                                                   float *resting_potentials,
                                                   float* last_spike_time_of_each_neuron,
                                                   unsigned char* bitarray_of_neuron_spikes,
                                                   int bitarray_length,
                                                   int bitarray_maximum_axonal_delay_in_timesteps,
                                                   float current_time_in_seconds,
                                                   float timestep,
                                                   size_t total_number_of_neurons,
                                                   bool high_fidelity_spike_flag) {
      // Get thread IDs
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {
        if (membrane_potentials_v[idx] >= thresholds_for_action_potential_spikes[idx]) {

          // Set current time as last spike time of neuron
          last_spike_time_of_each_neuron[idx] = current_time_in_seconds;

          // Reset membrane potential
          membrane_potentials_v[idx] = resting_potentials[idx];

          // High fidelity spike storage
          if (high_fidelity_spike_flag){
            // Get start of the given neuron's bits
            int neuron_id_spike_store_start = idx * bitarray_length;
            // Get offset depending upon the current timestep
            int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
            int offset_byte = offset_index / 8;
            int offset_bit_pos = offset_index - (8 * offset_byte);
            // Get the specific position at which we should be putting the current value
            unsigned char byte = bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
            // Set the specific bit in the byte to on 
            byte |= (1 << offset_bit_pos);
            // Assign the byte
            bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
          }

        } else {
          // High fidelity spike storage
          if (high_fidelity_spike_flag){
            // Get start of the given neuron's bits
            int neuron_id_spike_store_start = idx * bitarray_length;
            // Get offset depending upon the current timestep
            int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
            int offset_byte = offset_index / 8;
            int offset_bit_pos = offset_index - (8 * offset_byte);
            // Get the specific position at which we should be putting the current value
            unsigned char byte = bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
            // Set the specific bit in the byte to on 
            byte &= ~(1 << offset_bit_pos);
            // Assign the byte
            bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
          }
        }

        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }

  } // ::Backend::CUDA
} // ::Backend
