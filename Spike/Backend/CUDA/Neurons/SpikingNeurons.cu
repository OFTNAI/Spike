#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/Neurons/SpikingNeurons.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, SpikingNeurons);

namespace Backend {
  namespace CUDA {
    SpikingNeurons::SpikingNeurons() {
    }
    SpikingNeurons::~SpikingNeurons() {
      CudaSafeCall(hipFree(last_spike_time_of_each_neuron));
      CudaSafeCall(hipFree(membrane_potentials_v));
      CudaSafeCall(hipFree(thresholds_for_action_potential_spikes));
      CudaSafeCall(hipFree(resting_potentials_v0));
      CudaSafeCall(hipFree(after_spike_reset_potentials_vreset));
      CudaSafeCall(hipFree(d_neuron_data));

      CudaSafeCall(hipFree(neuron_spike_time_bitbuffer));
      CudaSafeCall(hipFree(neuron_spike_time_bitbuffer_currentloc));
      CudaSafeCall(hipFree(neuron_spike_time_bitbuffer));
    }

    void SpikingNeurons::allocate_device_pointers() {
      
      CudaSafeCall(hipMalloc((void **)&last_spike_time_of_each_neuron, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&membrane_potentials_v, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&resting_potentials_v0, sizeof(float)*frontend()->total_number_of_neurons));
      CudaSafeCall(hipMalloc((void **)&after_spike_reset_potentials_vreset, sizeof(float)*frontend()->total_number_of_neurons));

      CudaSafeCall(hipMalloc((void **)&d_neuron_data, sizeof(spiking_neurons_data_struct)));
      
      h_neuron_spike_time_bitbuffer_bytesize = (int)ceil((float)(frontend()->model->spiking_synapses->maximum_axonal_delay_in_timesteps + frontend()->model->timestep_grouping) / 8.0f);
      CudaSafeCall(hipMalloc((void **)&neuron_spike_time_bitbuffer_bytesize, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&neuron_spike_time_bitbuffer_currentloc, sizeof(int)));
      CudaSafeCall(hipMalloc((void **)&neuron_spike_time_bitbuffer, sizeof(uint8_t)*h_neuron_spike_time_bitbuffer_bytesize));
    }

    void SpikingNeurons::copy_constants_to_device() {
      CudaSafeCall(hipMemcpy(thresholds_for_action_potential_spikes, frontend()->thresholds_for_action_potential_spikes, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(resting_potentials_v0, frontend()->resting_potentials_v0, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(after_spike_reset_potentials_vreset, frontend()->after_spike_reset_potentials_vreset, sizeof(float)*frontend()->total_number_of_neurons, hipMemcpyHostToDevice));
    }

    void SpikingNeurons::prepare() {
      Neurons::prepare();
      allocate_device_pointers();
      copy_constants_to_device();

      neuron_data = new spiking_neurons_data_struct();
      memcpy(neuron_data, (static_cast<SpikingNeurons*>(this)->Neurons::neuron_data), sizeof(neurons_data_struct));
      neuron_data->last_spike_time_of_each_neuron = last_spike_time_of_each_neuron;
      neuron_data->membrane_potentials_v = membrane_potentials_v;
      neuron_data->thresholds_for_action_potential_spikes = thresholds_for_action_potential_spikes;
      neuron_data->resting_potentials_v0 = resting_potentials_v0;
      neuron_data->total_number_of_neurons = frontend()->total_number_of_neurons;
      neuron_data->after_spike_reset_potentials_vreset = after_spike_reset_potentials_vreset;

      neuron_data->neuron_spike_time_bitbuffer = neuron_spike_time_bitbuffer;
      neuron_data->neuron_spike_time_bitbuffer_currentloc = neuron_spike_time_bitbuffer_currentloc;
      neuron_data->neuron_spike_time_bitbuffer_bytesize = neuron_spike_time_bitbuffer_bytesize;


      CudaSafeCall(hipMemcpy(
		d_neuron_data, 
		neuron_data,
		sizeof(spiking_neurons_data_struct), hipMemcpyHostToDevice));
    }

    void SpikingNeurons::reset_state() {
      Neurons::reset_state();

      // Set last spike times to -1000 so that the times do not affect current simulation.
      float* tmp_last_spike_times;
      tmp_last_spike_times = (float*)malloc(sizeof(float)*frontend()->total_number_of_neurons);
      for (int i=0; i < frontend()->total_number_of_neurons; i++){
        tmp_last_spike_times[i] = -1000.0f;
      }

      CudaSafeCall(hipMemcpy(last_spike_time_of_each_neuron,
                              tmp_last_spike_times,
                              frontend()->total_number_of_neurons*sizeof(float),
                              hipMemcpyHostToDevice));
      CudaSafeCall(hipMemcpy(membrane_potentials_v,
                              frontend()->resting_potentials_v0,
                              sizeof(float)*frontend()->total_number_of_neurons,
                              hipMemcpyHostToDevice));

      CudaSafeCall(hipMemset(neuron_spike_time_bitbuffer_currentloc, 0, sizeof(int)));
      CudaSafeCall(hipMemset(neuron_spike_time_bitbuffer, 0, sizeof(uint8_t)*h_neuron_spike_time_bitbuffer_bytesize));
      CudaSafeCall(hipMemcpy(neuron_spike_time_bitbuffer_bytesize,
                              &h_neuron_spike_time_bitbuffer_bytesize,
                              sizeof(int),
                              hipMemcpyHostToDevice));
      // Free tmp_last_spike_times
      free (tmp_last_spike_times);
    }
    
    void SpikingNeurons::state_update(float current_time_in_seconds, float timestep) {
    }


  } // ::Backend::CUDA
} // ::Backend
