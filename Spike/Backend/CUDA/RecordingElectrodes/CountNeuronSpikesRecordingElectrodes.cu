#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/RecordingElectrodes/CountNeuronSpikesRecordingElectrodes.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, CountNeuronSpikesRecordingElectrodes);

namespace Backend {
  namespace CUDA {
    CountNeuronSpikesRecordingElectrodes::~CountNeuronSpikesRecordingElectrodes() {
      CudaSafeCall(hipFree(per_neuron_spike_counts));
    }

    void CountNeuronSpikesRecordingElectrodes::reset_state() {
      RecordingElectrodes::reset_state();

      CudaSafeCall(hipMemset(per_neuron_spike_counts, 0, sizeof(int) * frontend()->neurons->total_number_of_neurons));
    }

    void CountNeuronSpikesRecordingElectrodes::prepare() {
      RecordingElectrodes::prepare();

      allocate_pointers_for_spike_count();
    }

    void CountNeuronSpikesRecordingElectrodes::allocate_pointers_for_spike_count() {
      CudaSafeCall(hipMalloc((void **)&per_neuron_spike_counts,
                              sizeof(int) * frontend()->neurons->total_number_of_neurons));
    }

    void CountNeuronSpikesRecordingElectrodes::add_spikes_to_per_neuron_spike_count
    (float current_time_in_seconds) {
      add_spikes_to_per_neuron_spike_count_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
        (neurons_backend->last_spike_time_of_each_neuron,
         per_neuron_spike_counts,
         current_time_in_seconds,
         frontend()->neurons->total_number_of_neurons);
	CudaCheckError();
    }

    __global__ void add_spikes_to_per_neuron_spike_count_kernel
    (float* d_last_spike_time_of_each_neuron,
     int* d_per_neuron_spike_counts,
     float current_time_in_seconds,
     size_t total_number_of_neurons) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
          atomicAdd(&d_per_neuron_spike_counts[idx], 1);
        }

        // if (idx == 1000) printf("d_per_neuron_spike_counts[idx]: %d\n", d_per_neuron_spike_counts[idx]);
        idx += blockDim.x * gridDim.x;
      }
    }
  }
}

