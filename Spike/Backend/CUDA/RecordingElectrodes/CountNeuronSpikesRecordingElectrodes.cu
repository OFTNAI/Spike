#include "hip/hip_runtime.h"
#include "Spike/Backend/CUDA/RecordingElectrodes/CountNeuronSpikesRecordingElectrodes.hpp"

namespace Backend {
  namespace CUDA {
    CountNeuronSpikesRecordingElectrodes::~CountNeuronSpikesRecordingElectrodes() {
      CudaSafeCall(hipFree(d_per_neuron_spike_counts));
    }

    CountNeuronSpikesRecordingElectrodes::reset_state() {
      CudaSafeCall(hipMemset(d_per_neuron_spike_counts, 0, sizeof(int) * neurons->total_number_of_neurons));
    }

    CountNeuronSpikesRecordingElectrodes::prepare() {
    }

    CountNeuronSpikesRecordingElectrodes::allocate_pointers_for_spike_count() {
      CudaSafeCall(hipMalloc((void **)&d_per_neuron_spike_counts, sizeof(int) * neurons->total_number_of_neurons));
    }

    CountNeuronSpikesRecordingElectrodes::add_spikes_to_per_neuron_spike_count
    (::CountNeuronSpikesRecordingElectrodes* front,
     float current_time_in_seconds) {
      add_spikes_to_per_neuron_spike_count_kernel<<<neurons->number_of_neuron_blocks_per_grid, neurons->threads_per_block>>>
        (neurons->d_last_spike_time_of_each_neuron,
         d_per_neuron_spike_counts,
         current_time_in_seconds,
         neurons->total_number_of_neurons);
	CudaCheckError();
    }

    __global__ void add_spikes_to_per_neuron_spike_count_kernel
    (float* d_last_spike_time_of_each_neuron,
     int* d_per_neuron_spike_counts,
     float current_time_in_seconds,
     size_t total_number_of_neurons) {

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
          atomicAdd(&d_per_neuron_spike_counts[idx], 1);
        }

        // if (idx == 1000) printf("d_per_neuron_spike_counts[idx]: %d\n", d_per_neuron_spike_counts[idx]);
        idx += blockDim.x * gridDim.x;
      }
    }
  }
}

