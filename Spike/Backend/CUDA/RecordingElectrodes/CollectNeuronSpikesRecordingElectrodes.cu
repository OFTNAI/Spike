#include "hip/hip_runtime.h"
#include "Spike/Backend/CUDA/RecordingElectrodes/CollectNeuronSpikesRecordingElectrodes.hpp"

namespace Backend {
  namespace CUDA {
    CollectNeuronSpikesRecordingElectrodes::~CollectNeuronSpikesRecordingElectrodes() {
      CudaSafeCall(hipFree(d_neuron_ids_of_stored_spikes_on_device));
      CudaSafeCall(hipFree(d_total_number_of_spikes_stored_on_device));
      CudaSafeCall(hipFree(d_time_in_seconds_of_stored_spikes_on_device));
    }

    CollectNeuronSpikesRecordingElectrodes::reset_state() {
      CudaSafeCall(hipMemset(&(d_total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
      CudaSafeCall(hipMemset(d_neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*neurons->total_number_of_neurons));
      CudaSafeCall(hipMemset(d_time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*neurons->total_number_of_neurons));
    }
    
    // Collect Spikes
    __global__ void collect_spikes_for_timestep_kernel
    (float* d_last_spike_time_of_each_neuron,
     int* d_total_number_of_spikes_stored_on_device,
     int* d_neuron_ids_of_stored_spikes_on_device,
     float* d_time_in_seconds_of_stored_spikes_on_device,
     float current_time_in_seconds,
     size_t total_number_of_neurons){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        // If a neuron has fired
        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
          // Increase the number of spikes stored
          // NOTE: atomicAdd return value is actually original (atomic) value BEFORE incrementation!
          //		- So first value is actually 0 not 1!!!
          int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);
          __syncthreads();

          // In the location, add the id and the time
          d_neuron_ids_of_stored_spikes_on_device[i] = idx;
          d_time_in_seconds_of_stored_spikes_on_device[i] = current_time_in_seconds;
        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
