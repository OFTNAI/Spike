#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "Spike/Backend/CUDA/RecordingElectrodes/CollectNeuronSpikesRecordingElectrodes.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, CollectNeuronSpikesRecordingElectrodes);

namespace Backend {
  namespace CUDA {
    CollectNeuronSpikesRecordingElectrodes::~CollectNeuronSpikesRecordingElectrodes() {
      CudaSafeCall(hipFree(neuron_ids_of_stored_spikes_on_device));
      CudaSafeCall(hipFree(total_number_of_spikes_stored_on_device));
      CudaSafeCall(hipFree(time_in_seconds_of_stored_spikes_on_device));
    }

    void CollectNeuronSpikesRecordingElectrodes::reset_state() {
      RecordingElectrodes::reset_state();

      CudaSafeCall(hipMemset(&(total_number_of_spikes_stored_on_device[0]), 0, sizeof(int)));
      CudaSafeCall(hipMemset(neuron_ids_of_stored_spikes_on_device, -1, sizeof(int)*neurons_frontend->total_number_of_neurons));
      CudaSafeCall(hipMemset(time_in_seconds_of_stored_spikes_on_device, -1.0f, sizeof(float)*neurons_frontend->total_number_of_neurons));
    }

    void CollectNeuronSpikesRecordingElectrodes::prepare() {
      RecordingElectrodes::prepare();

      CudaSafeCall(hipMalloc((void **)&neuron_ids_of_stored_spikes_on_device, sizeof(int)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&time_in_seconds_of_stored_spikes_on_device, sizeof(float)*frontend()->size_of_device_spike_store));
      CudaSafeCall(hipMalloc((void **)&total_number_of_spikes_stored_on_device, sizeof(int)));
    }

    void CollectNeuronSpikesRecordingElectrodes::copy_spikes_to_front() {
      CudaSafeCall(hipMemcpy((void*)&frontend()->neuron_ids_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              neuron_ids_of_stored_spikes_on_device, 
                              (sizeof(int)*frontend()->total_number_of_spikes_stored_on_device[0]), 
                              hipMemcpyDeviceToHost));
      CudaSafeCall(hipMemcpy((void*)&frontend()->time_in_seconds_of_stored_spikes_on_host[frontend()->total_number_of_spikes_stored_on_host], 
                              time_in_seconds_of_stored_spikes_on_device, 
                              sizeof(float)*frontend()->total_number_of_spikes_stored_on_device[0], 
                              hipMemcpyDeviceToHost));
    }

    void CollectNeuronSpikesRecordingElectrodes::copy_spike_counts_to_front() {
      CudaSafeCall(hipMemcpy(&(frontend()->total_number_of_spikes_stored_on_device[0]), &(total_number_of_spikes_stored_on_device[0]), (sizeof(int)), hipMemcpyDeviceToHost));
    }

    void CollectNeuronSpikesRecordingElectrodes::collect_spikes_for_timestep
    (float current_time_in_seconds) {
      collect_spikes_for_timestep_kernel<<<neurons_backend->number_of_neuron_blocks_per_grid, neurons_backend->threads_per_block>>>
        (neurons_backend->last_spike_time_of_each_neuron,
         total_number_of_spikes_stored_on_device,
         neuron_ids_of_stored_spikes_on_device,
         time_in_seconds_of_stored_spikes_on_device,
         current_time_in_seconds,
         neurons_frontend->total_number_of_neurons);

      CudaCheckError();
    }


    // Collect Spikes
    __global__ void collect_spikes_for_timestep_kernel
    (float* d_last_spike_time_of_each_neuron,
     int* d_total_number_of_spikes_stored_on_device,
     int* d_neuron_ids_of_stored_spikes_on_device,
     float* d_time_in_seconds_of_stored_spikes_on_device,
     float current_time_in_seconds,
     size_t total_number_of_neurons){

      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      while (idx < total_number_of_neurons) {

        // If a neuron has fired
        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
          // Increase the number of spikes stored
          // NOTE: atomicAdd return value is actually original (atomic) value BEFORE incrementation!
          //		- So first value is actually 0 not 1!!!
          int i = atomicAdd(&d_total_number_of_spikes_stored_on_device[0], 1);
          __syncthreads();

          // In the location, add the id and the time
          d_neuron_ids_of_stored_spikes_on_device[i] = idx;
          d_time_in_seconds_of_stored_spikes_on_device[i] = current_time_in_seconds;
        }
        idx += blockDim.x * gridDim.x;
      }
      __syncthreads();
    }
  }
}
