#include "Memory.hpp"
#include "ErrorCheck.hpp"

namespace Backend {
  namespace CUDA {
    std::size_t MemoryManager::total_bytes() const {
      size_t tmp_free, tmp_total;
      hipError_t cuda_status = hipMemGetInfo(&tmp_free, &tmp_total) ;
      if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
      }
      return tmp_total;
    }

    std::size_t MemoryManager::free_bytes() const {
      size_t tmp_free, tmp_total;
      hipError_t cuda_status = hipMemGetInfo(&tmp_free, &tmp_total) ;
      if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
      }
      return tmp_free;
    }
  }
}

SPIKE_EXPORT_BACKEND_TYPE(CUDA, MemoryManager);
