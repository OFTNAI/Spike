#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
#include "RandomStateManager.hpp"
#include "Spike/Helpers/TimerWithMessages.hpp"
#include "Spike/Helpers/TerminalHelpers.hpp"

SPIKE_EXPORT_BACKEND_TYPE(CUDA, RandomStateManager);

namespace Backend {
  namespace CUDA {
    RandomStateManager::~RandomStateManager() {
      CudaSafeCall(hipFree(states));
    }

    void RandomStateManager::setup_random_states(int threads_per_blocks_x, int number_of_blocks_x, int seed) {
      // TimerWithMessages * set_up_random_states_timer = new TimerWithMessages("Setting up random states for RandomStateManager...\n");

      threads_per_block = dim3(threads_per_blocks_x);
      block_dimensions = dim3(number_of_blocks_x);
      total_number_of_states = threads_per_blocks_x * number_of_blocks_x;

      // In case it has already been allocated
      if (states) {
        CudaSafeCall(hipFree(states));
        states = nullptr;
      }

      // Allocate the random states
      CudaSafeCall(hipMalloc((void**) &states, sizeof(hiprandState_t)*threads_per_blocks_x*number_of_blocks_x));
      generate_random_states_kernel<<<block_dimensions, threads_per_block>>>(seed, states, threads_per_blocks_x * number_of_blocks_x);
      CudaCheckError();

      // set_up_random_states_timer->stop_timer_and_log_time_and_message("Random states set up...", true);
    }

    void RandomStateManager::prepare() {
      setup_random_states();
    }

    __global__ void generate_random_states_kernel(unsigned int seed, hiprandState_t* d_states, size_t total_number) {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      // int idx_g = idx;
      if (idx < total_number) {
        hiprand_init(seed, // the seed can be the same for each core, here we pass the time in from the CPU
                    idx,  // the sequence number should be different for each core (unless you want all
                    // cores to get the same sequence of numbers for some reason - use thread id!
                    0,    // the offset is how much extra we advance in the sequence for each call; can be 0
                    &d_states[idx]);

        __syncthreads();
      }
    }
  }
}
