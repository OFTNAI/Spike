#include "catch.hpp"
#include "../Helpers/CUDAErrorCheckHelpers.h"
#include <iostream>
#include <stdio.h>
#include <fstream>

/**
		RECORDINGELECTRODES.CU Test Set
**/
#include "../RecordingElectrodes/RecordingElectrodes.h"
#include "../Neurons/SpikingNeurons.h"
#include "../Synapses/SpikingSynapses.h"
TEST_CASE("RecordingElectrode") {
	SpikingSynapses test_synapses;
	SpikingNeurons test_neurons;

	// Creating the network
	// Pre-synaptic Population
	neuron_parameters_struct neuron_params_1;
	int dim1 = 1;
	int dim2 = 100;
	neuron_params_1.group_shape[0] = dim1;
	neuron_params_1.group_shape[1] = dim2;

	int presynaptic_population = test_neurons.AddGroup(&neuron_params_1);
	
	// Post-synaptic Population
	neuron_parameters_struct neuron_params_2;
	int dim1_2 = 1;
	int dim2_2 = 100;
	neuron_params_2.group_shape[0] = dim1_2;
	neuron_params_2.group_shape[1] = dim2_2;

	int postsynaptic_population = test_neurons.AddGroup(&neuron_params_2);

	// Setting times
	float timestep = 0.01f;
	float current_time = 0.09f;

	// Setting Synapses up!
	spiking_synapse_parameters_struct synapse_params;
	synapse_params.stdp_on = true;
	synapse_params.delay_range[0] = 0.1f;
	synapse_params.delay_range[1] = 0.1f;
	// Set a fixed weight range
	synapse_params.weight_range_bottom = 0.5f;
	synapse_params.weight_range_top = 0.5f;
	// Connect
	synapse_params.connectivity_type = CONNECTIVITY_TYPE_ALL_TO_ALL;
	test_synapses.AddGroup(
		presynaptic_population,
		postsynaptic_population,
		&test_neurons,
		&test_neurons,
		timestep,
		&synapse_params);

	// Checking for spikes on every timestep, with a store of length the number of neurons. Save spikes whenever they come.
	RecordingElectrodes test_record = RecordingElectrodes(&test_neurons, "test", 1, 1, 0.0f);

	// Setting up simulation
	test_neurons.set_threads_per_block_and_blocks_per_grid(512);
	test_synapses.set_threads_per_block_and_blocks_per_grid(512);
	// Allocating Pointers
	test_neurons.allocate_device_pointers(test_synapses.maximum_axonal_delay_in_timesteps, true);
	test_synapses.allocate_device_pointers();
	// Set-up Variables
	test_neurons.copy_constants_to_device();
	test_synapses.copy_constants_and_initial_efficacies_to_device();
	test_neurons.reset_neuron_activities();
	test_synapses.reset_synapse_activities();

	// Setting up the recording electrode
	test_record.allocate_pointers_for_spike_store();
	test_record.reset_pointers_for_spike_store();

	test_record.allocate_pointers_for_spike_count();
	test_record.reset_pointers_for_spike_count();

	SECTION("Testing Initial Weight Save"){
		// Saving the initial synaptic weights to file
		test_record.write_initial_synaptic_weights_to_file(&test_synapses);
		// Check the Results dir.
		std::ifstream weightfile;
		weightfile.open("./Results/test_NetworkWeights_Initial.bin", std::ios::binary);
		
		// Check weights
		for (int i=0; i < test_synapses.total_number_of_synapses; i++){
			float test_val;
			weightfile.read((char*)&test_val, sizeof(float));
			REQUIRE(test_val == test_synapses.synaptic_efficacies_or_weights[i]);
		}
	}

	SECTION("Spike Collection Test"){
		// Set neuron last spike indices to those required:
		int indices[5] = {0, 12, 78, 9, 11};
		float* last_neuron_spike_times;
		last_neuron_spike_times = (float*)malloc(sizeof(float)*test_neurons.total_number_of_neurons);
		CudaSafeCall(hipMemcpy(last_neuron_spike_times, test_neurons.d_last_spike_time_of_each_neuron, sizeof(float)*test_neurons.total_number_of_neurons, hipMemcpyDeviceToHost));
		// Set of the neuron spike times to now
		for (int i=0; i < 5; i++){
			last_neuron_spike_times[indices[i]] = current_time;
		}
		// Return the data to the device
		CudaSafeCall(hipMemcpy(test_neurons.d_last_spike_time_of_each_neuron, last_neuron_spike_times, sizeof(float)*test_neurons.total_number_of_neurons, hipMemcpyHostToDevice));

		// Collect Spikes and check
		test_record.collect_spikes_for_timestep(current_time);
		// Copy spikes back and ensure that the values are correct
		int* number_of_spikes = (int*)malloc(sizeof(int));
		int* spiked_neuron_ids = (int*)malloc(sizeof(int)*number_of_spikes[0]);
		float* spiked_neuron_times = (float*)malloc(sizeof(float)*number_of_spikes[0]);
		CudaSafeCall(hipMemcpy(number_of_spikes, test_record.d_total_number_of_spikes_stored_on_device, sizeof(int), hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(spiked_neuron_ids, test_record.d_neuron_ids_of_stored_spikes_on_device, sizeof(int)*number_of_spikes[0], hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(spiked_neuron_times, test_record.d_time_in_seconds_of_stored_spikes_on_device, sizeof(float)*number_of_spikes[0], hipMemcpyDeviceToHost));

		// Check Values
		REQUIRE(number_of_spikes[0] == 5);
		// Values will not necessarily be in order. So check that they are only checked once
		bool checked[5];
		for (int i=0; i < 5; i++){
			checked[i] = false;
		}

		for (int i=0; i < 5; i++){
			for (int j=0; j < 5; j++){
				if (spiked_neuron_ids[i] == indices[j]){
					if (checked[i] == false){
						checked[i] = true;
						REQUIRE(spiked_neuron_ids[i] == indices[j]);
						REQUIRE(spiked_neuron_times[i] == current_time);
					} else {
						printf("Multiple copies of a single spike!");
						REQUIRE(true == false);
					}
				}
			}
		}

		// Test the effect when we use the spike copy function
		test_record.copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(current_time, 0, 1);
		REQUIRE(test_record.h_total_number_of_spikes_stored_on_host == 5);
		for (int i=0; i < 5; i++){
			checked[i] = false;
		}
		for (int i=0; i < 5; i++){
			for (int j=0; j < 5; j++){
				if (test_record.h_neuron_ids_of_stored_spikes_on_device[i] == indices[j]){
					if (checked[i] == false){
						checked[i] = true;
						REQUIRE(test_record.h_neuron_ids_of_stored_spikes_on_device[i] == indices[j]);
						REQUIRE(test_record.h_time_in_seconds_of_stored_spikes_on_device[i] == current_time);
					} else {
						printf("Multiple copies of a single spike!");
						REQUIRE(true == false);
					}
				}
			}
		}
	}
}