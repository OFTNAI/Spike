#include "hip/hip_runtime.h"
//	Synapse Class C++
//	Synapse.cpp
//
//	Author: Nasir Ahmad
//	Date: 7/12/2015

#include "Connections.h"
// stdlib allows random numbers
#include <stdlib.h>
// Input Output
#include <stdio.h>
// allows maths
#include <math.h>

#include "Constants.h"
#include <hip/hip_runtime.h>
#include "CUDAErrorCheckHelpers.h"


// Macro to get the gaussian prob
//	INPUT:
//			x = The pre-population input neuron position that is being checked
//			u = The post-population neuron to which the connection is forming (taken as mean)
//			sigma = Standard Deviation of the gaussian distribution
#define GAUS(distance, sigma) ( (1.0f/(sigma*(sqrt(2.0f*M_PI)))) * (exp(-1.0f * (pow((distance),(2.0f))) / (2.0f*(pow(sigma,(2.0f)))))) )

// Connections Constructor
Connections::Connections() {
	// Initialise my parameters
	// Variables;
	total_number_of_connections = 0;
	// Full Matrices
	presynaptic_neuron_indices = NULL;
	postsynaptic_neuron_indices = NULL;
	weights = NULL;
	delays = NULL;
	stdp = NULL;

	// On construction, seed
	srand(42);	// Seeding the random numbers
}

// Connections Destructor
Connections::~Connections() {
	// Just need to free up the memory
	// Full Matrices
	free(presynaptic_neuron_indices);
	free(postsynaptic_neuron_indices);
	free(weights);
	free(delays);
	free(stdp);
}

// Setting personal STDP parameters
void Connections::SetSTDP(float w_max_new,
				float a_minus_new,
				float a_plus_new,
				float tau_minus_new,
				float tau_plus_new){
	// Set the values
	stdp_vars.w_max = w_max_new;
	stdp_vars.a_minus = a_minus_new;
	stdp_vars.a_plus = a_plus_new;
	stdp_vars.tau_minus = tau_minus_new;
	stdp_vars.tau_plus = tau_plus_new;
}

// Connection Detail implementation
//	INPUT:
//		Pre-neuron population ID
//		Post-neuron population ID
//		An array of the exclusive sum of neuron populations
//		CONNECTIVITY_TYPE (Constants.h)
//		2 number float array for weight range
//		2 number float array for delay range
//		Boolean value to indicate if population is STDP based
//		Parameter = either probability for random connections or S.D. for Gaussian
void Connections::AddGroup(	int presynaptic_group_id, 
								int postsynaptic_group_id, 
								int* last_neuron_indices_for_each_neuron_group,
								int** group_shapes, 
								int connectivity_type,
								float weight_range[2],
								int delay_range[2],
								bool stdp_on,
								float parameter,
								float parameter_two){
	// Find the right set of indices
	// Take everything in 2D
	// Pre-Population Indices
	int prestart = 0;
	if (presynaptic_group_id > 0){
		prestart = last_neuron_indices_for_each_neuron_group[presynaptic_group_id-1];
		printf("prestart: %d\n", prestart);
	}
	int preend = last_neuron_indices_for_each_neuron_group[presynaptic_group_id];
	printf("preend: %d\n", preend);
	// Post-Population Indices
	int poststart = 0;
	if (postsynaptic_group_id > 0){
		poststart = last_neuron_indices_for_each_neuron_group[postsynaptic_group_id-1];
	}
	int postend = last_neuron_indices_for_each_neuron_group[postsynaptic_group_id];

	int original_number_of_connections = total_number_of_connections;

	// Carry out the creation of the connectivity matrix
	switch (connectivity_type){
            
		case CONNECTIVITY_TYPE_ALL_TO_ALL:
		{
            
            int increment = (preend-prestart)*(postend-poststart);
            increment_number_of_connections(increment);
            
			// If the connectivity is all_to_all
			for (int i = prestart; i < preend; i++){
				for (int j = poststart; j < postend; j++){
					// Index
					int idx = original_number_of_connections + (i-prestart) + (j-poststart)*(preend-prestart);
					// Setup Synapses
					presynaptic_neuron_indices[idx] = i;
					postsynaptic_neuron_indices[idx] = j;
				}
			}
			break;
		}
		case CONNECTIVITY_TYPE_ONE_TO_ONE:
		{
            int increment = (preend-prestart);
            increment_number_of_connections(increment);
            
			// If the connectivity is one_to_one
			if ((preend-prestart) != (postend-poststart)){
				printf("Unequal populations for one_to_one. Exiting.\n");
				exit(-1);
			}
			// Create the connectivity
			for (int i = 0; i < (preend-prestart); i++){
				presynaptic_neuron_indices[original_number_of_connections + i] = prestart + i;
				postsynaptic_neuron_indices[original_number_of_connections + i] = poststart + i;
			}

			break;
		}
		case CONNECTIVITY_TYPE_RANDOM: //JI DO
		{
			// If the connectivity is random
			// Begin a count
			for (int i = prestart; i < preend; i++){
				for (int j = poststart; j < postend; j++){
					// Probability of connection
					float prob = ((float)rand() / (RAND_MAX));
					// If it is within the probability range, connect!
					if (prob < parameter){
						
						increment_number_of_connections(1);

						// Setup Synapses
						presynaptic_neuron_indices[total_number_of_connections - 1] = i;
						postsynaptic_neuron_indices[total_number_of_connections - 1] = j;
					}
				}
			}
			break;
		}
		
		case CONNECTIVITY_TYPE_GAUSSIAN: // 1-D or 2-D
		{
			// For gaussian connectivity, the shape of the layers matters.
			// If we desire a given number of neurons, we must scale the gaussian
			float gaussian_scaling_factor = 1.0f;
			if (parameter_two != 0.0f){
				gaussian_scaling_factor = 0.0f;
				int pre_x = group_shapes[presynaptic_group_id][0] / 2;
				int pre_y = group_shapes[presynaptic_group_id][1] / 2;
				for (int i = 0; i < group_shapes[postsynaptic_group_id][0]; i++){
					for (int j = 0; j < group_shapes[postsynaptic_group_id][1]; j++){
						// Post XY
						int post_x = i;
						int post_y = j;
						// Distance
						float distance = pow((pow((float)(pre_x - post_x), 2.0f) + pow((float)(pre_y - post_y), 2.0f)), 0.5f);
						// Gaussian Probability
						gaussian_scaling_factor += GAUS(distance, parameter);
					}
				}
				// Multiplying the gaussian scaling factor by the number of connections you require:
				gaussian_scaling_factor = gaussian_scaling_factor / parameter_two;
			}
			// Running through our neurons
			for (int i = prestart; i < preend; i++){
				for (int j = poststart; j < postend; j++){
					// Probability of connection
					float prob = ((float) rand() / (RAND_MAX));
					// Get the relative distance from the two neurons
					// Pre XY
					int pre_x = (i-prestart) % group_shapes[presynaptic_group_id][0];
					int pre_y = floor((float)(i-prestart) / group_shapes[presynaptic_group_id][0]);
					// Post XY
					int post_x = (j-poststart) % group_shapes[postsynaptic_group_id][0];
					int post_y = floor((float)(j-poststart) / group_shapes[postsynaptic_group_id][0]);
					// Distance
					float distance = sqrt((pow((float)(pre_x - post_x), 2.0f) + pow((float)(pre_y - post_y), 2.0f)));
					// If it is within the probability range, connect!
					if (prob <= ((GAUS(distance, parameter)) / gaussian_scaling_factor)){
						
						increment_number_of_connections(1);

						// Setup Synapses
						presynaptic_neuron_indices[total_number_of_connections - 1] = i;
						postsynaptic_neuron_indices[total_number_of_connections - 1] = j;
					}
				}
			}
			break;
		}
		case CONNECTIVITY_TYPE_IRINA_GAUSSIAN: // 1-D only
		{
			// Getting the population sizes
			int in_size = preend - prestart;
			int out_size = postend - poststart;
			// Diagonal Width value
			float diagonal_width = parameter;
			// Irina's application of some sparse measure
			float in2out_sparse = 0.67f*0.67f;
			// Irina's implementation of some kind of stride
			int dist = 1;
			if ( (float(out_size)/float(in_size)) > 1.0f ){
				dist = int(out_size/in_size);
			}
			// Irina's version of sigma
			double sigma = dist*diagonal_width;
			// Number of connections to form
			int conn_num = int((sigma/in2out_sparse));
			int conn_tgts = 0;
			int temp = 0;
			// Running through the input neurons
			for (int i = prestart; i < preend; i++){
				double mu = int(float(dist)/2.0f) + (i-prestart)*dist;
				conn_tgts = 0;
				while (conn_tgts < conn_num) {
					temp = int(randn(mu, sigma));
					if ((temp >= 0) && (temp < out_size)){
						
						increment_number_of_connections(1);

						// Setup the synapses:
						// Setup Synapses
						presynaptic_neuron_indices[total_number_of_connections - 1] = i;
						postsynaptic_neuron_indices[total_number_of_connections - 1] = poststart + temp;

						// Increment conn_tgts
						++conn_tgts;
					}
				}
			}
			break;
		}
		case CONNECTIVITY_TYPE_SINGLE:
		{
			// If we desire a single connection
			increment_number_of_connections(1);

			// Setup Synapses
			presynaptic_neuron_indices[original_number_of_connections] = prestart + int(parameter);
			postsynaptic_neuron_indices[original_number_of_connections] = poststart + int(parameter_two);

			break;
		}
		default:
		{
			printf("\n\nUnknown Connection Type: %d\n\n", CONNECTIVITY_TYPE_SINGLE);
			exit(-1);
			break;
		}
	}

	for (int i = original_number_of_connections; i < total_number_of_connections-1; i++){
		// Setup Weights
		if (weight_range[0] == weight_range[1]) {
			weights[i] = weight_range[0];
		} else {
			float rndweight = weight_range[0] + (weight_range[1] - weight_range[0])*((float)rand() / (RAND_MAX));
			weights[i] = rndweight;
		}
		// Setup Delays
		// Get the randoms
		if (delay_range[0] == delay_range[1]) {
			delays[i] = delay_range[0];
		} else {
			float rnddelay = delay_range[0] + (delay_range[1] - delay_range[0])*((float)rand() / (RAND_MAX));
			delays[i] = round(rnddelay);
		}
		// Setup STDP
		if (stdp_on){
			stdp[i] = 1;
		} else {
			stdp[i] = 0;
		}
	}

}

void Connections::increment_number_of_connections(int increment) {
	presynaptic_neuron_indices = (int*)realloc(presynaptic_neuron_indices, (total_number_of_connections + increment)*sizeof(int));
    postsynaptic_neuron_indices = (int*)realloc(postsynaptic_neuron_indices, (total_number_of_connections + increment)*sizeof(int));
    weights = (float*)realloc(weights, (total_number_of_connections + increment)*sizeof(float));
    delays = (int*)realloc(delays, (total_number_of_connections + increment)*sizeof(int));
    stdp = (int*)realloc(stdp, (total_number_of_connections + increment)*sizeof(int));

    total_number_of_connections += increment;
}


void Connections::initialise_device_pointers() {
	CudaSafeCall(hipMalloc((void **)&d_presynaptic_neuron_indices, sizeof(int)*total_number_of_connections));
	CudaSafeCall(hipMalloc((void **)&d_postsynaptic_neuron_indices, sizeof(int)*total_number_of_connections));
	CudaSafeCall(hipMalloc((void **)&d_delays, sizeof(int)*total_number_of_connections));
	CudaSafeCall(hipMalloc((void **)&d_weights, sizeof(float)*total_number_of_connections));
	CudaSafeCall(hipMalloc((void **)&d_spikes, sizeof(int)*total_number_of_connections));


	CudaSafeCall(hipMemcpy(d_presynaptic_neuron_indices, presynaptic_neuron_indices, sizeof(int)*total_number_of_connections, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_postsynaptic_neuron_indices, postsynaptic_neuron_indices, sizeof(int)*total_number_of_connections, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_delays, delays, sizeof(int)*total_number_of_connections, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_weights, weights, sizeof(float)*total_number_of_connections, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemset(d_spikes, 0, sizeof(int)*total_number_of_connections));
}



// An implementation of the polar gaussian random number generator which I need
double randn (double mu, double sigma)
{
  double U1, U2, W, mult;
  static double X1, X2;
  static int call = 0;

  if (call == 1)
    {
      call = !call;
      return (mu + sigma * (double) X2);
    }

  do
    {
      U1 = -1 + ((double) rand () / RAND_MAX) * 2;
      U2 = -1 + ((double) rand () / RAND_MAX) * 2;
      W = pow (U1, 2) + pow (U2, 2);
    }
  while (W >= 1 || W == 0);

  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;

  call = !call;

  return (mu + sigma * (double) X1);
}