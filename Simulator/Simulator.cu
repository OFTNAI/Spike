#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm> // For random shuffle
#include <time.h>
#include <string>
#include <sys/stat.h>

#include "Simulator.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"
#include "../Helpers/RandomStateManager.h"

#include "../Models/FourLayerVisionSpikingModel.h"

using namespace std;

// Constructors
Simulator::Simulator() {}


Simulator::Simulator(SpikingModel * spiking_model_param, Simulator_Options * simulator_options_param) {

	spiking_model = spiking_model_param;
	simulator_options = simulator_options_param;

	simulations_run_count = 0;

	full_directory_name_for_simulation_data_files = "output/"; // Put into struct!!

	#ifndef SILENCE_SIMULATOR_SETUP
	TimerWithMessages * timer = new TimerWithMessages("Setting up recording electrodes...\n");
	#endif


	if (simulator_options->recording_electrodes_options->count_neuron_spikes_recording_electrodes_bool) {
		count_neuron_spikes_recording_electrodes = new CountNeuronSpikesRecordingElectrodes(spiking_model->spiking_neurons, spiking_model->spiking_synapses, full_directory_name_for_simulation_data_files, "Neurons");
		count_neuron_spikes_recording_electrodes->initialise_count_neuron_spikes_recording_electrodes();
	} else {
		count_neuron_spikes_recording_electrodes = NULL;
	}

	if (simulator_options->recording_electrodes_options->count_input_neuron_spikes_recording_electrodes_bool) {
		count_input_neuron_spikes_recording_electrodes = new CountNeuronSpikesRecordingElectrodes(spiking_model->input_spiking_neurons, spiking_model->spiking_synapses, full_directory_name_for_simulation_data_files, "Input_Neurons");
		count_input_neuron_spikes_recording_electrodes->initialise_count_neuron_spikes_recording_electrodes();
	} else {
		count_input_neuron_spikes_recording_electrodes = NULL;
	}

	if (simulator_options->recording_electrodes_options->collect_neuron_spikes_recording_electrodes_bool) {
		collect_neuron_spikes_recording_electrodes = new CollectNeuronSpikesRecordingElectrodes(spiking_model->spiking_neurons, spiking_model->spiking_synapses, full_directory_name_for_simulation_data_files, "Neurons");
		collect_neuron_spikes_recording_electrodes->initialise_collect_neuron_spikes_recording_electrodes(simulator_options->recording_electrodes_options->collect_neuron_spikes_optional_parameters);
	} else {
		collect_neuron_spikes_recording_electrodes = NULL;
	}

	if (simulator_options->recording_electrodes_options->collect_input_neuron_spikes_recording_electrodes_bool) {
		collect_input_neuron_spikes_recording_electrodes = new CollectNeuronSpikesRecordingElectrodes(spiking_model->input_spiking_neurons, spiking_model->spiking_synapses, full_directory_name_for_simulation_data_files, "Input_Neurons");
		collect_input_neuron_spikes_recording_electrodes->initialise_collect_neuron_spikes_recording_electrodes(simulator_options->recording_electrodes_options->collect_input_neuron_spikes_optional_parameters);
	} else {
		collect_input_neuron_spikes_recording_electrodes = NULL;
	}

	if (simulator_options->recording_electrodes_options->network_state_archive_recording_electrodes_bool) {
		network_state_archive_recording_electrodes->initialise_network_state_archive_recording_electrodes(simulator_options->recording_electrodes_options->network_state_archive_optional_parameters);
		network_state_archive_recording_electrodes = new NetworkStateArchiveRecordingElectrodes(spiking_model->spiking_neurons, spiking_model->spiking_synapses, full_directory_name_for_simulation_data_files, "Synapses");
	} else {
		network_state_archive_recording_electrodes = NULL;
	}

	#ifndef SILENCE_SIMULATOR_SETUP
	timer->stop_timer_and_log_time_and_message("Recording electrodes setup.\n", true);
	#endif

}

// Destructor
Simulator::~Simulator(){

	delete count_neuron_spikes_recording_electrodes;
	delete count_input_neuron_spikes_recording_electrodes;
	delete collect_neuron_spikes_recording_electrodes;
	delete collect_input_neuron_spikes_recording_electrodes;
	delete network_state_archive_recording_electrodes;

}

void Simulator::CreateDirectoryForSimulationDataFiles(string directory_name_for_simulation_data_files) {
	if (mkdir(("output/"+directory_name_for_simulation_data_files).c_str(),S_IRUSR | S_IWUSR | S_IXUSR | S_IRGRP | S_IWGRP | S_IXGRP | S_IROTH | S_IWOTH | S_IXOTH)==0)
		printf("\nDirectory created\n");
	else
		print_message_and_exit("\nERROR: You must set a different experiment name to avoid overwriting the results\n");
	full_directory_name_for_simulation_data_files = "output/"+directory_name_for_simulation_data_files+"/";
}


void Simulator::reset_all_recording_electrodes() {

	if (simulator_options->recording_electrodes_options->count_neuron_spikes_recording_electrodes_bool) {
		count_neuron_spikes_recording_electrodes->reset_pointers_for_spike_count();
	}

	if (simulator_options->recording_electrodes_options->count_input_neuron_spikes_recording_electrodes_bool) {
		count_input_neuron_spikes_recording_electrodes->reset_pointers_for_spike_count();
	}

	if (simulator_options->recording_electrodes_options->collect_neuron_spikes_recording_electrodes_bool) {
		collect_neuron_spikes_recording_electrodes->delete_and_reset_collected_spikes();
	}

	if (simulator_options->recording_electrodes_options->collect_input_neuron_spikes_recording_electrodes_bool) {
		collect_input_neuron_spikes_recording_electrodes->delete_and_reset_collected_spikes();
	}

}


void Simulator::RunSimulation(SpikeAnalyser *spike_analyser) {

	// check_for_epochs_and_begin_simulation_message(spiking_model->timestep, spiking_model->input_spiking_neurons->total_number_of_input_stimuli, number_of_epochs, collect_spikes, save_collected_spikes_and_states_to_file, spiking_model->spiking_neurons->total_number_of_neurons, spiking_model->input_spiking_neurons->total_number_of_neurons, spiking_model->spiking_synapses->total_number_of_synapses);
	// Should print something about simulator_options->stimuli_presentation_options as old stuff removed from check_for_epochs...
	TimerWithMessages * simulation_timer = new TimerWithMessages();

	// Set seed for stimulus presentation order
	srand(simulator_options->run_simulation_general_options->stimulus_presentation_order_seed);


	reset_all_recording_electrodes();

	if (simulator_options->file_storage_options->write_initial_synaptic_weights_to_file_bool) {
	
		network_state_archive_recording_electrodes->write_initial_synaptic_weights_to_file();
	
	}

	for (int epoch_number = 0; epoch_number < simulator_options->run_simulation_general_options->number_of_epochs; epoch_number++) {

		TimerWithMessages * epoch_timer = new TimerWithMessages();
		printf("Starting Epoch: %d\n", epoch_number);

		spiking_model->reset_model_activities();

		float current_time_in_seconds = 0.0f;

		int* stimuli_presentation_order = setup_stimuli_presentation_order();
		for (int stimulus_index = 0; stimulus_index < spiking_model->input_spiking_neurons->total_number_of_input_stimuli; stimulus_index++) {

			if (simulator_options->stimuli_presentation_options->reset_current_time_between_each_stimulus) current_time_in_seconds = 0.0f; // For GeneratorInputSpikingNeurons?

			perform_pre_stimulus_presentation_instructions(stimuli_presentation_order[stimulus_index]);

			//TEMP 
			// delete ((FourLayerVisionSpikingModel*)spiking_model)->image_poisson_input_spiking_neurons->random_state_manager;
			// ((FourLayerVisionSpikingModel*)spiking_model)->image_poisson_input_spiking_neurons->setup_random_states_on_device();

			int number_of_timesteps_per_stimulus_per_epoch = simulator_options->run_simulation_general_options->presentation_time_per_stimulus_per_epoch / spiking_model->timestep;
		

			for (int timestep_index = 0; timestep_index < number_of_timesteps_per_stimulus_per_epoch; timestep_index++){
				
				spiking_model->perform_per_timestep_model_instructions(current_time_in_seconds, simulator_options->run_simulation_general_options->apply_stdp_to_relevant_synapses);

				perform_per_timestep_recording_electrode_instructions(current_time_in_seconds, timestep_index, number_of_timesteps_per_stimulus_per_epoch);

				current_time_in_seconds += float(spiking_model->timestep);

			}

			perform_post_stimulus_presentation_instructions(spike_analyser);
			
		}

		perform_post_epoch_instructions(epoch_number, epoch_timer);
		
	}
	
	perform_end_of_simulation_instructions(simulation_timer);
	
}


int* Simulator::setup_stimuli_presentation_order() {

	int total_number_of_input_stimuli = spiking_model->input_spiking_neurons->total_number_of_input_stimuli;
	int total_number_of_objects = spiking_model->input_spiking_neurons->total_number_of_objects;
	int total_number_of_transformations_per_object = spiking_model->input_spiking_neurons->total_number_of_transformations_per_object;
	
	int* stimuli_presentation_order = (int*)malloc(total_number_of_input_stimuli*sizeof(int));

	// From InputSpikingNeurons
	
	for (int i = 0; i < total_number_of_input_stimuli; i++){
		stimuli_presentation_order[i] = i;
	}

	switch (simulator_options->stimuli_presentation_options->presentation_format) {

		case PRESENTATION_FORMAT_RANDOM_RESET_BETWEEN_EACH_STIMULUS: case PRESENTATION_FORMAT_RANDOM_NO_RESET: {
			std::random_shuffle(&stimuli_presentation_order[0], &stimuli_presentation_order[total_number_of_input_stimuli]);
			break;
		}

		case PRESENTATION_FORMAT_OBJECT_BY_OBJECT_RESET_BETWEEN_OBJECTS: case PRESENTATION_FORMAT_OBJECT_BY_OBJECT_NO_RESET: {
			
			int* object_order_indices = (int*)malloc(total_number_of_objects * sizeof(int));

			for (int object_index = 0; object_index < total_number_of_objects; object_index++) {
				object_order_indices[object_index] = object_index;			
			}

			switch (simulator_options->stimuli_presentation_options->object_order) {
		
				case OBJECT_ORDER_ORIGINAL:

					break;

				case OBJECT_ORDER_RANDOM:
					std::random_shuffle(&object_order_indices[0], &object_order_indices[total_number_of_objects]);
					break;

			}

			int* transform_order_indices = (int*)malloc(total_number_of_transformations_per_object * sizeof(int));
			for (int transform_index = 0; transform_index < total_number_of_transformations_per_object; transform_index++) {
				transform_order_indices[transform_index] = transform_index;			
			}

			for (int object_index = 0; object_index < total_number_of_objects; object_index++) {
				
				if (simulator_options->stimuli_presentation_options->transform_order == TRANSFORM_ORDER_RANDOM) std::random_shuffle(&transform_order_indices[0], &transform_order_indices[total_number_of_transformations_per_object]);

				for (int transform_index = 0; transform_index < total_number_of_transformations_per_object; transform_index++) {
					stimuli_presentation_order[object_index * total_number_of_transformations_per_object + transform_index] = object_order_indices[object_index] * total_number_of_transformations_per_object + transform_order_indices[transform_index]; 
				}					
			}

			break;

		}

		default:
			break;
	}

	return stimuli_presentation_order;
}



void Simulator::perform_per_timestep_recording_electrode_instructions(float current_time_in_seconds, int timestep_index, int number_of_timesteps_per_stimulus_per_epoch) {

	// Do various recording electrode operations


	// JI PSEUDO CODE FOR COLLECTING EVENTS START

	// if synapse_spike_arrived

	// 	int neuron_spike_count_for_postsynaptic_neuron = recording_electrodes->d_per_neuron_spike_counts[postsynaptic_neuron_id];

	// 	if (current_time_in_seconds > (d_ordered_spike_times_data[d_beginning_spike_time_int_indices_per_neuron[neuron_index] + neuron_spike_count_for_postsynaptic_neuron] - window) {
	// 		d_events_as_bools_per_neuron_and_spike_data[d_beginning_event_bool_indices_per_neuron + neuron_spike_count_for_postsynaptic_neuron * neurons->per_neuron_afferent_synapse_count[postsynaptic_neuron_index] + synapses->d_synapse_postsynaptic_neuron_count_index[synapse_index]] = true;
	// }


	// JI PSEUDO CODE FOR COLLECTING EVENTS END



	if (simulator_options->recording_electrodes_options->count_neuron_spikes_recording_electrodes_bool) {
	
		count_neuron_spikes_recording_electrodes->add_spikes_to_per_neuron_spike_count(current_time_in_seconds);
	
	}

	if (simulator_options->recording_electrodes_options->count_input_neuron_spikes_recording_electrodes_bool) {
	
		count_input_neuron_spikes_recording_electrodes->add_spikes_to_per_neuron_spike_count(current_time_in_seconds);
	
	}

	if (simulator_options->recording_electrodes_options->collect_neuron_spikes_recording_electrodes_bool){

		collect_neuron_spikes_recording_electrodes->collect_spikes_for_timestep(current_time_in_seconds);
		collect_neuron_spikes_recording_electrodes->copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(current_time_in_seconds, timestep_index, number_of_timesteps_per_stimulus_per_epoch );
	
	}

	if (simulator_options->recording_electrodes_options->collect_input_neuron_spikes_recording_electrodes_bool) {
		
		collect_input_neuron_spikes_recording_electrodes->collect_spikes_for_timestep(current_time_in_seconds);
		collect_input_neuron_spikes_recording_electrodes->copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(current_time_in_seconds, timestep_index, number_of_timesteps_per_stimulus_per_epoch );

	}

}


void Simulator::perform_pre_stimulus_presentation_instructions(int stimulus_index) {

	printf("Stimulus Index: %d\n", stimulus_index);
	// printf("simulator_options->stimuli_presentation_options->presentation_format: %d\n", simulator_options->stimuli_presentation_options->presentation_format);

	switch (simulator_options->stimuli_presentation_options->presentation_format) {
		case PRESENTATION_FORMAT_OBJECT_BY_OBJECT_RESET_BETWEEN_STIMULI: case PRESENTATION_FORMAT_RANDOM_RESET_BETWEEN_EACH_STIMULUS:
		{
			spiking_model->reset_model_activities();

			break;
		}
		case PRESENTATION_FORMAT_OBJECT_BY_OBJECT_RESET_BETWEEN_OBJECTS:
		{
			bool stimulus_is_new_object = spiking_model->input_spiking_neurons->stimulus_is_new_object_for_object_by_object_presentation(stimulus_index);
			// (stimulus_is_new_object) ? printf("Stimulus is new object\n") : printf("Stimulus is not new object\n");

			if (stimulus_is_new_object) {
				spiking_model->reset_model_activities();
			}

			break;
		}
		default:
			break;

	}


	spiking_model->input_spiking_neurons->current_stimulus_index = stimulus_index;
	spiking_model->input_spiking_neurons->reset_neuron_activities();


}



void Simulator::perform_post_stimulus_presentation_instructions(SpikeAnalyser * spike_analyser) {

	if (simulator_options->recording_electrodes_options->count_neuron_spikes_recording_electrodes_bool && spike_analyser) {

		spike_analyser->store_spike_counts_for_stimulus_index(spiking_model->input_spiking_neurons->current_stimulus_index, count_neuron_spikes_recording_electrodes->d_per_neuron_spike_counts);
		count_neuron_spikes_recording_electrodes->reset_pointers_for_spike_count();

	}

}


void Simulator::perform_post_epoch_instructions(int epoch_number, TimerWithMessages * epoch_timer) {

	printf("Epoch %d, Complete.\n", epoch_number);
	epoch_timer->stop_timer_and_log_time_and_message(" ", true);
	
	if (simulator_options->recording_electrodes_options->collect_neuron_spikes_recording_electrodes_bool) printf(" Number of Spikes: %d\n", collect_neuron_spikes_recording_electrodes->h_total_number_of_spikes_stored_on_host);
	if (simulator_options->recording_electrodes_options->collect_input_neuron_spikes_recording_electrodes_bool) printf(" Number of Input Spikes: %d\n", collect_input_neuron_spikes_recording_electrodes->h_total_number_of_spikes_stored_on_host);

	if (simulator_options->file_storage_options->save_recorded_neuron_spikes_to_file) collect_neuron_spikes_recording_electrodes->write_spikes_to_file(epoch_number, simulator_options->file_storage_options->network_is_trained);
	if (simulator_options->file_storage_options->save_recorded_input_neuron_spikes_to_file) collect_input_neuron_spikes_recording_electrodes->write_spikes_to_file(epoch_number, simulator_options->file_storage_options->network_is_trained);

}


void Simulator::perform_end_of_simulation_instructions(TimerWithMessages * simulation_timer) {

	simulation_timer->stop_timer_and_log_time_and_message("Simulation Complete!", true);

	if (simulator_options->recording_electrodes_options->network_state_archive_recording_electrodes_bool) {

		network_state_archive_recording_electrodes->write_network_state_to_file();

	}

	simulations_run_count++;

}