#include "hip/hip_runtime.h"
// 	Simulator Class
// 	Simulator.cu

//	Authors: Nasir Ahmad (7/12/2015), James Isbister (23/3/2016)

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm> // For random shuffle
#include <time.h>

#include "Simulator.h"
#include "../RecordingElectrodes/RecordingElectrodes.h"
#include "../Neurons/GeneratorSpikingNeurons.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"


// Constructor
Simulator::Simulator(){
	// Spike Generators

	synapses = NULL;
	neurons = NULL;
	input_neurons = NULL;

	number_of_stimuli = 0;
	numEntries = NULL;
	genids = NULL;
	gentimes = NULL;
	// Default parameters
	timestep = 0.001f;
	
	#ifndef QUIETSTART
		// Say Hi to the user:
		printf("\nWelcome to the SPIKE.\n\n");
		printf("Setting up Populations and Synapses: \n\n");
		fflush(stdout);
	#endif
}


// Destructor
Simulator::~Simulator(){

	free(neurons);
	free(input_neurons);
	free(synapses);

	free(numEntries);
	free(genids);
	free(gentimes);
}



void Simulator::SetTimestep(float timest){

	if ((synapses == NULL) || (synapses->total_number_of_synapses == 0)) {
		timestep = timest;
	} else {
		print_message_and_exit("You must set the timestep before creating any synapses.");
	}
}

void Simulator::SetNeuronType(SpikingNeurons * neurons_parameter) {

	neurons = neurons_parameter;

}

void Simulator::SetInputNeuronType(PoissonSpikingNeurons * inputs_parameter) {

	input_neurons = inputs_parameter;

}

void Simulator::SetSynapseType(SpikingSynapses * synapses_parameter) {

	synapses = synapses_parameter;

}



int Simulator::AddNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {

	if (neurons == NULL) print_message_and_exit("Please call SetNeuronType before adding neuron groups.");

	int neuron_group_id = neurons->AddGroup(group_params, group_shape);
	return neuron_group_id;

}


int Simulator::AddInputNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {

	if (input_neurons == NULL) print_message_and_exit("Please call SetInputNeuronType before adding inputs groups.");

	int input_group_id = input_neurons->AddGroup(group_params, group_shape);
	return input_group_id;

}


void Simulator::AddSynapseGroup(int presynaptic_group_id, 
							int postsynaptic_group_id, 
							int connectivity_type,
							float weight_range[2], 
							float delay_range[2],
							bool stdp_on,
							float parameter,
							float parameter_two) {

	if (synapses == NULL) print_message_and_exit("Please call SetSynapseType before adding synapses.");

	
	// Convert delay range from time to number of timesteps
	int delay_range_in_timesteps[2] = {int(round(delay_range[0]/timestep)), int(round(delay_range[1]/timestep))};

	if ((delay_range_in_timesteps[0] < 1) || (delay_range_in_timesteps[1] < 1)) {
		print_message_and_exit("Delay range must be at least one timestep.");
	}

	synapses->AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							connectivity_type, 
							weight_range,
							delay_range_in_timesteps,
							stdp_on,
							parameter,
							parameter_two);
}




void Simulator::Run(float total_time_per_epoch, int number_of_epochs, int temp_model_type, bool save_spikes, bool present_stimuli_in_random_order){

	begin_simulation_message(timestep, number_of_stimuli, number_of_epochs, save_spikes, present_stimuli_in_random_order, neurons->total_number_of_neurons, synapses->total_number_of_synapses);

	// Check how many stimuli their are and do something about it:
	if (number_of_stimuli == 0){
		++number_of_stimuli;
		numEntries = (int*)realloc(numEntries, sizeof(int)*number_of_stimuli);
		numEntries[0] = 0;
	}
	
	if (number_of_epochs == 0) print_message_and_exit("Error. There must be at least one epoch.");

	GeneratorSpikingNeurons * temp_test_generator = new GeneratorSpikingNeurons();
	RecordingElectrodes * recording_electrodes = new RecordingElectrodes(neurons);
	RecordingElectrodes * input_recording_electrodes = new RecordingElectrodes(input_neurons);

	int threads_per_block_neurons = 512;
	int threads_per_block_synapses = 512;
	synapses->set_threads_per_block_and_blocks_per_grid(threads_per_block_synapses);
	neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);
	input_neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);

	// Provides order of magnitude speedup for LIF (All to all atleast). 
	// Because all synapses contribute to current_injection on every iteration, having all threads in a block accessing only 1 or 2 positions in memory causing massive slowdown.
	// Randomising order of synapses means that each block is accessing a larger number of points in memory.
	if (temp_model_type == 1) synapses->shuffle_synapses();
	// synapses->shuffle_synapses();

	neurons->allocate_device_pointers();
	synapses->allocate_device_pointers();
	input_neurons->allocate_device_pointers();

	recording_electrodes->initialise_device_pointers();
	recording_electrodes->initialise_host_pointers();
	input_recording_electrodes->initialise_device_pointers();
	input_recording_electrodes->initialise_host_pointers();

	// SEEDING
	srand(42);

	// STIMULUS ORDER (Put into function + variable)
	int stimuli_presentation_order[number_of_stimuli];
	for (int i = 0; i < number_of_stimuli; i++){
		stimuli_presentation_order[i] = i;
	}

	recording_electrodes->write_initial_synaptic_weights_to_file(synapses);
	
	input_neurons->generate_random_states();


	clock_t begin = clock();

	for (int epoch_number = 0; epoch_number < number_of_epochs; epoch_number++) {

		if (present_stimuli_in_random_order) {
			std::random_shuffle(&stimuli_presentation_order[0], &stimuli_presentation_order[number_of_stimuli]);
		}
		// Running through every Stimulus
		for (int stimulus_index = 0; stimulus_index < number_of_stimuli; stimulus_index++){
			// Get the presentation position:
			int present = stimuli_presentation_order[stimulus_index];
			// Get the number of entries for this specific stimulus
			size_t numEnts = numEntries[present];
			if (numEnts > 0){
				temp_test_generator->initialise_device_pointers_for_ents(numEnts, present);
				temp_test_generator->set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);
			}
			// Reset the variables necessary
			neurons->reset_neurons();
			input_neurons->reset_neurons();
			synapses->reset_synapse_spikes();

			int number_of_timesteps_per_epoch = total_time_per_epoch / timestep;
			float current_time_in_seconds = 0.0f;
		
			for (int timestep_index = 0; timestep_index < number_of_timesteps_per_epoch; timestep_index++){
				
				current_time_in_seconds = float(timestep_index)*float(timestep);
				
				neurons->reset_current_injections();

				// Temporary seperation of izhikevich and LIF per timestep instructions. Eventually hope to share as much execuation as possible between both models for generality
				if (temp_model_type == 0) temp_izhikevich_per_timestep_instructions(current_time_in_seconds);
				if (temp_model_type == 1) temp_lif_per_timestep_instructions(current_time_in_seconds);

				// // Only save the spikes if necessary
				if (save_spikes){
					recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch);
					input_recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch);
				}
			}
			if (numEnts > 0){
				// CudaSafeCall(hipFree(d_genids));
				// CudaSafeCall(hipFree(d_gentimes));
			}
		}
		#ifndef QUIETSTART
		clock_t mid = clock();
		if (save_spikes) {
			printf("Epoch %d, Complete.\n Running Time: %f\n Number of Spikes: %d\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC), recording_electrodes->h_total_number_of_spikes);
			printf("Number of Input Spikes: %d\n\n", input_recording_electrodes->h_total_number_of_spikes);
		
		} else {
			printf("Epoch %d, Complete.\n Running Time: %f\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC));
		}
		#endif
		// Output Spikes list after each epoch:
		// Only save the spikes if necessary
		if (save_spikes){
			recording_electrodes->write_spikes_to_file(neurons, epoch_number);
			input_recording_electrodes->write_spikes_to_file(input_neurons, epoch_number);
		}
	}
	
	// SIMULATION COMPLETE!
	#ifndef QUIETSTART
	// Finish the simulation and check time
	clock_t end = clock();
	float timed = float(end-begin) / CLOCKS_PER_SEC;
	printf("Simulation Complete! Time Elapsed: %f\n\n", timed);
	#endif

	recording_electrodes->save_network_state(synapses);

	delete recording_electrodes;
	delete input_recording_electrodes;

}


// Temporary seperation of izhikevich and LIF per timestep instructions. Eventually hope to share as much execuation as possible between both models for generality
void Simulator::temp_izhikevich_per_timestep_instructions(float current_time_in_seconds) {

	// // If there are any spike generators
	// 	temp_test_generator->generupdate2_wrapper(current_time_in_seconds, timestep);
	
	synapses->check_for_synapse_spike_arrival(current_time_in_seconds);
	synapses->calculate_postsynaptic_current_injection(neurons, current_time_in_seconds);

	synapses->apply_ltd_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

	neurons->update_membrane_potentials(timestep);
	input_neurons->update_membrane_potentials(timestep);

	neurons->check_for_neuron_spikes(current_time_in_seconds);
	input_neurons->check_for_neuron_spikes(current_time_in_seconds);
					
	synapses->move_spikes_towards_synapses(neurons->d_last_spike_time_of_each_neuron, input_neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

	synapses->apply_ltp_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

}

void Simulator::temp_lif_per_timestep_instructions(float current_time_in_seconds) {

	// // If there are any spike generators
	// 	temp_test_generator->generupdate2_wrapper(current_time_in_seconds, timestep);
	
	synapses->check_for_synapse_spike_arrival(current_time_in_seconds);
	synapses->calculate_postsynaptic_current_injection(neurons, current_time_in_seconds);
	synapses->update_synaptic_conductances(timestep, current_time_in_seconds);


	// synapses->apply_ltd_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

	neurons->update_membrane_potentials(timestep);
	input_neurons->update_membrane_potentials(timestep);

	neurons->check_for_neuron_spikes(current_time_in_seconds);
	input_neurons->check_for_neuron_spikes(current_time_in_seconds);
					
	synapses->move_spikes_towards_synapses(neurons->d_last_spike_time_of_each_neuron, input_neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

	// synapses->apply_ltp_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

}




// Spike Generator Spike Creation
// INPUT:
//		Population ID
//		Stimulus ID
//		Number of Neurons
//		Number of entries in our arrays
//		Array of generator indices (neuron IDs)
//		Corresponding array of the spike times for each instance
void Simulator::CreateGenerator(int popID, int stimulusid, int spikenumber, int* ids, float* spiketimes){
	// We have to ensure that we have created space for the current stimulus.
	if ((number_of_stimuli - 1) < stimulusid) {

		// Check what the difference is and quit if it is too high
		if ((stimulusid - (number_of_stimuli - 1)) > 1)	print_message_and_exit("Error: Stimuli not created in order.");

		// If it isn't greater than 1, make space!
		++number_of_stimuli;
		numEntries = (int*)realloc(numEntries, sizeof(int)*number_of_stimuli);
		genids = (int**)realloc(genids, sizeof(int*)*number_of_stimuli);
		gentimes = (float**)realloc(gentimes, sizeof(float*)*number_of_stimuli);
		// Initialize stuff
		genids[stimulusid] = NULL;
		gentimes[stimulusid] = NULL;
		numEntries[stimulusid] = 0;
	}
	// Spike generator populations are necessary
	// Create space for the new ids
	
	genids[stimulusid] = (int*)realloc(genids[stimulusid], 
								sizeof(int)*(spikenumber + numEntries[stimulusid]));
	gentimes[stimulusid] = (float*)realloc(gentimes[stimulusid], 
								sizeof(float)*(spikenumber + numEntries[stimulusid]));
	
	// Check where the neuron population starts
	int startnum = 0;
	if (popID > 0) {
		startnum = neurons->last_neuron_indices_for_each_group[popID-1];
	}
	
	// Assign the genid values according to how many neurons exist already
	for (int i = 0; i < spikenumber; i++){
		genids[stimulusid][numEntries[stimulusid]+i] = ids[i] + startnum;
		gentimes[stimulusid][numEntries[stimulusid]+i] = spiketimes[i];
	}
	// Increment the number of entries the generator population
	numEntries[stimulusid] += spikenumber;
	
}



// // Synapse weight loading
// // INPUT:
// //		Number of weights that you are inputting
// //		The array in which the weights are located
// void Simulator::LoadWeights(int numWeights,
// 						float* newWeights){
// 	// Check if you have the correct number of weights
// 	if (numWeights != synconnects.numsynapses){
// 		print_message_and_exit("The number of weights being loaded is not equivalent to the model.");
// 	}
// 	// Continuing and applying the weights
// 	for (int i=0; i < numWeights; i++){
// 		synconnects.weights[i] = newWeights[i];
// 	}
// }