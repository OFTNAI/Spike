#include "hip/hip_runtime.h"
// 	Simulator Class
// 	Simulator.cu

//	Authors: Nasir Ahmad (7/12/2015), James Isbister (23/3/2016)

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm> // For random shuffle
#include <time.h>

#include "Simulator.h"
#include "../Neurons/GeneratorSpikingNeurons.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"


// Constructor
Simulator::Simulator(){
	// Spike Generators

	synapses = NULL;
	neurons = NULL;
	input_neurons = NULL;

	number_of_stimuli = 0;
	numEntries = NULL;
	genids = NULL;
	gentimes = NULL;
	// Default parameters
	timestep = 0.001f;

	recording_electrodes = NULL;
	input_recording_electrodes = NULL;
	
	#ifndef QUIETSTART
		// Say Hi to the user:
		printf("\nWelcome to the SPIKE.\n");
		print_line_of_dashes_with_blank_lines_either_side();
		printf("Setting up Neurons and Synapses:\n");
		print_line_of_dashes_with_blank_lines_either_side();
		fflush(stdout);
	#endif
}


// Destructor
Simulator::~Simulator(){

	free(neurons);
	free(input_neurons);
	free(synapses);

	free(numEntries);
	free(genids);
	free(gentimes);
}



void Simulator::SetTimestep(float timest){

	if ((synapses == NULL) || (synapses->total_number_of_synapses == 0)) {
		timestep = timest;
	} else {
		print_message_and_exit("You must set the timestep before creating any synapses.");
	}
}

void Simulator::SetNeuronType(SpikingNeurons * neurons_parameter) {

	neurons = neurons_parameter;

}

void Simulator::SetInputNeuronType(PoissonSpikingNeurons * inputs_parameter) {

	input_neurons = inputs_parameter;

}

void Simulator::SetSynapseType(SpikingSynapses * synapses_parameter) {

	synapses = synapses_parameter;

}



int Simulator::AddNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {

	if (neurons == NULL) print_message_and_exit("Please call SetNeuronType before adding neuron groups.");

	int neuron_group_id = neurons->AddGroup(group_params, group_shape);
	return neuron_group_id;

}


int Simulator::AddInputNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {

	if (input_neurons == NULL) print_message_and_exit("Please call SetInputNeuronType before adding inputs groups.");

	int input_group_id = input_neurons->AddGroup(group_params, group_shape);
	return input_group_id;

}


void Simulator::AddSynapseGroup(int presynaptic_group_id, 
							int postsynaptic_group_id, 
							float delay_range[2],
							synapse_parameters_struct * synapse_params,
							float parameter,
							float parameter_two) {

	if (synapses == NULL) print_message_and_exit("Please call SetSynapseType before adding synapses.");

	
	// Convert delay range from time to number of timesteps
	int delay_range_in_timesteps[2] = {int(round(delay_range[0]/timestep)), int(round(delay_range[1]/timestep))};

	if ((delay_range_in_timesteps[0] < 1) || (delay_range_in_timesteps[1] < 1)) {
		print_message_and_exit("Delay range must be at least one timestep.");
	}

	synapses->AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							delay_range_in_timesteps,
							synapse_params,
							parameter,
							parameter_two);
}

void Simulator::AddSynapseGroupsForNeuronGroupAndEachInputGroup(int postsynaptic_group_id, 
							float delay_range[2],
							synapse_parameters_struct * synapse_params,
							float parameter,
							float parameter_two) {

	for (int i = 0; i < input_neurons->total_number_of_groups; i++) {

		AddSynapseGroup(CORRECTED_PRESYNAPTIC_ID(i, true), 
							postsynaptic_group_id,
							delay_range,
							synapse_params,
							parameter,
							parameter_two);

	}

}


void Simulator::setup_network(bool temp_model_type) {

	printf("Setting Up Network...\n");
	clock_t initialise_network_start = clock();

	int threads_per_block_neurons = 512;
	int threads_per_block_synapses = 512;
	synapses->set_threads_per_block_and_blocks_per_grid(threads_per_block_synapses);
	neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);
	input_neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block_neurons);

	// Provides order of magnitude speedup for Conductance (All to all atleast). 
	// Because all synapses contribute to current_injection on every iteration, having all threads in a block accessing only 1 or 2 positions in memory causes massive slowdown.
	// Randomising order of synapses means that each block is accessing a larger number of points in memory.
	// if (temp_model_type == 1) synapses->shuffle_synapses();

	neurons->allocate_device_pointers();
	synapses->allocate_device_pointers();
	input_neurons->allocate_device_pointers();

	input_neurons->generate_random_states();

	clock_t initialise_network_end = clock();
	float initialise_network_total_time = float(initialise_network_end - initialise_network_start) / CLOCKS_PER_SEC;
	printf("Network Setup. Time taken: %f\n", initialise_network_total_time);
	print_line_of_dashes_with_blank_lines_either_side();

}

void Simulator::setup_recording_electrodes_for_neurons(int number_of_timesteps_per_device_spike_copy_check_param, int device_spike_store_size_multiple_of_total_neurons_param, float proportion_of_device_spike_store_full_before_copy_param) {

	printf("Setting Up recording electrodes for neurons...\n");
	clock_t setup_recording_electrodes_start = clock();

	recording_electrodes = new RecordingElectrodes(neurons, "Neurons", number_of_timesteps_per_device_spike_copy_check_param, device_spike_store_size_multiple_of_total_neurons_param, proportion_of_device_spike_store_full_before_copy_param);
	recording_electrodes->initialise_device_pointers();
	recording_electrodes->initialise_host_pointers();

	clock_t setup_recording_electrodes_end = clock();
	float setup_recording_electrodes_total_time = float(setup_recording_electrodes_end - setup_recording_electrodes_start) / CLOCKS_PER_SEC;
	printf("Recording Electrodes Setup For Neurons. Time taken: %f\n", setup_recording_electrodes_total_time);
	print_line_of_dashes_with_blank_lines_either_side();
}


void Simulator::setup_recording_electrodes_for_input_neurons(int number_of_timesteps_per_device_spike_copy_check_param, int device_spike_store_size_multiple_of_total_neurons_param, float proportion_of_device_spike_store_full_before_copy_param) {

	printf("Setting Up recording electrodes for input neurons...\n");
	clock_t setup_recording_electrodes_start = clock();

	input_recording_electrodes = new RecordingElectrodes(input_neurons, "Input_Neurons", number_of_timesteps_per_device_spike_copy_check_param, device_spike_store_size_multiple_of_total_neurons_param, proportion_of_device_spike_store_full_before_copy_param);
	input_recording_electrodes->initialise_device_pointers();
	input_recording_electrodes->initialise_host_pointers();

	clock_t setup_recording_electrodes_end = clock();
	float setup_recording_electrodes_total_time = float(setup_recording_electrodes_end - setup_recording_electrodes_start) / CLOCKS_PER_SEC;
	printf("Recording Electrodes Setup For Input Neurons. Time taken: %f\n", setup_recording_electrodes_total_time);
	print_line_of_dashes_with_blank_lines_either_side();

}


void Simulator::Run(float presentation_time_per_stimulus_per_epoch, int number_of_epochs, int temp_model_type, bool save_spikes, bool apply_stdp_to_relevant_synapses, bool count_spikes_per_neuron, bool present_stimuli_in_random_order){
	
	if (number_of_epochs == 0) print_message_and_exit("Error. There must be at least one epoch.");

	// SEEDING
	srand(42);

	// STIMULUS ORDER (Put into function + variable)
	int number_of_stimuli = input_neurons->total_number_of_input_images;
	int stimuli_presentation_order[number_of_stimuli];
	for (int i = 0; i < number_of_stimuli; i++){
		stimuli_presentation_order[i] = i;
	}

	recording_electrodes->write_initial_synaptic_weights_to_file(synapses);


	begin_simulation_message(timestep, number_of_stimuli, number_of_epochs, save_spikes, present_stimuli_in_random_order, neurons->total_number_of_neurons, input_neurons->total_number_of_neurons, synapses->total_number_of_synapses);
	clock_t simulation_begin = clock();

	for (int epoch_number = 0; epoch_number < number_of_epochs; epoch_number++) {

		printf("\nStarting Epoch: %d\n", epoch_number);

		if (present_stimuli_in_random_order) {
			std::random_shuffle(&stimuli_presentation_order[0], &stimuli_presentation_order[number_of_stimuli]);
		}

		neurons->reset_neurons();
		synapses->reset_synapse_spikes();

		float current_time_in_seconds = 0.0f;

		// Running through every Stimulus
		for (int stimulus_index = 0; stimulus_index < number_of_stimuli; stimulus_index++){

			printf("Stimulus: %d, Current time in seconds: %1.2f\n", stimuli_presentation_order[stimulus_index], current_time_in_seconds);

			input_neurons->reset_neurons();

			int number_of_timesteps_per_stimulus_per_epoch = presentation_time_per_stimulus_per_epoch / timestep;
		
			for (int timestep_index = 0; timestep_index < number_of_timesteps_per_stimulus_per_epoch; timestep_index++){
				
				neurons->reset_current_injections();

				// Temporary seperation of izhikevich and Conductance per timestep instructions. Eventually hope to share as much execuation as possible between both models for generality
				if (temp_model_type == 0) temp_izhikevich_per_timestep_instructions(current_time_in_seconds);
				if (temp_model_type == 1) temp_conductance_per_timestep_instructions(current_time_in_seconds, apply_stdp_to_relevant_synapses);

				if (count_spikes_per_neuron) recording_electrodes->add_spikes_to_per_neuron_spike_count(current_time_in_seconds);

				// // Only save the spikes if necessary
				if (save_spikes){
					if (recording_electrodes) {
						recording_electrodes->collect_spikes_for_timestep(current_time_in_seconds);
						recording_electrodes->copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(current_time_in_seconds, timestep_index, number_of_timesteps_per_stimulus_per_epoch );
					}
					if (input_recording_electrodes) {
						input_recording_electrodes->collect_spikes_for_timestep(current_time_in_seconds);
						input_recording_electrodes->copy_spikes_from_device_to_host_and_reset_device_spikes_if_device_spike_count_above_threshold(current_time_in_seconds, timestep_index, number_of_timesteps_per_stimulus_per_epoch );
					}
				}

				current_time_in_seconds += float(timestep);

			}
		}
		#ifndef QUIETSTART
		clock_t simulation_mid = clock();
		printf("Epoch %d, Complete.\n Running Time: %f\n", epoch_number, (float(simulation_mid-simulation_begin) / CLOCKS_PER_SEC));
		
		if (save_spikes) {
			if (recording_electrodes) printf(" Number of Spikes: %d\n", recording_electrodes->h_total_number_of_spikes_stored_on_host);
			if (input_recording_electrodes) printf(" Number of Input Spikes: %d\n", input_recording_electrodes->h_total_number_of_spikes_stored_on_host);
		}

		#endif
		// Output Spikes list after each epoch:
		// Only save the spikes if necessary
		if (save_spikes){
			if (recording_electrodes) recording_electrodes->write_spikes_to_file(epoch_number);
			if (input_recording_electrodes) input_recording_electrodes->write_spikes_to_file(epoch_number);
		}
	}
	
	// SIMULATION COMPLETE!
	#ifndef QUIETSTART
	// Finish the simulation and check time
	clock_t simulation_end = clock();
	float simulation_timed = float(simulation_end-simulation_begin) / CLOCKS_PER_SEC;
	print_line_of_dashes_with_blank_lines_either_side();
	printf("Simulation Complete! Time Elapsed: %f\n", simulation_timed);
	#endif

	recording_electrodes->save_network_state(synapses);

	// delete recording_electrodes;
	// delete input_recording_electrodes;

}


// Temporary seperation of izhikevich and Conductance per timestep instructions. Eventually hope to share as much execuation as possible between both models for generality
void Simulator::temp_izhikevich_per_timestep_instructions(float current_time_in_seconds) {


	// --------------- SAME ---------------
	synapses->check_for_synapse_spike_arrival(current_time_in_seconds);
	synapses->calculate_postsynaptic_current_injection(neurons, input_neurons, current_time_in_seconds);
	// --------------- SAME ---------------

	synapses->apply_ltd_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);



	// --------------- SAME ---------------
	neurons->update_membrane_potentials(timestep);
	input_neurons->update_membrane_potentials(timestep);

	neurons->check_for_neuron_spikes(current_time_in_seconds);
	input_neurons->check_for_neuron_spikes(current_time_in_seconds);
					
	synapses->move_spikes_towards_synapses(neurons->d_last_spike_time_of_each_neuron, input_neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);
	// --------------- SAME ---------------


	synapses->apply_ltp_to_synapse_weights(neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

}

void Simulator::temp_conductance_per_timestep_instructions(float current_time_in_seconds, bool apply_stdp_to_relevant_synapses) {


	// Check for NEURON_SPIKES(t+delta_t) from V(t+delta_t) and if so reset V(t+delta_t)
	neurons->check_for_neuron_spikes(current_time_in_seconds);
	input_neurons->check_for_neuron_spikes(current_time_in_seconds);
					
	synapses->move_spikes_towards_synapses(neurons->d_last_spike_time_of_each_neuron, input_neurons->d_last_spike_time_of_each_neuron, current_time_in_seconds);

	// --------------- SAME ---------------
	synapses->check_for_synapse_spike_arrival(current_time_in_seconds);

	// Calculate I(t) from delta_g(t) and V(t)
	synapses->calculate_postsynaptic_current_injection(neurons, input_neurons, current_time_in_seconds);
	// --------------- SAME ---------------

	// Calculate g(t+delta_t) and delta_g(t)
	synapses->update_synaptic_conductances(timestep, current_time_in_seconds);
	
	if (apply_stdp_to_relevant_synapses) {
		// Calculate delta_g(t+delta_t) from C(t) and D(t)
		synapses->update_synaptic_efficacies_or_weights(neurons->d_recent_postsynaptic_activities_D, timestep, current_time_in_seconds, neurons->d_last_spike_time_of_each_neuron);

		// Calculate C(t+delta_t) from C(t)
		synapses->update_presynaptic_activities(timestep, current_time_in_seconds);

		// Calculate D(t+delta_t) from D(t)
		neurons->update_postsynaptic_activities(timestep, current_time_in_seconds);
	}

	// --------------- SAME ---------------
	// Caculate V(t+delta_t) from V(t) and I(t)
	neurons->update_membrane_potentials(timestep);
	input_neurons->update_membrane_potentials(timestep);

	
	// --------------- SAME ---------------


}




// Spike Generator Spike Creation
// INPUT:
//		Population ID
//		Stimulus ID
//		Number of Neurons
//		Number of entries in our arrays
//		Array of generator indices (neuron IDs)
//		Corresponding array of the spike times for each instance
void Simulator::CreateGenerator(int popID, int stimulusid, int spikenumber, int* ids, float* spiketimes){
	// We have to ensure that we have created space for the current stimulus.
	if ((number_of_stimuli - 1) < stimulusid) {

		// Check what the difference is and quit if it is too high
		if ((stimulusid - (number_of_stimuli - 1)) > 1)	print_message_and_exit("Error: Stimuli not created in order.");

		// If it isn't greater than 1, make space!
		++number_of_stimuli;
		numEntries = (int*)realloc(numEntries, sizeof(int)*number_of_stimuli);
		genids = (int**)realloc(genids, sizeof(int*)*number_of_stimuli);
		gentimes = (float**)realloc(gentimes, sizeof(float*)*number_of_stimuli);
		// Initialize stuff
		genids[stimulusid] = NULL;
		gentimes[stimulusid] = NULL;
		numEntries[stimulusid] = 0;
	}
	// Spike generator populations are necessary
	// Create space for the new ids
	
	genids[stimulusid] = (int*)realloc(genids[stimulusid], 
								sizeof(int)*(spikenumber + numEntries[stimulusid]));
	gentimes[stimulusid] = (float*)realloc(gentimes[stimulusid], 
								sizeof(float)*(spikenumber + numEntries[stimulusid]));
	
	// Check where the neuron population starts
	int startnum = 0;
	if (popID > 0) {
		startnum = neurons->last_neuron_indices_for_each_group[popID-1];
	}
	
	// Assign the genid values according to how many neurons exist already
	for (int i = 0; i < spikenumber; i++){
		genids[stimulusid][numEntries[stimulusid]+i] = ids[i] + startnum;
		gentimes[stimulusid][numEntries[stimulusid]+i] = spiketimes[i];
	}
	// Increment the number of entries the generator population
	numEntries[stimulusid] += spikenumber;
	
}



// // Synapse weight loading
// // INPUT:
// //		Number of weights that you are inputting
// //		The array in which the weights are located
// void Simulator::LoadWeights(int numWeights,
// 						float* newWeights){
// 	// Check if you have the correct number of weights
// 	if (numWeights != synconnects.numsynapses){
// 		print_message_and_exit("The number of weights being loaded is not equivalent to the model.");
// 	}
// 	// Continuing and applying the weights
// 	for (int i=0; i < numWeights; i++){
// 		synconnects.weights[i] = newWeights[i];
// 	}
// }