#include "SpikeAnalyser.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"

// SpikeAnalyser Constructor
SpikeAnalyser::SpikeAnalyser(Neurons * neurons_parameter, PoissonSpikingNeurons * input_neurons_parameter) {
	neurons = neurons_parameter;
	input_neurons = input_neurons_parameter;

	per_stimulus_per_neuron_spike_counts = new int*[input_neurons->total_number_of_input_images];

	for (int stimulus_index = 0; stimulus_index < input_neurons->total_number_of_input_images; stimulus_index++) {
		per_stimulus_per_neuron_spike_counts[stimulus_index] = new int[neurons->total_number_of_neurons];
	}
	
}


// SpikeAnalyser Destructor
SpikeAnalyser::~SpikeAnalyser() {

}


void SpikeAnalyser::store_spike_counts_for_stimulus_index(int stimulus_index, int * d_neuron_spike_counts_for_stimulus) {
	
	CudaSafeCall(hipMemcpy(per_stimulus_per_neuron_spike_counts[stimulus_index], 
									d_neuron_spike_counts_for_stimulus, 
									sizeof(float) * neurons->total_number_of_neurons, 
									hipMemcpyDeviceToHost));

}