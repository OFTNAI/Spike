//	Neurons Class C++
//	Neurons.cpp
//
//	Author: Nasir Ahmad
//	Date: 7/12/2015
//
//  Adapted from NeuronPopulations by Nasir Ahmad and James Isbister
//	Date: 6/4/2016

#include "Neurons.h"
#include <stdlib.h>
#include "CUDAErrorCheckHelpers.h"


// Neurons Constructor
Neurons::Neurons() {

	d_last_spike_time = NULL;
	d_current_injections = NULL;

	// Set totals to zero
	total_number_of_neurons = 0;
	total_number_of_groups = 0;

	// neuron_variablesNew = NULL;

	// Initialise pointers
	group_shapes = NULL;
	last_neuron_indices_for_each_group = NULL;

}


// Neurons Destructor
Neurons::~Neurons() {

	// Free up memory
	free(group_shapes);
	free(last_neuron_indices_for_each_group);

}


int Neurons::AddGroupNew(neuron_parameters_struct * group_params, int group_shape[2]){
	
	number_of_neurons_in_new_group = group_shape[0]*group_shape[1];
 
	if (number_of_neurons_in_new_group < 0) {
		printf("\nError: Group must have at least 1 neuron.\n\n");
		exit(-1);
	}

	// Update totals
	total_number_of_neurons += number_of_neurons_in_new_group;
	++total_number_of_groups;
	printf("total_number_of_groups: %d\n", total_number_of_groups); // Temp helper

	// Calculate new group id
	int new_group_id = total_number_of_groups - 1;

	// Add last neuron index for new group
	last_neuron_indices_for_each_group = (int*)realloc(last_neuron_indices_for_each_group,(total_number_of_groups*sizeof(int)));
	last_neuron_indices_for_each_group[new_group_id] = total_number_of_neurons;

	// Add new group shape
	group_shapes = (int**)realloc(group_shapes,(total_number_of_groups*sizeof(int*)));
	group_shapes[new_group_id] = (int*)malloc(2*sizeof(int));
	group_shapes[new_group_id] = group_shape;
	
	return new_group_id;
}


void Neurons::initialise_device_pointersNew() {

	CudaSafeCall(hipMalloc((void **)&d_last_spike_time, sizeof(float)*total_number_of_neurons));
	CudaSafeCall(hipMalloc((void **)&d_current_injections, sizeof(float)*total_number_of_neurons));

	Neurons::reset_neuron_variables_and_spikesNew();
	Neurons::reset_device_current_injections();
}

void Neurons::reset_neuron_variables_and_spikesNew() {

	CudaSafeCall(hipMemset(d_last_spike_time, -1000.0f, total_number_of_neurons*sizeof(float)));
}

void Neurons::reset_device_current_injections() {
	CudaSafeCall(hipMemset(d_current_injections, 0.0f, total_number_of_neurons*sizeof(float)));
}


void Neurons::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	threads_per_block.x = threads;

	int number_of_neuron_blocks = (total_number_of_neurons + threads) / threads;
	number_of_neuron_blocks_per_grid.x = number_of_neuron_blocks;
}
