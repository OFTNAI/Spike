#include "hip/hip_runtime.h"
//	STDP Dynamics CUDA Code
//	STDPDynamics.cu
//
//	Author: Nasir Ahmad
//	Date: 15/03/2016

#include "STDPDynamics.h"
#include <stdlib.h>
#include <stdio.h>

// LTD of weights
__global__ void ltdweights(float* d_lastactive,
							float* d_weights,
							int* d_stdp,
							float* d_lastspiketime,
							int* d_postsyns,
							float currtime,
							struct stdp_struct stdp_vars,
							size_t numConns,
							size_t numNeurons){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < (numConns)) {
		// Get the locations for updating
		// Get the synapses that are to be LTD'd
		if ((d_lastactive[idx] == currtime) && (d_stdp[idx] == 1)) {
			float diff = d_lastspiketime[d_postsyns[idx]] - currtime;
			// STDP Update Rule
			float weightscale = stdp_vars.w_max * stdp_vars.a_minus * expf(diff / stdp_vars.tau_minus);
			// Now scale the weight (using an inverted column/row)
			d_weights[idx] += weightscale; 
		}
	}
}

// LTP on synapses
__global__ void synapseLTP(int* d_postsyns,
							float* d_lastspiketime,
							int* d_stdp,
							float* d_lastactive,
							float* d_weights,
							struct stdp_struct stdp_vars,
							float currtime,
							size_t numConns,
							size_t numNeurons) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numConns) {
		// Get the synapses upon which we should do LTP
		// Reversed indexing to check post->pre connections
		if ((d_lastspiketime[d_postsyns[idx]] == currtime) && (d_stdp[idx] == 1)){
			// Get the last active time / weight of the synapse
			// Calc time difference and weight change
			float diff = currtime - d_lastactive[idx];
			float weightchange = (stdp_vars.w_max - d_weights[idx]) * (stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus));
			// Update weights
			d_weights[idx] += weightchange;
		}

	}
}