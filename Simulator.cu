#include "hip/hip_runtime.h"
// 	Simulator Class
// 	Simulator.cpp
//
//	Original Author: Nasir Ahmad
//	Date: 8/12/2015
//	Originally Spike.cpp
// 
//  Adapted by Nasir Ahmad and James Isbister
//	Date: 23/3/2016

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm> // For random shuffle
#include <time.h>

#include "Simulator.h"
#include "RecordingElectrodes.h"
#include "Neurons/GeneratorSpikingNeurons.h"

#include "CUDAErrorCheckHelpers.h"


// Constructor
Simulator::Simulator(){
	// Spike Generators
	number_of_stimuli = 0;
	numEntries = NULL;
	genids = NULL;
	gentimes = NULL;
	// Default parameters
	timestep = 0.001f;

	connections = new Connections();
	
	#ifndef QUIETSTART
		// Say Hi to the user:
		printf("\nWelcome to the SPIKE.\n\n");
		printf("Setting up Populations and Synapses: \n\n");
		fflush(stdout);
	#endif
}


// Destructor
Simulator::~Simulator(){
	free(numEntries);
	free(genids);
	free(gentimes);
}


// Timestep Setting function
void Simulator::SetTimestep(float timest){

	printf("timest = %d\n\n", connections->total_number_of_connections);
	if (connections->total_number_of_connections == 0){
		timestep = timest;
	} else {
		printf("You must set the timestep before creating any synapses. Exiting ...\n\n");
		exit(-1);
	}
}


void Simulator::SetNeuronType(SpikingNeurons * neurons_parameter) {
	neurons = neurons_parameter;
}

void Simulator::SetInputNeuronType(PoissonSpikingNeurons * inputs_parameter) {
	input_neurons = inputs_parameter;
}

int Simulator::AddNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {
	if (neurons == NULL) {
		printf("Please call SetNeuronType before adding neuron groups. Exiting ...\n\n");
		exit(-1);
	}
	int neuron_group_id = neurons->AddGroup(group_params, group_shape);
	return neuron_group_id;
}


int Simulator::AddInputNeuronGroup(neuron_parameters_struct * group_params, int group_shape[2]) {
	if (input_neurons == NULL) {
		printf("Please call SetInputNeuronType before adding inputs groups. Exiting ...\n\n");
		exit(-1);
	}
	int input_group_id = input_neurons->AddGroup(group_params, group_shape);
	return input_group_id;
}


void Simulator::AddConnectionGroup(int presynaptic_group_id, 
							int postsynaptic_group_id, 
							int connectivity_type,
							float weight_range[2], 
							float delay_range[2],
							bool stdp_on,
							float parameter,
							float parameter_two) {
	
	// Convert delay range from time to number of timesteps
	int delay_range_in_timesteps[2] = {int(round(delay_range[0]/timestep)), int(round(delay_range[1]/timestep))};
	if ((delay_range_in_timesteps[0] < 1) || (delay_range_in_timesteps[1] < 1)) {
		printf("\nDelay range must be at least one timestep. Exiting ...\n\n");
		exit(-1);
	}

	connections->AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							connectivity_type, 
							weight_range,
							delay_range_in_timesteps,
							stdp_on,
							parameter,
							parameter_two);
}




void Simulator::Run(float total_time_per_epoch, int number_of_epochs, bool save_spikes, bool present_stimuli_in_random_order){
	#ifndef QUIETSTART
	printf("\n\n----------------------------------\n");
	printf("Simulation Beginning\n");
	printf("Time Step: %f\nNumber of Stimuli: %d\nNumber of Epochs: %d\n\n", timestep, number_of_stimuli, number_of_epochs);
	printf("Total Number of Neurons: %d\n", neurons->total_number_of_neurons);
	printf("Total Number of Synapses: %d\n\n", connections->total_number_of_connections);
	if (present_stimuli_in_random_order)
		printf("Stimuli to be presented in a random order.\n");
	if (save_spikes)
		printf("Spikes shall be saved.\n");
	printf("----------------------------------\n\nBeginning ...\n\n");
	#endif

	// Check how many stimuli their are and do something about it:
	if (number_of_stimuli == 0){
		++number_of_stimuli;
		numEntries = (int*)realloc(numEntries, sizeof(int)*number_of_stimuli);
		numEntries[0] = 0;
	}
	// Ensure that there is at least one epoch
	if (number_of_epochs == 0){
		printf("Error. There must be at least one epoch. Exiting ...\n\n");
		exit(-1);
	}


	GeneratorSpikingNeurons * temp_test_generator = new GeneratorSpikingNeurons();
	RecordingElectrodes * recording_electrodes = new RecordingElectrodes(neurons);
	RecordingElectrodes * input_recording_electrodes = new RecordingElectrodes(input_neurons);

	neurons->initialise_device_pointers();
	connections->initialise_device_pointers();
	input_neurons->initialise_device_pointers();

	recording_electrodes->initialise_device_pointers();
	recording_electrodes->initialise_host_pointers();
	input_recording_electrodes->initialise_device_pointers();
	input_recording_electrodes->initialise_host_pointers();


	int threads_per_block = 128;
	connections->set_threads_per_block_and_blocks_per_grid(threads_per_block);
	neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block);
	input_neurons->set_threads_per_block_and_blocks_per_grid(threads_per_block);

	// SEEDING
	srand(42);

	// STIMULUS ORDER (Put into function + variable)
	int stimuli_presentation_order[number_of_stimuli];
	for (int i = 0; i < number_of_stimuli; i++){
		stimuli_presentation_order[i] = i;
	}


	recording_electrodes->write_initial_synaptic_weights_to_file(connections);
	
	input_neurons->generate_random_states_wrapper();


	clock_t begin = clock();

	for (int epoch_number = 0; epoch_number < number_of_epochs; epoch_number++) {

		if (present_stimuli_in_random_order) {
			std::random_shuffle(&stimuli_presentation_order[0], &stimuli_presentation_order[number_of_stimuli]);
		}
		// Running through every Stimulus
		for (int stimulus_index = 0; stimulus_index < number_of_stimuli; stimulus_index++){
			// Get the presentation position:
			int present = stimuli_presentation_order[stimulus_index];
			// Get the number of entries for this specific stimulus
			size_t numEnts = numEntries[present];
			if (numEnts > 0){
				temp_test_generator->initialise_device_pointers_for_ents(numEnts, present);
				temp_test_generator->set_threads_per_block_and_blocks_per_grid(threads_per_block);
			}
			// Reset the variables necessary
			neurons->reset_neurons();
			input_neurons->reset_neurons();
			connections->reset_connection_spikes();

			int number_of_timesteps_per_epoch = total_time_per_epoch / timestep;
			float current_time_in_seconds = 0.0f;
		
			for (int timestep_index = 0; timestep_index < number_of_timesteps_per_epoch; timestep_index++){
				
				current_time_in_seconds = float(timestep_index)*float(timestep);
				
				neurons->reset_current_injections();
				
				input_neurons->update_poisson_state_wrapper(timestep);

				// // If there are any spike generators
				// if (numEnts > 0) {
				// 	// Update those neurons corresponding to the Spike Generators
				// 	temp_test_generator->generupdate2_wrapper(current_time_in_seconds, timestep);
				// } 
				
				connections->calculate_postsynaptic_current_injection_for_connection_wrapper(neurons->d_current_injections, current_time_in_seconds);

				connections->apply_ltd_to_connection_weights(neurons->d_last_spike_times, current_time_in_seconds);

				neurons->update_neuron_states(timestep);

				neurons->check_for_neuron_spikes_wrapper(current_time_in_seconds);
				input_neurons->check_for_neuron_spikes_wrapper(current_time_in_seconds);
								
				connections->check_for_synapse_spike_arrival(neurons->d_last_spike_times, input_neurons->d_last_spike_times, current_time_in_seconds);

				connections->apply_ltp_to_connection_weights(neurons->d_last_spike_times, current_time_in_seconds);
				

				// // Only save the spikes if necessary
				if (save_spikes){
					recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch);
					input_recording_electrodes->save_spikes_to_host(current_time_in_seconds, timestep_index, number_of_timesteps_per_epoch);

				}
			}
			if (numEnts > 0){
				// CudaSafeCall(hipFree(d_genids));
				// CudaSafeCall(hipFree(d_gentimes));
			}
		}
		#ifndef QUIETSTART
		clock_t mid = clock();
		if (save_spikes) {
			printf("Epoch %d, Complete.\n Running Time: %f\n Number of Spikes: %d\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC), recording_electrodes->h_total_number_of_spikes);
			printf("Number of Input Spikes: %d\n\n", input_recording_electrodes->h_total_number_of_spikes);
		
		} else {
			printf("Epoch %d, Complete.\n Running Time: %f\n\n", epoch_number, (float(mid-begin) / CLOCKS_PER_SEC));
		}
		#endif
		// Output Spikes list after each epoch:
		// Only save the spikes if necessary
		if (save_spikes){
			recording_electrodes->write_spikes_to_file(neurons, epoch_number);
		}
	}
	
	// SIMULATION COMPLETE!
	#ifndef QUIETSTART
	// Finish the simulation and check time
	clock_t end = clock();
	float timed = float(end-begin) / CLOCKS_PER_SEC;
	printf("Simulation Complete! Time Elapsed: %f\n\n", timed);
	#endif

	recording_electrodes->save_network_state(connections);


	delete neurons;
	delete connections;
	delete recording_electrodes;

	// Free Memory on CPU
	free(recording_electrodes->h_spikestoretimes);
	free(recording_electrodes->h_spikestoreID);

}


// Spike Generator Spike Creation
// INPUT:
//		Population ID
//		Stimulus ID
//		Number of Neurons
//		Number of entries in our arrays
//		Array of generator indices (neuron IDs)
//		Corresponding array of the spike times for each instance
void Simulator::CreateGenerator(int popID, int stimulusid, int spikenumber, int* ids, float* spiketimes){
	// We have to ensure that we have created space for the current stimulus.
	if ((number_of_stimuli - 1) < stimulusid) {
		// Check what the difference is and quit if it is too high
		if ((stimulusid - (number_of_stimuli - 1)) > 1){
			// Error Quit
			printf("Error: Stimuli not created in order. Exiting ...\n");
			exit(-1);
		}
		// If it isn't greater than 1, make space!
		++number_of_stimuli;
		numEntries = (int*)realloc(numEntries, sizeof(int)*number_of_stimuli);
		genids = (int**)realloc(genids, sizeof(int*)*number_of_stimuli);
		gentimes = (float**)realloc(gentimes, sizeof(float*)*number_of_stimuli);
		// Initialize stuff
		genids[stimulusid] = NULL;
		gentimes[stimulusid] = NULL;
		numEntries[stimulusid] = 0;
	}
	// Spike generator populations are necessary
	// Create space for the new ids
	
	genids[stimulusid] = (int*)realloc(genids[stimulusid], 
								sizeof(int)*(spikenumber + numEntries[stimulusid]));
	gentimes[stimulusid] = (float*)realloc(gentimes[stimulusid], 
								sizeof(float)*(spikenumber + numEntries[stimulusid]));
	
	// Check where the neuron population starts
	int startnum = 0;
	if (popID > 0) {
		startnum = neurons->last_neuron_indices_for_each_group[popID-1];
	}
	
	// Assign the genid values according to how many neurons exist already
	for (int i = 0; i < spikenumber; i++){
		genids[stimulusid][numEntries[stimulusid]+i] = ids[i] + startnum;
		gentimes[stimulusid][numEntries[stimulusid]+i] = spiketimes[i];
	}
	// Increment the number of entries the generator population
	numEntries[stimulusid] += spikenumber;
	
}



// // Synapse weight loading
// // INPUT:
// //		Number of weights that you are inputting
// //		The array in which the weights are located
// void Simulator::LoadWeights(int numWeights,
// 						float* newWeights){
// 	// Check if you have the correct number of weights
// 	if (numWeights != synconnects.numconnections){
// 		// Error if not
// 		printf("The number of weights being loaded is not equivalent to the model. Exiting \n");
// 		exit(-1);
// 	}
// 	// Continuing and applying the weights
// 	for (int i=0; i < numWeights; i++){
// 		synconnects.weights[i] = newWeights[i];
// 	}
// }