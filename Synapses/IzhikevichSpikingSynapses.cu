#include "hip/hip_runtime.h"
#include "IzhikevichSpikingSynapses.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"

// IzhikevichSpikingSynapses Constructor
IzhikevichSpikingSynapses::IzhikevichSpikingSynapses() {

}

// IzhikevichSpikingSynapses Destructor
IzhikevichSpikingSynapses::~IzhikevichSpikingSynapses() {
	// Just need to free up the memory
}

// Connection Detail implementation
//	INPUT:
//		Pre-neuron population ID
//		Post-neuron population ID
//		An array of the exclusive sum of neuron populations
//		CONNECTIVITY_TYPE (Constants.h)
//		2 number float array for weight range
//		2 number float array for delay range
//		Boolean value to indicate if population is STDP based
//		Parameter = either probability for random synapses or S.D. for Gaussian
void IzhikevichSpikingSynapses::AddGroup(int presynaptic_group_id, 
						int postsynaptic_group_id, 
						Neurons * neurons,
						Neurons * input_neurons,
						int connectivity_type,
						float weight_range[2],
						int delay_range[2],
						bool stdp_on,
						float parameter,
						float parameter_two) {
	
	
	SpikingSynapses::AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							connectivity_type, 
							weight_range,
							delay_range,
							stdp_on,
							parameter,
							parameter_two);

	for (int i = (total_number_of_synapses - temp_number_of_synapses_in_last_group); i < total_number_of_synapses-1; i++){
		// Setup Delays
		// Get the randoms
		if (delay_range[0] == delay_range[1]) {
			delays[i] = delay_range[0];
		} else {
			float rnddelay = delay_range[0] + (delay_range[1] - delay_range[0])*((float)rand() / (RAND_MAX));
			delays[i] = round(rnddelay);
		}
		// Setup STDP
		if (stdp_on){
			stdp[i] = 1;
		} else {
			stdp[i] = 0;
		}
	}

}

void IzhikevichSpikingSynapses::increment_number_of_synapses(int increment) {

	SpikingSynapses::increment_number_of_synapses(increment);

}


void IzhikevichSpikingSynapses::initialise_device_pointers() {

	SpikingSynapses::initialise_device_pointers();

	reset_synapse_spikes();
}

void IzhikevichSpikingSynapses::reset_synapse_spikes() {

}


void IzhikevichSpikingSynapses::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	SpikingSynapses::set_threads_per_block_and_blocks_per_grid(threads);
	
}


__global__ void apply_ltd_to_synapse_weights_kernal(float* d_lastactive,
							float* d_weights,
							int* d_stdp,
							float* d_lastspiketime,
							int* d_postsyns,
							float currtime,
							struct stdp_struct stdp_vars,
							size_t numConns);

__global__ void apply_ltp_to_synapse_weights_kernal(int* d_postsyns,
							float* d_lastspiketime,
							int* d_stdp,
							float* d_lastactive,
							float* d_weights,
							struct stdp_struct stdp_vars,
							float currtime,
							size_t numConns);



void IzhikevichSpikingSynapses::apply_ltd_to_synapse_weights(float* d_lastspiketime, float current_time_in_seconds) {

	apply_ltd_to_synapse_weights_kernal<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_lastactive,
																	d_weights,
																	d_stdp,
																	d_lastspiketime,
																	d_postsynaptic_neuron_indices,
																	current_time_in_seconds,
																	stdp_vars, // Should make device copy?
																	total_number_of_synapses);

	CudaCheckError();
}


void IzhikevichSpikingSynapses::apply_ltp_to_synapse_weights(float* d_lastspiketime, float current_time_in_seconds) {
	// Carry out the last step, LTP!
	apply_ltp_to_synapse_weights_kernal<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_postsynaptic_neuron_indices,
																	d_lastspiketime,
																	d_stdp,
																	d_lastactive,
																	d_weights,
																	stdp_vars, 
																	current_time_in_seconds,
																	total_number_of_synapses);

	CudaCheckError();
}


__global__ void apply_ltd_to_synapse_weights_kernal(float* d_lastactive,
							float* d_weights,
							int* d_stdp,
							float* d_lastspiketime,
							int* d_postsyns,
							float currtime,
							struct stdp_struct stdp_vars,
							size_t numConns){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < (numConns)) {
		// Get the locations for updating
		// Get the synapses that are to be LTD'd
		if ((d_lastactive[idx] == currtime) && (d_stdp[idx] == 1)) {
			float diff = d_lastspiketime[d_postsyns[idx]] - currtime;
			// STDP Update Rule
			float weightscale = stdp_vars.w_max * stdp_vars.a_minus * expf(diff / stdp_vars.tau_minus);
			// Now scale the weight (using an inverted column/row)
			d_weights[idx] += weightscale; 
		}
	}
}


// LTP on synapses
__global__ void apply_ltp_to_synapse_weights_kernal(int* d_postsyns,
							float* d_lastspiketime,
							int* d_stdp,
							float* d_lastactive,
							float* d_weights,
							struct stdp_struct stdp_vars,
							float currtime,
							size_t numConns) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numConns) {
		// Get the synapses upon which we should do LTP
		// Reversed indexing to check post->pre synapses
		if ((d_lastspiketime[d_postsyns[idx]] == currtime) && (d_stdp[idx] == 1)){
			// Get the last active time / weight of the synapse
			// Calc time difference and weight change
			float diff = currtime - d_lastactive[idx];
			float weightchange = (stdp_vars.w_max - d_weights[idx]) * (stdp_vars.a_plus * expf(-diff / stdp_vars.tau_plus));
			// Update weights
			d_weights[idx] += weightchange;
		}

	}
}