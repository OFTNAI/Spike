#include "hip/hip_runtime.h"
#include "SpikingSynapses.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"

// SpikingSynapses Constructor
SpikingSynapses::SpikingSynapses() {

	delays = NULL;
	stdp = NULL;

	d_delays = NULL;
	d_spikes_travelling_to_synapse = NULL;
	d_stdp = NULL;
	d_time_of_last_spike_to_reach_synapse = NULL;
	d_spikes_travelling_to_synapse_buffer = NULL;
}

// SpikingSynapses Destructor
SpikingSynapses::~SpikingSynapses() {
	// Just need to free up the memory
	// Full Matrices
	free(delays);
	free(stdp);

	CudaSafeCall(hipFree(d_delays));
	CudaSafeCall(hipFree(d_spikes_travelling_to_synapse));
	CudaSafeCall(hipFree(d_stdp));
	CudaSafeCall(hipFree(d_time_of_last_spike_to_reach_synapse));
	CudaSafeCall(hipFree(d_spikes_travelling_to_synapse_buffer));

}

// Connection Detail implementation
//	INPUT:
//		Pre-neuron population ID
//		Post-neuron population ID
//		An array of the exclusive sum of neuron populations
//		CONNECTIVITY_TYPE (Constants.h)
//		2 number float array for weight range
//		2 number float array for delay range
//		Boolean value to indicate if population is STDP based
//		Parameter = either probability for random synapses or S.D. for Gaussian
void SpikingSynapses::AddGroup(int presynaptic_group_id, 
						int postsynaptic_group_id, 
						Neurons * neurons,
						Neurons * input_neurons,
						float timestep,
						synapse_parameters_struct * synapse_params,
						float parameter,
						float parameter_two) {
	
	
	Synapses::AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							timestep,
							synapse_params,
							parameter,
							parameter_two);

	spiking_synapse_parameters_struct * spiking_synapse_group_params = (spiking_synapse_parameters_struct*)synapse_params;

	for (int i = (total_number_of_synapses - temp_number_of_synapses_in_last_group); i < total_number_of_synapses-1; i++){
		
		// Convert delay range from time to number of timesteps
		int delay_range_in_timesteps[2] = {int(round(spiking_synapse_group_params->delay_range[0]/timestep)), int(round(spiking_synapse_group_params->delay_range[1]/timestep))};

		// Check delay range bounds greater than timestep
		if ((delay_range_in_timesteps[0] < 1) || (delay_range_in_timesteps[1] < 1)) {
			printf("%d\n", delay_range_in_timesteps[0]);
			printf("%d\n", delay_range_in_timesteps[1]);
			print_message_and_exit("Delay range must be at least one timestep.");
		}

		// Setup Delays
		if (delay_range_in_timesteps[0] == delay_range_in_timesteps[1]) {
			delays[i] = delay_range_in_timesteps[0];
		} else {
			float random_delay = delay_range_in_timesteps[0] + (delay_range_in_timesteps[1] - delay_range_in_timesteps[0]) * ((float)rand() / (RAND_MAX));
			delays[i] = round(random_delay);
		}

		//Set STDP on or off for synapse
		stdp[i] = spiking_synapse_group_params->stdp_on;
	}

}

void SpikingSynapses::increment_number_of_synapses(int increment) {

	Synapses::increment_number_of_synapses(increment);

    delays = (int*)realloc(delays, total_number_of_synapses * sizeof(int));
    stdp = (bool*)realloc(stdp, total_number_of_synapses * sizeof(bool));

}


void SpikingSynapses::allocate_device_pointers() {

	Synapses::allocate_device_pointers();

	CudaSafeCall(hipMalloc((void **)&d_delays, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_spikes_travelling_to_synapse, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_stdp, sizeof(bool)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_time_of_last_spike_to_reach_synapse, sizeof(float)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_spikes_travelling_to_synapse_buffer, sizeof(int)*total_number_of_synapses));

	CudaSafeCall(hipMemcpy(d_delays, delays, sizeof(int)*total_number_of_synapses, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_stdp, stdp, sizeof(bool)*total_number_of_synapses, hipMemcpyHostToDevice));

}

void SpikingSynapses::reset_synapse_spikes() {
	
	CudaSafeCall(hipMemset(d_spikes_travelling_to_synapse, 0, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMemset(d_time_of_last_spike_to_reach_synapse, -1000.0f, sizeof(float)*total_number_of_synapses));
	CudaSafeCall(hipMemset(d_spikes_travelling_to_synapse_buffer, -1, sizeof(int)*total_number_of_synapses));
}


void SpikingSynapses::shuffle_synapses() {
	
	Synapses::shuffle_synapses();

	int * temp_delays = (int *)malloc(total_number_of_synapses*sizeof(int));
	bool * temp_stdp = (bool *)malloc(total_number_of_synapses*sizeof(bool));
	for(int i = 0; i < total_number_of_synapses; i++) {

		temp_delays[i] = delays[original_synapse_indices[i]];
		temp_stdp[i] = stdp[original_synapse_indices[i]];

	}

	delays = temp_delays;
	stdp = temp_stdp;

}


void SpikingSynapses::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	Synapses::set_threads_per_block_and_blocks_per_grid(threads);
	
}



void SpikingSynapses::check_for_synapse_spike_arrival(float current_time_in_seconds) {

	// printf("check_for_synapse_spike_arrival. number_of_synapse_blocks_per_grid.x: %d. threads_per_block.x: %d\n", number_of_synapse_blocks_per_grid.x, threads_per_block.x);

	check_for_synapse_spike_arrival_kernel<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_spikes_travelling_to_synapse,
																	d_time_of_last_spike_to_reach_synapse,
																	current_time_in_seconds,
																	total_number_of_synapses);

	CudaCheckError();
}

void SpikingSynapses::move_spikes_towards_synapses(float* d_last_spike_time_of_each_neuron, float* d_input_neurons_last_spike_time, float current_time_in_seconds) {

	move_spikes_towards_synapses_kernel<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_presynaptic_neuron_indices,
																		d_delays,
																		d_spikes_travelling_to_synapse,
																		d_last_spike_time_of_each_neuron,
																		d_input_neurons_last_spike_time,
																		d_spikes_travelling_to_synapse_buffer,
																		current_time_in_seconds,
																		total_number_of_synapses,
																		d_time_of_last_spike_to_reach_synapse);

	CudaCheckError();
}



void SpikingSynapses::calculate_postsynaptic_current_injection(SpikingNeurons * neurons, float current_time_in_seconds, float timestep) {

}


__global__ void check_for_synapse_spike_arrival_kernel(int* d_spikes_travelling_to_synapse,
							float* d_time_of_last_spike_to_reach_synapse,
							float current_time_in_seconds,
							size_t total_number_of_synapses){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_synapses) {
		// Decrememnt Spikes
		d_spikes_travelling_to_synapse[idx] -= 1;
		if (d_spikes_travelling_to_synapse[idx] == 0) {
			d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
		}
		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

void SpikingSynapses::update_synaptic_conductances(float timestep, float current_time_in_seconds) {

}

__global__ void move_spikes_towards_synapses_kernel(int* d_presynaptic_neuron_indices,
								int* d_delays,
								int* d_spikes_travelling_to_synapse,
								float* d_last_spike_time_of_each_neuron,
								float* d_input_neurons_last_spike_time,
								int* d_spikes_travelling_to_synapse_buffer,
								float current_time_in_seconds,
								size_t total_number_of_synapses,
								float* d_time_of_last_spike_to_reach_synapse){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_synapses) {


		int timesteps_until_spike_reaches_synapse = d_spikes_travelling_to_synapse[idx];
		timesteps_until_spike_reaches_synapse -= 1;

		if (timesteps_until_spike_reaches_synapse == 0) {
			d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
		}

		if (timesteps_until_spike_reaches_synapse < 0) {

			// Get presynaptic neurons last spike time
			int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
			bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
			float presynaptic_neurons_last_spike_time = presynaptic_is_input ? d_input_neurons_last_spike_time[CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input)] : d_last_spike_time_of_each_neuron[presynaptic_neuron_index];

			if (presynaptic_neurons_last_spike_time == current_time_in_seconds){

				timesteps_until_spike_reaches_synapse = d_delays[idx];

			}
		} 

		d_spikes_travelling_to_synapse[idx] = timesteps_until_spike_reaches_synapse;


		// // Reduce the spikebuffer by 1
		// // d_spikes_travelling_to_synapse_buffer[idx] -= 1;

		// // Get presynaptic neurons last spike time
		// int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
		// bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
		// float presynaptic_neurons_last_spike_time = presynaptic_is_input ? d_input_neurons_last_spike_time[CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input)] : d_last_spike_time_of_each_neuron[presynaptic_neuron_index];

		// // If the presynaptic neuron has JUST fired, add spike to spikes_travelling_to_synapse or buffer, with delay time
		// if (presynaptic_neurons_last_spike_time == current_time_in_seconds){

		// 	if (d_spikes_travelling_to_synapse[idx] <= 0){
				
		// 	} 
		// 	// else if (d_spikes_travelling_to_synapse_buffer[idx] <= 0){
		// 	// 	d_spikes_travelling_to_synapse_buffer[idx] = d_delays[idx];
		// 	// }

		// }

		// // If there is spike in main array has expired, add buffer spike (if exists) to main array. Or set both to negative if no spikes.
		// if (d_spikes_travelling_to_synapse[idx] <= 0) {
		// 	if (d_spikes_travelling_to_synapse_buffer[idx] > 0) {
		// 		d_spikes_travelling_to_synapse[idx] = d_spikes_travelling_to_synapse_buffer[idx];
		// 	} else {
		// 		d_spikes_travelling_to_synapse[idx] = -1;
		// 		d_spikes_travelling_to_synapse_buffer[idx] = -1;
		// 	}

		// }

		// // If the buffer has a smaller time than the spike, switch them
		// if ((d_spikes_travelling_to_synapse_buffer[idx] > 0) && (d_spikes_travelling_to_synapse_buffer[idx] < d_spikes_travelling_to_synapse[idx])){
		// 	int temp = d_spikes_travelling_to_synapse[idx];
		// 	d_spikes_travelling_to_synapse[idx] = d_spikes_travelling_to_synapse_buffer[idx];
		// 	d_spikes_travelling_to_synapse_buffer[idx] = temp;

		// }

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}