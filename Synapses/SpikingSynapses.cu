#include "hip/hip_runtime.h"
#include "SpikingSynapses.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"

// SpikingSynapses Constructor
SpikingSynapses::SpikingSynapses() {

	delays = NULL;
	stdp = NULL;

	d_delays = NULL;
	d_spikes_travelling_to_synapse = NULL;
	d_stdp = NULL;
	d_time_of_last_postsynaptic_activation_for_each_synapse = NULL;
	d_spikes_travelling_to_synapse_buffer = NULL;
}

// SpikingSynapses Destructor
SpikingSynapses::~SpikingSynapses() {
	// Just need to free up the memory
	// Full Matrices
	free(delays);
	free(stdp);

	CudaSafeCall(hipFree(d_delays));
	CudaSafeCall(hipFree(d_spikes_travelling_to_synapse));
	CudaSafeCall(hipFree(d_stdp));
	CudaSafeCall(hipFree(d_time_of_last_postsynaptic_activation_for_each_synapse));
	CudaSafeCall(hipFree(d_spikes_travelling_to_synapse_buffer));

}

// Connection Detail implementation
//	INPUT:
//		Pre-neuron population ID
//		Post-neuron population ID
//		An array of the exclusive sum of neuron populations
//		CONNECTIVITY_TYPE (Constants.h)
//		2 number float array for weight range
//		2 number float array for delay range
//		Boolean value to indicate if population is STDP based
//		Parameter = either probability for random synapses or S.D. for Gaussian
void SpikingSynapses::AddGroup(int presynaptic_group_id, 
						int postsynaptic_group_id, 
						Neurons * neurons,
						Neurons * input_neurons,
						int connectivity_type,
						float weight_range[2],
						int delay_range[2],
						bool stdp_on,
						float parameter,
						float parameter_two) {
	
	
	Synapses::AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							connectivity_type, 
							weight_range,
							delay_range,
							stdp_on,
							parameter,
							parameter_two);

	for (int i = (total_number_of_synapses - temp_number_of_synapses_in_last_group); i < total_number_of_synapses-1; i++){
		// Setup Delays
		// Get the randoms
		if (delay_range[0] == delay_range[1]) {
			delays[i] = delay_range[0];
		} else {
			float rnddelay = delay_range[0] + (delay_range[1] - delay_range[0])*((float)rand() / (RAND_MAX));
			delays[i] = round(rnddelay);
		}
		// Setup STDP
		if (stdp_on){
			stdp[i] = 1;
		} else {
			stdp[i] = 0;
		}
	}

}

void SpikingSynapses::increment_number_of_synapses(int increment) {

	Synapses::increment_number_of_synapses(increment);

    delays = (int*)realloc(delays, total_number_of_synapses * sizeof(int));
    stdp = (int*)realloc(stdp, total_number_of_synapses * sizeof(int));

}


void SpikingSynapses::initialise_device_pointers() {

	Synapses::initialise_device_pointers();

	CudaSafeCall(hipMalloc((void **)&d_delays, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_spikes_travelling_to_synapse, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_stdp, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_time_of_last_postsynaptic_activation_for_each_synapse, sizeof(float)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_spikes_travelling_to_synapse_buffer, sizeof(int)*total_number_of_synapses));

	CudaSafeCall(hipMemcpy(d_delays, delays, sizeof(int)*total_number_of_synapses, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_stdp, stdp, sizeof(int)*total_number_of_synapses, hipMemcpyHostToDevice));

	reset_synapse_spikes();
}

void SpikingSynapses::reset_synapse_spikes() {
	CudaSafeCall(hipMemset(d_spikes_travelling_to_synapse, 0, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMemset(d_time_of_last_postsynaptic_activation_for_each_synapse, -1000.0f, sizeof(float)*total_number_of_synapses));
	CudaSafeCall(hipMemset(d_spikes_travelling_to_synapse_buffer, -1, sizeof(int)*total_number_of_synapses));
}


void SpikingSynapses::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	Synapses::set_threads_per_block_and_blocks_per_grid(threads);
	
}



__global__ void calculate_postsynaptic_current_injection_for_synapse_kernal(int* d_spikes_travelling_to_synapse,
							float* d_synaptic_efficacies_or_weights,
							float* d_time_of_last_postsynaptic_activation_for_each_synapse,
							int* d_postsynaptic_neuron_indices,
							float* d_neurons_current_injections,
							float current_time_in_seconds,
							size_t total_number_of_synapses);

__global__ void check_for_synapse_spike_arrival_kernal(int* d_presynaptic_neuron_indices,
								int* d_delays,
								int* d_spikes_travelling_to_synapse,
								float* d_neurons_last_spike_time,
								float* d_input_neurons_last_spike_time,
								int* d_spikes_travelling_to_synapse_buffer,
								float currtime,
								size_t total_number_of_synapses);



void SpikingSynapses::calculate_postsynaptic_current_injection_for_synapse(float* d_neurons_current_injections, float current_time_in_seconds) {

	calculate_postsynaptic_current_injection_for_synapse_kernal<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_spikes_travelling_to_synapse,
																	d_synaptic_efficacies_or_weights,
																	d_time_of_last_postsynaptic_activation_for_each_synapse,
																	d_postsynaptic_neuron_indices,
																	d_neurons_current_injections,
																	current_time_in_seconds,
																	total_number_of_synapses);

	CudaCheckError();
}

void SpikingSynapses::check_for_synapse_spike_arrival(float* d_neurons_last_spike_time, float* d_input_neurons_last_spike_time, float current_time_in_seconds) {

	check_for_synapse_spike_arrival_kernal<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_presynaptic_neuron_indices,
																		d_delays,
																		d_spikes_travelling_to_synapse,
																		d_neurons_last_spike_time,
																		d_input_neurons_last_spike_time,
																		d_spikes_travelling_to_synapse_buffer,
																		current_time_in_seconds,
																		total_number_of_synapses);

	CudaCheckError();
}

void SpikingSynapses::apply_ltd_to_synapse_weights(float* d_neurons_last_spike_time, float current_time_in_seconds) {

}


void SpikingSynapses::apply_ltp_to_synapse_weights(float* d_neurons_last_spike_time, float current_time_in_seconds) {

}



// If spike has reached synapse add synapse weight to postsyn current injection
// Was currentcalc
__global__ void calculate_postsynaptic_current_injection_for_synapse_kernal(int* d_spikes_travelling_to_synapse,
							float* d_synaptic_efficacies_or_weights,
							float* d_time_of_last_postsynaptic_activation_for_each_synapse,
							int* d_postsynaptic_neuron_indices,
							float* d_neurons_current_injections,
							float current_time_in_seconds,
							size_t total_number_of_synapses){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < (total_number_of_synapses)) {
		// Decrememnt Spikes
		d_spikes_travelling_to_synapse[idx] -= 1;
		if (d_spikes_travelling_to_synapse[idx] == 0) {

			atomicAdd(&d_neurons_current_injections[d_postsynaptic_neuron_indices[idx]], d_synaptic_efficacies_or_weights[idx]);

			d_time_of_last_postsynaptic_activation_for_each_synapse[idx] = current_time_in_seconds;
		}
	}
	__syncthreads();
}


__global__ void check_for_synapse_spike_arrival_kernal(int* d_presynaptic_neuron_indices,
								int* d_delays,
								int* d_spikes_travelling_to_synapse,
								float* d_neurons_last_spike_time,
								float* d_input_neurons_last_spike_time,
								int* d_spikes_travelling_to_synapse_buffer,
								float current_time_in_seconds,
								size_t total_number_of_synapses){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < total_number_of_synapses) {
		// Reduce the spikebuffer by 1
		d_spikes_travelling_to_synapse_buffer[idx] -= 1;

		int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
		float presynaptic_neurons_last_spike_time;
		if (presynaptic_neuron_index < 0) {
			presynaptic_neurons_last_spike_time = d_input_neurons_last_spike_time[-1*presynaptic_neuron_index - 1];
		} else {
			presynaptic_neurons_last_spike_time = d_neurons_last_spike_time[presynaptic_neuron_index];
		}

		// Check if the neuron PRE has just fired and if the synapse exists
		if (presynaptic_neurons_last_spike_time == current_time_in_seconds){
			// Update the spikes with the correct delay
			if (d_spikes_travelling_to_synapse[idx] <= 0){
				d_spikes_travelling_to_synapse[idx] = d_delays[idx];
			} else if (d_spikes_travelling_to_synapse_buffer[idx] <= 0){
				d_spikes_travelling_to_synapse_buffer[idx] = d_delays[idx];
			}
		}
		// If there is no waiting spike
		if (d_spikes_travelling_to_synapse[idx] <= 0) {
			// Use the buffer if necessary
			if (d_spikes_travelling_to_synapse_buffer[idx] > 0) {
				d_spikes_travelling_to_synapse[idx] = d_spikes_travelling_to_synapse_buffer[idx];
			} else {
				d_spikes_travelling_to_synapse[idx] = -1;
				d_spikes_travelling_to_synapse_buffer[idx] = -1;
			}
		}
		// If the buffer has a smaller time than the spike, switch them
		if ((d_spikes_travelling_to_synapse_buffer[idx] > 0) && (d_spikes_travelling_to_synapse_buffer[idx] < d_spikes_travelling_to_synapse[idx])){
			int temp = d_spikes_travelling_to_synapse[idx];
			d_spikes_travelling_to_synapse[idx] = d_spikes_travelling_to_synapse_buffer[idx];
			d_spikes_travelling_to_synapse_buffer[idx] = temp;
		}
	}
}